#include "hip/hip_runtime.h"
#include "s_pyramid.h"

#include "gauss_filter.h"
#include "clamp.h"
#include "debug_macros.h"
#include "assist.h"
#include <hip/hip_runtime.h>

/*************************************************************
 * V11: device side
 *************************************************************/

#define V11_EDGE_LEN    32
#define V11_RANGE    4 // RANGES from 1 to 8 are possible
#define V11_GAUSS_BASE   ( GAUSS_ONE_SIDE_RANGE - V11_RANGE )
#define V11_FILTERSIZE   ( V11_RANGE + 1        + V11_RANGE )
#define V11_LEVELS       _levels

namespace popart {

__global__
void filter_gauss_horiz_v11( Plane2D_float src_data,
                             Plane2D_float dst_data )
{

    __shared__ float loaddata[V11_EDGE_LEN][V11_RANGE + V11_EDGE_LEN + V11_RANGE];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int block_x = blockIdx.x * V11_EDGE_LEN;
    int block_y = blockIdx.y * V11_EDGE_LEN;
    int idx     = threadIdx.x;
    int idy     = threadIdx.y;
    for( ; idx < V11_EDGE_LEN+2*V11_RANGE; idx += V11_EDGE_LEN) {
        int read_x = clamp( block_x + idx - V11_RANGE, src_w );
        int read_y = clamp( block_y + idy,            src_h );
        loaddata[idy][idx] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = loaddata[threadIdx.y][idx+V11_RANGE];
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = loaddata[threadIdx.y][idx+V11_RANGE];
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = loaddata[threadIdx.y][idx+V11_RANGE];
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    if( idx >= src_w ) return;
    if( idy >= src_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void filter_gauss_horiz_v11( hipTextureObject_t src_data,
                             Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void filter_gauss_horiz_v11_by_2( hipTextureObject_t src_data,
                                  Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}


__device__ inline
float filter_gauss_vert_v11_sub( hipTextureObject_t src_data,
                                 Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return 0;
    if( idy >= dst_h ) return 0;

    dst_data.ptr(idy)[idx] = out;

    return out;
}

__global__
void filter_gauss_vert_v11( hipTextureObject_t src_data,
                            Plane2D_float       dst_data )
{
    filter_gauss_vert_v11_sub( src_data, dst_data );
}

__global__
void filter_gauss_vert_v11_dog( hipTextureObject_t src_data,
                                Plane2D_float       dst_data,
                                hipTextureObject_t top_data,
                                Plane2D_float       dog_data )
{
    float b = filter_gauss_vert_v11_sub( src_data, dst_data );

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float a;
    a = tex2D<float>( top_data, idx, idy );
    a = fabs( a - b );

    const int width  = dog_data.getWidth();
    const int height = dog_data.getHeight();
    if( idx >= width ) return;
    if( idy >= height ) return;

    dog_data.ptr(idy)[idx] = a;
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_v11( Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    for( int octave=0; octave<_num_octaves; octave++ ) {
        for( int level=0; level<V11_LEVELS; level++ ) {
#if 0
        cerr << "Configuration for octave " << octave << endl
             << "  Horiz: layer size: "
             << _octaves[octave].getData(level).getWidth() << "x" << _octaves[octave].getData(level).getHeight() << endl
             << "  Vert: layer size: "
             << _octaves[octave].getIntermediateData().getWidth() << "x" << _octaves[octave].getIntermediateData().getHeight() << endl
             << "  grid: "
             << "(" << grid.x << "," << grid.y << "," << grid.z << ")"
             << " block: "
             << "(" << block.x << "," << block.y << "," << block.z << ")" << endl;
#endif

            const int width  = _octaves[octave].getData(0).getWidth();
            const int height = _octaves[octave].getData(0).getHeight();

            dim3 h_block( 64, 2 );
            dim3 h_grid;
            h_grid.x = grid_divide( width,  h_block.x );
            h_grid.y = grid_divide( height, h_block.y );

            dim3 v_block( 64, 2 );
            dim3 v_grid;
            v_grid.x = grid_divide( width,  v_block.x );
            v_grid.y = grid_divide( height, v_block.y );

            dim3 d_block( 32, 1 );
            dim3 d_grid;
            d_grid.x = grid_divide( width,  d_block.x );
            d_grid.y = grid_divide( height, d_block.y );

            if( level == 0 ) {
                if( octave == 0 ) {
                    dim3 block;
                    block.x = V11_EDGE_LEN;
                    block.y = V11_EDGE_LEN;

                    dim3 grid;
                    const int width  = _octaves[octave].getData(0).getWidth();
                    const int height = _octaves[octave].getData(0).getHeight();
                    grid.x = grid_divide( width,  V11_EDGE_LEN );
                    grid.y = grid_divide( height, V11_EDGE_LEN );

                    filter_gauss_horiz_v11
                        <<<grid,block>>>
                        ( base->array,
                          _octaves[octave].getIntermediateData( ) );
                } else {
                    filter_gauss_horiz_v11_by_2
                        <<<h_grid,h_block>>>
                        ( _octaves[octave-1]._data_tex[ V11_LEVELS-3 ],
                          _octaves[octave].getIntermediateData( ) );
                }
            } else {
                filter_gauss_horiz_v11
                    <<<h_grid,h_block>>>
                    ( _octaves[octave]._data_tex[ level-1 ],
                      _octaves[octave].getIntermediateData( ) );
            }

            if( level == 0 ) {
                filter_gauss_vert_v11
                    <<<v_grid,v_block>>>
                    ( _octaves[octave]._interm_data_tex,
                      _octaves[octave].getData( level ) );
            } else {
                filter_gauss_vert_v11_dog
                    <<<d_grid,d_block>>>
                    ( _octaves[octave]._interm_data_tex,
                      _octaves[octave].getData( level ),
                      _octaves[octave]._data_tex[level-1],
                      _octaves[octave].getDogData( level-1 ) );
            }
        }
    }
    hipDeviceSynchronize( );
    hipError_t err = hipGetLastError();
    POP_CUDA_FATAL_TEST( err, "filter_gauss_horiz_v11 failed: " );
}

} // namespace popart

