#include "hip/hip_runtime.h"
#include "s_pyramid.h"

#include "write_plane_2d.h"
#include "gauss_filter.h"
#include "clamp.h"
#include "debug_macros.h"
#include "assist.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <stdio.h>

/*************************************************************
 * V11: device side
 *************************************************************/

#define V11_EDGE_LEN 32

#define HORIZ_NO_SHARED_128x1

namespace popart {

__global__
void filter_gauss_horiz_tex_128x1( hipTextureObject_t src_data,
                                   Plane2D_float       dst_data,
                                   int                 level )
{
    const float dst_w  = dst_data.getWidth();
    const float dst_h  = dst_data.getHeight();
    const float read_y = ( blockIdx.y + 0.5 ) / dst_h;

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];
        const float read_x_l = ( off_x - offset );
        const float  v1 = tex2D<float>( src_data, ( read_x_l + 0.5 ) / dst_w, read_y );
        out += ( v1 * g );

        const float read_x_r = ( off_x + offset );
        const float  v2 = tex2D<float>( src_data, ( read_x_r + 0.5 ) / dst_w, read_y );
        out += ( v2 * g );
    }
    const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    const float read_x = off_x;
    const float v3 = tex2D<float>( src_data, ( read_x + 0.5 ) / dst_w, read_y );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out;
}


__global__
void filter_gauss_horiz_v11_128x1( hipTextureObject_t src_data,
                                   Plane2D_float       dst_data,
                                   int                 level )
{
    const int dst_w = dst_data.getWidth();

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];
        const float  v1 = tex2D<float>( src_data, off_x - offset + 0.5, blockIdx.y + 0.5 );
        out += ( v1 * g );

        const float  v2 = tex2D<float>( src_data, off_x + offset + 0.5, blockIdx.y + 0.5 );
        out += ( v2 * g );
    }
    const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    const float v3 = tex2D<float>( src_data, off_x+0.5, blockIdx.y+0.5 );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out;
}


#if 0
__global__
void filter_gauss_horiz_v11( hipTextureObject_t src_data,
                             Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}
#endif

__global__
void get_by_2( hipTextureObject_t src_data,
               Plane2D_float       dst_data,
               int level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const float val = tex2D<float>( src_data, 2.0f * idx + 1.0f, 2.0f * idy + 1.0f );
    dst_data.ptr(idy)[idx] = val;
}

__global__
void filter_gauss_horiz_v11_by_2( hipTextureObject_t src_data,
                                  Plane2D_float       dst_data,
                                  int level )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idx = threadIdx.x - offset;
        // add +1.0f because we must shift by 0.5 pixels upscaled by 2 in the previous octave
        val = tex2D<float>( src_data, 2 * ( block_x + idx ) + 1.0, 2 * ( block_y + idy ) + 1.0 );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ) + 1.0, 2 * ( block_y + idy ) + 1.0 );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, 2 * ( block_x + idx ) + 1.0, 2 * ( block_y + idy ) + 1.0 );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

    //input texture (src_data) has twize the size of dst_data.
    //the block and thread dimensions are that of dst_data.
#if 0
__global__
void downscale_by_2(Plane2D_float src_data,
                    Plane2D_float dst_data)
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy     = threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    //todo: cant do tex2d lookup in Plane2D_float array (not texture memory).
    //      Need to either use another input buffer, or change to slower global memory lookup.
    //add 0.5f to lookup coords to get interpolated values? Does it work here?
    dst_data.ptr(idy)[idx] = tex2D<float>( src_data,
                                           2 * ( block_x + idx ),
                                           2 * ( block_y + idy ));
}
#endif
__global__
void filter_gauss_vert_v11( hipTextureObject_t src_data,
                            Plane2D_float       dst_data,
                            int level )
{
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5, block_y + idy + 0.5 );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5, block_y + idy + 0.5 );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx + 0.5, block_y + idy + 0.5 );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}


__global__
void make_dog( hipTextureObject_t this_data,
               hipTextureObject_t top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const float b = tex2D<float>( this_data, idx, idy );
    const float a = tex2D<float>( top_data, idx, idy );
    const float c = a - b; // c = fabs( a - b );

    surf2DLayeredwrite( c, dog_data, idx*4, idy, level, hipBoundaryModeZero );
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_v11( Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

#if 0
    //Creating the octaves
    for(uint32_t octave=0; octave<_num_octaves; octave++){
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        dim3 h_block( 64, 2 );
        dim3 h_grid;

        h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
        h_grid.y = (unsigned int)grid_divide( height, h_block.y );

        dim3 v_block( 64, 2 );
        dim3 v_grid;
        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

        dim3 d_block( 32, 1 );
        dim3 d_grid;
        d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
        d_grid.y = (unsigned int)grid_divide( height, d_block.y );

        if(octave==0){
            downscale_by_2<<<h_grid,h_block>>>(base->array,
                                               _octaves[octave  ].getData(0));
        }else{
            downscale_by_2<<<h_grid,h_block>>>(_octaves[octave-1].getData(0),
                                               _octaves[octave  ].getData(0));
        }

    }

    //Performing the gaussing
    for(uint32_t octave=0; octave<_num_octaves; octave++) {
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        dim3 h_block( 64, 2 );
        dim3 h_grid;

        h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
        h_grid.y = (unsigned int)grid_divide( height, h_block.y );

        dim3 v_block( 64, 2 );
        dim3 v_grid;
        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

        dim3 d_block( 32, 1 );
        dim3 d_grid;
        d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
        d_grid.y = (unsigned int)grid_divide( height, d_block.y );

        //horizontal
        //  input : _data_tex[level-1]
        //  output: getIntermediateData();
        //vertical:
        //  input : _interm_data_tex
        //  output:
        //      lvl0 : getData(level)
        //      lvl>0: getDogSurface()
        for (uint32_t level = 0; level < _levels; level++) {
            if(level == 0){
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
            }


            if( level == 0 ) {
                if( octave == 0 ) {
                    dim3 block(V11_EDGE_LEN,V11_EDGE_LEN);
                    dim3 grid((unsigned int)grid_divide( width,  V11_EDGE_LEN ),
                              (unsigned int)grid_divide( height, V11_EDGE_LEN ));

                    filter_gauss_horiz_v11 <<<grid,block>>> (
                        _octaves[octave]._data_tex[level-1],
                        _octaves[octave].getIntermediateData() );
                } else {
                    filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                        _octaves[octave-1]._data_tex[ _levels-3 ],
                        _octaves[octave].getIntermediateData( ) );
                }
            }
            else {
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                        _octaves[octave]._data_tex[ level-1 ],
                                _octaves[octave].getIntermediateData( ) );
            }



            if( level == 0 ) {
                filter_gauss_vert_v11 <<<v_grid,v_block>>> (
                        _octaves[octave]._interm_data_tex,
                                _octaves[octave].getData( level ) );
            }
            else {
                filter_gauss_vert_v11_dog <<<d_grid,d_block>>> (
                        _octaves[octave]._interm_data_tex,
                                _octaves[octave].getData( level ),
                                _octaves[octave]._data_tex[level-1],
                                _octaves[octave].getDogSurface( ),
                                level-1 );
            }
        }
    }

#else

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
        for( uint32_t level=0; level<_levels; level++ ) {

            const int width  = _octaves[octave].getData(0).getWidth();
            const int height = _octaves[octave].getData(0).getHeight();

            Octave&      oct_obj   = _octaves[octave];
            hipStream_t oct_str_0 = oct_obj.getStream(0);

            if( level == 0 ) {
                if( _scaling_mode == Config::DirectDownscaling ) {
                    dim3 block( 128, 1 );
                    dim3 grid;
                    grid.x  = grid_divide( width,  128 );
                    grid.y  = height;
                    filter_gauss_horiz_tex_128x1
                        <<<grid,block,0,oct_str_0>>>
                        ( base->getUpscaledTexture(),
                          oct_obj.getIntermediateData( ),
                          level );
                } else {
                    if( octave == 0 ) {
#if 0
                        dim3 block( 32, 1 );
                        dim3 grid;
                        grid.x  = grid_divide( width,  128 );
                        grid.y  = height;
                        filter_gauss_horiz_v11
                            <<<grid,block,0,oct_str_0>>>
                            ( base->array,
                            oct_obj.getIntermediateData( ) );
#else
                        dim3 block( 128, 1 );
                        dim3 grid;
                        grid.x  = grid_divide( width,  128 );
                        grid.y  = height;
                        filter_gauss_horiz_tex_128x1
                            <<<grid,block,0,oct_str_0>>>
                            ( base->getUpscaledTexture(),
                              oct_obj.getIntermediateData( ),
                              level );
#endif
                    } else {
#define PREV_LEVEL 3
// #define PREV_LEVEL 5
                        Octave& prev_oct_obj  = _octaves[octave-1];
                        hipStreamWaitEvent( oct_str_0, prev_oct_obj.getEventGaussDone( _levels-PREV_LEVEL ), 0 );

                        cout << "We are blurring from level " << _levels-PREV_LEVEL << endl;
                        if( _scaling_mode == Config::IndirectUnfilteredDownscaling ) {
                            dim3 h_block( 64, 2 );
                            dim3 h_grid;
                            h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
                            h_grid.y = (unsigned int)grid_divide( height, h_block.y );

                            get_by_2
                                <<<h_grid,h_block,0,oct_str_0>>>
                                ( prev_oct_obj._data_tex[ _levels-PREV_LEVEL ],
                                  oct_obj.getData( level ),
                                  level );
                        } else if( _scaling_mode == Config::IndirectDownscaling ) {
                            dim3 h_block( 64, 2 );
                            dim3 h_grid;
                            h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
                            h_grid.y = (unsigned int)grid_divide( height, h_block.y );

                            filter_gauss_horiz_v11_by_2
                                <<<h_grid,h_block,0,oct_str_0>>>
                                ( prev_oct_obj._data_tex[ _levels-PREV_LEVEL ],
                                  oct_obj.getIntermediateData( ),
                                  level );
                        } else {
                            cerr << __FILE__ << ":" << __LINE__ << ": unknown scaling mode" << endl;
                        }
                    }
                }
            } else {
#if 0
                dim3 h_block( 64, 2 );
                dim3 h_grid;
                h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
                h_grid.y = (unsigned int)grid_divide( height, h_block.y );

                filter_gauss_horiz_v11
                    <<<h_grid,h_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[ level-1 ],
                      oct_obj.getIntermediateData( ) );
#else
                // const int width  = _octaves[octave].getData(0).getWidth();
                // const int height = _octaves[octave].getData(0).getHeight();
                dim3 block( 128, 1 );
                dim3 grid;
                grid.x  = grid_divide( width,  128 );
                grid.y  = height;
                filter_gauss_horiz_v11_128x1
                    <<<grid,block,0,oct_str_0>>>
                    ( oct_obj._data_tex[ level-1 ],
                      oct_obj.getIntermediateData( ),
                      level );
#endif
            }

            if( level == 0 ) {
                switch( _scaling_mode )
                {
                case Config::IndirectUnfilteredDownscaling :
                    if( octave != 0 )
                        break;
                case Config::DirectDownscaling :
                case Config::IndirectDownscaling :
                    {
                        dim3 v_block( 64, 2 );
                        dim3 v_grid;
                        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
                        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

                        filter_gauss_vert_v11
                            <<<v_grid,v_block,0,oct_str_0>>>
                            ( oct_obj._interm_data_tex,
                              oct_obj.getData( level ),
                              level );
                    }
                    break;
                default :
                    cerr << __FILE__ << ":" << __LINE__ << ": Missing scaling mode" << endl;
                    exit( -1 );
                }
            } else {
                dim3 v_block( 64, 2 );
                dim3 v_grid;
                v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
                v_grid.y = (unsigned int)grid_divide( height, v_block.y );

                filter_gauss_vert_v11
                    <<<v_grid,v_block,0,oct_str_0>>>
                    ( oct_obj._interm_data_tex,
                      oct_obj.getData( level ),
                      level );

                dim3 e_block( 128, 2 );
                dim3 e_grid;
                e_grid.x = grid_divide( width,  e_block.x );
                e_grid.y = grid_divide( height, e_block.y );

                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );
            }

            hipEventRecord( oct_obj.getEventGaussDone( level ), oct_str_0 );
        }
    }
#endif
}

} // namespace popart

