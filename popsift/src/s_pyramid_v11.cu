#include "hip/hip_runtime.h"
#include "s_pyramid.h"

#include "write_plane_2d.h"
#include "gauss_filter.h"
#include "clamp.h"
#include "debug_macros.h"
#include "assist.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <stdio.h>

/*************************************************************
 * V11: device side
 *************************************************************/

#define V11_EDGE_LEN 32

#define HORIZ_NO_SHARED_128x1

namespace popart {

__global__
void filter_gauss_horiz_v11_128x1( Plane2D_float src_data,
                                   Plane2D_float dst_data,
                                   int level )
{
    __shared__ float loaddata[GAUSS_SPAN + 128 + GAUSS_SPAN];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int       idx    = threadIdx.x;
    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;
    int       read_x;
    int       read_y = clamp( blockIdx.y, src_h );
    if( idx < GAUSS_SPAN ) {
        read_x = clamp( off_x - GAUSS_SPAN, src_w );
        loaddata[idx] = src_data.ptr(read_y)[read_x];
    } else if( idx >= 128-GAUSS_SPAN ) {
        read_x = clamp( off_x + GAUSS_SPAN, src_w );
        loaddata[idx+2*GAUSS_SPAN] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();
    read_x = clamp( off_x, src_w );
    loaddata[idx+GAUSS_SPAN] = src_data.ptr(read_y)[read_x];
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN+offset];
        val = loaddata[threadIdx.x+GAUSS_SPAN-offset];
        out += ( val * g );
        val = loaddata[threadIdx.x+GAUSS_SPAN+offset];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    val = loaddata[threadIdx.x+4];
    out += ( val * g );

    __syncthreads();

    if( off_x >= src_w )      return;
    if( blockIdx.y >= src_h ) return;

    dst_data.ptr(blockIdx.y)[off_x] = out;
}

__global__
void filter_gauss_horiz_v11_128x1_no_shared( Plane2D_float src_data,
                                             Plane2D_float dst_data,
                                             int level )
{
    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    const int off_x  = blockIdx.x * blockDim.x + threadIdx.x;
    int       read_x;
    const int read_y = clamp( blockIdx.y, src_h );

    if( off_x >= src_w )      return;
    if( blockIdx.y >= src_h ) return;

    float g;
    float val;
    float out = 0;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN+offset];

        read_x = clamp( off_x - offset, src_w );
        val = src_data.ptr(read_y)[read_x];
        out += ( val * g );

        read_x = clamp( off_x + offset, src_w );
        val = src_data.ptr(read_y)[read_x];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];

    read_x = clamp( off_x, src_w );
    val = src_data.ptr(read_y)[read_x];
    out += ( val * g );

    dst_data.ptr(blockIdx.y)[off_x] = out;
}
#if 0
__global__
void filter_gauss_horiz_v11( Plane2D_float src_data,
                             Plane2D_float dst_data )
{
    __shared__ float loaddata[V11_EDGE_LEN][V11_RANGE + V11_EDGE_LEN + V11_RANGE];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int idx     = threadIdx.x;
    int idy     = threadIdx.y;
    for( ; idx < V11_EDGE_LEN+2*V11_RANGE; idx += V11_EDGE_LEN) {
        int read_x = clamp( blockIdx.x * blockDim.x + idx - V11_RANGE, src_w );
        int read_y = clamp( blockIdx.y * blockDim.y + idy,             src_h );
        loaddata[idy][idx] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];
        val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE-offset];
        out += ( val * g );
        val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE+offset];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE];
    out += ( val * g );

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    idy = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= src_w ) return;
    if( idy >= src_h ) return;

    dst_data.ptr(idy)[idx] = out;
}
#endif

__global__
void filter_gauss_horiz_v11_128x1( hipTextureObject_t src_data,
                                   Plane2D_float       dst_data,
                                   int level )
{
    __shared__ float loaddata[GAUSS_SPAN + 128 + GAUSS_SPAN];

    const int idx   = threadIdx.x;
    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( idx < GAUSS_SPAN ) {
        loaddata[idx] = tex2D<float>( src_data, off_x-GAUSS_SPAN, blockIdx.y );
    } else if( idx >= 128-GAUSS_SPAN ) {
        loaddata[idx+2*GAUSS_SPAN] = tex2D<float>( src_data, off_x+GAUSS_SPAN, blockIdx.y );
    }
    __syncthreads();
    loaddata[idx+GAUSS_SPAN] = tex2D<float>( src_data, off_x, blockIdx.y );
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];
        val = loaddata[threadIdx.x+GAUSS_SPAN-offset];
        out += ( val * g );
        val = loaddata[threadIdx.x+GAUSS_SPAN+offset];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    val = loaddata[threadIdx.x+GAUSS_SPAN];
    out += ( val * g );

    const int dst_w = dst_data.getWidth();
    if( off_x >= dst_w )      return;

    dst_data.ptr(blockIdx.y)[off_x] = out;
}

__global__
void filter_gauss_horiz_v11_128x1_no_shared( hipTextureObject_t src_data,
                                             Plane2D_float       dst_data,
                                             int level )
{
    const int dst_w = dst_data.getWidth();

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];
        const float  v1 = tex2D<float>( src_data, off_x - offset, blockIdx.y );
        out += ( v1 * g );

        const float  v2 = tex2D<float>( src_data, off_x + offset, blockIdx.y );
        out += ( v2 * g );
    }
    const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    const float v3 = tex2D<float>( src_data, off_x, blockIdx.y );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out;
}

#if 0
__global__
void filter_gauss_horiz_v11( hipTextureObject_t src_data,
                             Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}
#endif

__global__
void filter_gauss_horiz_v11_by_2( hipTextureObject_t src_data,
                                  Plane2D_float       dst_data,
                                  int level )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

    //input texture (src_data) has twize the size of dst_data.
    //the block and thread dimensions are that of dst_data.
#if 0
__global__
void downscale_by_2(Plane2D_float src_data,
                    Plane2D_float dst_data)
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy     = threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    //todo: cant do tex2d lookup in Plane2D_float array (not texture memory).
    //      Need to either use another input buffer, or change to slower global memory lookup.
    //add 0.5f to lookup coords to get interpolated values? Does it work here?
    dst_data.ptr(idy)[idx] = tex2D<float>( src_data,
                                           2 * ( block_x + idx ),
                                           2 * ( block_y + idy ));
}
#endif
__global__
void filter_gauss_vert_v11( hipTextureObject_t src_data,
                            Plane2D_float       dst_data,
                            int level )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}


__global__
void make_dog( hipTextureObject_t this_data,
               hipTextureObject_t top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const float b = tex2D<float>( this_data, idx, idy );
    const float a = tex2D<float>( top_data, idx, idy );
    const float c = a - b; // c = fabs( a - b );

    surf2DLayeredwrite( c, dog_data, idx*4, idy, level, hipBoundaryModeZero );
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_v11( Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

#if 0
    //Creating the octaves
    for(uint32_t octave=0; octave<_num_octaves; octave++){
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        dim3 h_block( 64, 2 );
        dim3 h_grid;

        h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
        h_grid.y = (unsigned int)grid_divide( height, h_block.y );

        dim3 v_block( 64, 2 );
        dim3 v_grid;
        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

        dim3 d_block( 32, 1 );
        dim3 d_grid;
        d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
        d_grid.y = (unsigned int)grid_divide( height, d_block.y );

        if(octave==0){
            downscale_by_2<<<h_grid,h_block>>>(base->array,
                                               _octaves[octave  ].getData(0));
        }else{
            downscale_by_2<<<h_grid,h_block>>>(_octaves[octave-1].getData(0),
                                               _octaves[octave  ].getData(0));
        }

    }

    //Performing the gaussing
    for(uint32_t octave=0; octave<_num_octaves; octave++) {
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        dim3 h_block( 64, 2 );
        dim3 h_grid;

        h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
        h_grid.y = (unsigned int)grid_divide( height, h_block.y );

        dim3 v_block( 64, 2 );
        dim3 v_grid;
        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

        dim3 d_block( 32, 1 );
        dim3 d_grid;
        d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
        d_grid.y = (unsigned int)grid_divide( height, d_block.y );

        //horizontal
        //  input : _data_tex[level-1]
        //  output: getIntermediateData();
        //vertical:
        //  input : _interm_data_tex
        //  output:
        //      lvl0 : getData(level)
        //      lvl>0: getDogSurface()
        for (uint32_t level = 0; level < _levels; level++) {
            if(level == 0){
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
            }


            if( level == 0 ) {
                if( octave == 0 ) {
                    dim3 block(V11_EDGE_LEN,V11_EDGE_LEN);
                    dim3 grid((unsigned int)grid_divide( width,  V11_EDGE_LEN ),
                              (unsigned int)grid_divide( height, V11_EDGE_LEN ));

                    filter_gauss_horiz_v11 <<<grid,block>>> (
                        _octaves[octave]._data_tex[level-1],
                        _octaves[octave].getIntermediateData() );
                } else {
                    filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                        _octaves[octave-1]._data_tex[ _levels-3 ],
                        _octaves[octave].getIntermediateData( ) );
                }
            }
            else {
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                        _octaves[octave]._data_tex[ level-1 ],
                                _octaves[octave].getIntermediateData( ) );
            }



            if( level == 0 ) {
                filter_gauss_vert_v11 <<<v_grid,v_block>>> (
                        _octaves[octave]._interm_data_tex,
                                _octaves[octave].getData( level ) );
            }
            else {
                filter_gauss_vert_v11_dog <<<d_grid,d_block>>> (
                        _octaves[octave]._interm_data_tex,
                                _octaves[octave].getData( level ),
                                _octaves[octave]._data_tex[level-1],
                                _octaves[octave].getDogSurface( ),
                                level-1 );
            }
        }
    }

#else

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
        for( uint32_t level=0; level<_levels; level++ ) {

            const int width  = _octaves[octave].getData(0).getWidth();
            const int height = _octaves[octave].getData(0).getHeight();

            Octave&      oct_obj   = _octaves[octave];
            hipStream_t oct_str_0 = oct_obj.getStream(0);

            if( level == 0 ) {
                if( octave == 0 ) {
#if 0
                    dim3 block( 32, 1 );
                    dim3 grid;
                    grid.x  = grid_divide( width,  128 );
                    grid.y  = height;
                    filter_gauss_horiz_v11
                        <<<grid,block,0,oct_str_0>>>
                        ( base->array,
                          oct_obj.getIntermediateData( ) );
#else
                    dim3 block( 128, 1 );
                    dim3 grid;
                    grid.x  = grid_divide( width,  128 );
                    grid.y  = height;
#ifdef HORIZ_NO_SHARED_128x1
                    filter_gauss_horiz_v11_128x1_no_shared
                        <<<grid,block,0,oct_str_0>>>
                        ( base->array,
                          oct_obj.getIntermediateData( ),
                          level );
#else // HORIZ_NO_SHARED_128x1
                    filter_gauss_horiz_v11_128x1
                        <<<grid,block,0,oct_str_0>>>
                        ( base->array,
                          oct_obj.getIntermediateData( ),
                          level );
#endif // HORIZ_NO_SHARED_128x1
#endif
                } else {
                    dim3 h_block( 64, 2 );
                    dim3 h_grid;
                    h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
                    h_grid.y = (unsigned int)grid_divide( height, h_block.y );

                    Octave& prev_oct_obj  = _octaves[octave-1];
                    hipStreamWaitEvent( oct_str_0, prev_oct_obj.getEventGaussDone( _levels-3 ), 0 );

                    filter_gauss_horiz_v11_by_2
                        <<<h_grid,h_block,0,oct_str_0>>>
                        ( prev_oct_obj._data_tex[ _levels-3 ],
                          // _octaves[octave-1]._data_tex[ 0 ],
                          oct_obj.getIntermediateData( ),
                          level );
                }
            } else {
#if 0
                dim3 h_block( 64, 2 );
                dim3 h_grid;
                h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
                h_grid.y = (unsigned int)grid_divide( height, h_block.y );

                filter_gauss_horiz_v11
                    <<<h_grid,h_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[ level-1 ],
                      oct_obj.getIntermediateData( ) );
#else
                // const int width  = _octaves[octave].getData(0).getWidth();
                // const int height = _octaves[octave].getData(0).getHeight();
                dim3 block( 128, 1 );
                dim3 grid;
                grid.x  = grid_divide( width,  128 );
                grid.y  = height;
#ifdef HORIZ_NO_SHARED_128x1
                filter_gauss_horiz_v11_128x1_no_shared
                    <<<grid,block,0,oct_str_0>>>
                    ( oct_obj._data_tex[ level-1 ],
                      oct_obj.getIntermediateData( ),
                      level );
#else // HORIZ_NO_SHARED_128x1
                filter_gauss_horiz_v11_128x1
                    <<<grid,block,0,oct_str_0>>>
                    ( oct_obj._data_tex[ level-1 ],
                      oct_obj.getIntermediateData( ),
                      level );
#endif // HORIZ_NO_SHARED_128x1
#endif
            }

            if( level == 0 ) {
                dim3 v_block( 64, 2 );
                dim3 v_grid;
                v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
                v_grid.y = (unsigned int)grid_divide( height, v_block.y );

                filter_gauss_vert_v11
                    <<<v_grid,v_block,0,oct_str_0>>>
                    ( oct_obj._interm_data_tex,
                      oct_obj.getData( level ),
                      level );
            } else {
                dim3 v_block( 64, 2 );
                dim3 v_grid;
                v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
                v_grid.y = (unsigned int)grid_divide( height, v_block.y );

                filter_gauss_vert_v11
                    <<<v_grid,v_block,0,oct_str_0>>>
                    ( oct_obj._interm_data_tex,
                      oct_obj.getData( level ),
                      level );

                dim3 e_block( 128, 2 );
                dim3 e_grid;
                e_grid.x = grid_divide( width,  e_block.x );
                e_grid.y = grid_divide( height, e_block.y );

                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );
            }

            hipEventRecord( oct_obj.getEventGaussDone( level ), oct_str_0 );
        }
    }
#endif
}

} // namespace popart

