#include "hip/hip_runtime.h"
#include "s_pyramid.h"

#include "write_plane_2d.h"
#include "gauss_filter.h"
#include "clamp.h"
#include "debug_macros.h"
#include "assist.h"
#include <hip/hip_runtime.h>

/*************************************************************
 * V11: device side
 *************************************************************/

#define V11_EDGE_LEN 32
#define V11_RANGE    4 // RANGES from 1 to 8 are possible
#define V11_LEVELS   _levels

#define SEPARATE_DOG_BUILDING

namespace popart {

__global__
void filter_gauss_horiz_v11_128x1( Plane2D_float src_data,
                                    Plane2D_float dst_data )
{
    __shared__ float loaddata[4 + 128 + 4];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int       idx    = threadIdx.x;
    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;
    int       read_x;
    int       read_y = clamp( blockIdx.y, src_h );
    if( idx < 4 ) {
        read_x = clamp( off_x - 4, src_w );
        loaddata[idx] = src_data.ptr(read_y)[read_x];
    } else if( idx >= 128-4 ) {
        read_x = clamp( off_x + 4, src_w );
        loaddata[idx+8] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();
    read_x = clamp( off_x, src_w );
    loaddata[idx+4] = src_data.ptr(read_y)[read_x];

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];
        val = loaddata[threadIdx.x+4-offset];
        out += ( val * g );
        val = loaddata[threadIdx.x+4+offset];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    val = loaddata[threadIdx.x+V11_RANGE];
    out += ( val * g );

    if( off_x >= src_w )      return;
    if( blockIdx.y >= src_h ) return;

    dst_data.ptr(blockIdx.y)[off_x] = out;
}
#if 0
__global__
void filter_gauss_horiz_v11( Plane2D_float src_data,
                             Plane2D_float dst_data )
{
    __shared__ float loaddata[V11_EDGE_LEN][V11_RANGE + V11_EDGE_LEN + V11_RANGE];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int idx     = threadIdx.x;
    int idy     = threadIdx.y;
    for( ; idx < V11_EDGE_LEN+2*V11_RANGE; idx += V11_EDGE_LEN) {
        int read_x = clamp( blockIdx.x * blockDim.x + idx - V11_RANGE, src_w );
        int read_y = clamp( blockIdx.y * blockDim.y + idy,             src_h );
        loaddata[idy][idx] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];
        val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE-offset];
        out += ( val * g );
        val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE+offset];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE];
    out += ( val * g );

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    idy = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= src_w ) return;
    if( idy >= src_h ) return;

    dst_data.ptr(idy)[idx] = out;
}
#endif

__global__
void filter_gauss_horiz_v11( hipTextureObject_t src_data,
                             Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void filter_gauss_horiz_v11_by_2( hipTextureObject_t src_data,
                                  Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}


__device__ inline
float filter_gauss_vert_v11_sub( hipTextureObject_t src_data,
                                 Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return 0;
    if( idy >= dst_h ) return 0;

    dst_data.ptr(idy)[idx] = out;

    return out;
}

__global__
void filter_gauss_vert_v11( hipTextureObject_t src_data,
                            Plane2D_float       dst_data )
{
    filter_gauss_vert_v11_sub( src_data, dst_data );
}

#ifdef SEPARATE_DOG_BUILDING
__global__
void make_dog( hipTextureObject_t this_data,
               hipTextureObject_t top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float b;
    b = tex2D<float>( this_data, idx, idy );
    float a;
    a = tex2D<float>( top_data, idx, idy );
    a = fabs( a - b );

    surf2DLayeredwrite( a, dog_data,
                        idx*4, idy, level,
                        hipBoundaryModeZero );
}
__global__
void make_dog( Plane2D_float       this_data,
               Plane2D_float       top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float b = this_data.ptr(idy)[idx];
    float a = top_data .ptr(idy)[idx];
    a = fabs( a - b );

    surf2DLayeredwrite( a, dog_data,
                        idx*4, idy, level,
                        hipBoundaryModeZero );
}
__global__
void make_dog4( Plane2D_float       this_data,
                Plane2D_float       top_data,
                hipSurfaceObject_t dog_data,
                int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y * 4;

    float4 b = *(float4*)&this_data.ptr(idy)[idx];
    float4 a = *(float4*)&top_data .ptr(idy)[idx];
    a.x = fabs( a.x - b.x );
    a.y = fabs( a.y - b.y );
    a.z = fabs( a.z - b.z );
    a.w = fabs( a.w - b.w );

    surf2DLayeredwrite( a, dog_data,
                        idx*16, idy, level,
                        hipBoundaryModeZero );
}
#else // not SEPARATE_DOG_BUILDING
__global__
void filter_gauss_vert_v11_dog( hipTextureObject_t src_data,
                                Plane2D_float       dst_data,
                                hipTextureObject_t top_data,
                                hipSurfaceObject_t dog_data,
                                int                 level )
{
    float b = filter_gauss_vert_v11_sub( src_data, dst_data );

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float a;
    a = tex2D<float>( top_data, idx, idy );
    a = fabs( a - b );

    surf2DLayeredwrite( a, dog_data,
                        idx*4, idy, level,
                        hipBoundaryModeZero );
}
#endif // not SEPARATE_DOG_BUILDING

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_v11( Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    for( int octave=0; octave<_num_octaves; octave++ ) {
        for( int level=0; level<V11_LEVELS; level++ ) {
            const int width  = _octaves[octave].getData(0).getWidth();
            const int height = _octaves[octave].getData(0).getHeight();

            dim3 h_block( 64, 2 );
            dim3 h_grid;
            h_grid.x = grid_divide( width,  h_block.x );
            h_grid.y = grid_divide( height, h_block.y );

            dim3 v_block( 64, 2 );
            dim3 v_grid;
            v_grid.x = grid_divide( width,  v_block.x );
            v_grid.y = grid_divide( height, v_block.y );

            dim3 d_block( 32, 1 );
            dim3 d_grid;
            d_grid.x = grid_divide( width,  d_block.x );
            d_grid.y = grid_divide( height, d_block.y );

            Octave&      oct_obj   = _octaves[octave];
            hipStream_t oct_str_0 = oct_obj.getStream(0);

            if( level == 0 ) {
                if( octave == 0 ) {
                    dim3 block;
                    dim3 grid;
                    const int width  = _octaves[octave].getData(0).getWidth();
                    const int height = _octaves[octave].getData(0).getHeight();

                    block.x = 128;
                    block.y = 1;
                    grid.x  = grid_divide( width,  128 );
                    grid.y  = height;
                    filter_gauss_horiz_v11_128x1
                        <<<grid,block,0,oct_str_0>>>
                        ( base->array,
                          oct_obj.getIntermediateData( ) );
                } else {
                    Octave& prev_oct_obj  = _octaves[octave-1];
                    hipStreamWaitEvent( oct_str_0, prev_oct_obj.getEventGaussDone( V11_LEVELS-3 ), 0 );

                    filter_gauss_horiz_v11_by_2
                        <<<h_grid,h_block,0,oct_str_0>>>
                        ( prev_oct_obj._data_tex[ V11_LEVELS-3 ],
                          // _octaves[octave-1]._data_tex[ 0 ],
                          oct_obj.getIntermediateData( ) );
                }
            } else {
                filter_gauss_horiz_v11
                    <<<h_grid,h_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[ level-1 ],
                      oct_obj.getIntermediateData( ) );
            }

            if( level == 0 ) {
                filter_gauss_vert_v11
                    <<<v_grid,v_block,0,oct_str_0>>>
                    ( oct_obj._interm_data_tex,
                      oct_obj.getData( level ) );
            } else {
#ifdef SEPARATE_DOG_BUILDING
                filter_gauss_vert_v11
                    <<<v_grid,v_block,0,oct_str_0>>>
                    ( oct_obj._interm_data_tex,
                      oct_obj.getData( level ) );

                dim3 e_block;
                dim3 e_grid;

                e_block.x = 16;
                e_block.y = 1;
                e_grid.x = grid_divide( width,  d_block.x );
                e_grid.y = grid_divide( height, d_block.y );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj.getData( level ),
                      oct_obj.getData( level-1 ),
                      oct_obj.getDogSurface( ),
                      level-1 );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );

                e_block.x = 32;
                e_block.y = 1;
                e_grid.x = grid_divide( width,  d_block.x );
                e_grid.y = grid_divide( height, d_block.y );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj.getData( level ),
                      oct_obj.getData( level-1 ),
                      oct_obj.getDogSurface( ),
                      level-1 );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );

                e_block.x = 64;
                e_block.y = 1;
                e_grid.x = grid_divide( width,  d_block.x );
                e_grid.y = grid_divide( height, d_block.y );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj.getData( level ),
                      oct_obj.getData( level-1 ),
                      oct_obj.getDogSurface( ),
                      level-1 );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );

                e_block.x = 16;
                e_block.y = 2;
                e_grid.x = grid_divide( width,  d_block.x );
                e_grid.y = grid_divide( height, d_block.y );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj.getData( level ),
                      oct_obj.getData( level-1 ),
                      oct_obj.getDogSurface( ),
                      level-1 );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );

                e_block.x = 32;
                e_block.y = 2;
                e_grid.x = grid_divide( width,  d_block.x );
                e_grid.y = grid_divide( height, d_block.y );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj.getData( level ),
                      oct_obj.getData( level-1 ),
                      oct_obj.getDogSurface( ),
                      level-1 );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );

                e_block.x = 64;
                e_block.y = 2;
                e_grid.x = grid_divide( width,  d_block.x );
                e_grid.y = grid_divide( height, d_block.y );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj.getData( level ),
                      oct_obj.getData( level-1 ),
                      oct_obj.getDogSurface( ),
                      level-1 );
                make_dog
                    <<<e_grid,e_block,0,oct_str_0>>>
                    ( oct_obj._data_tex[level],
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );

#else // not SEPARATE_DOG_BUILDING
                filter_gauss_vert_v11_dog
                    <<<d_grid,d_block,0,oct_str_0>>>
                    ( oct_obj._interm_data_tex,
                      oct_obj.getData( level ),
                      oct_obj._data_tex[level-1],
                      oct_obj.getDogSurface( ),
                      level-1 );
#endif // not SEPARATE_DOG_BUILDING
            }

            hipEventRecord( oct_obj.getEventGaussDone( level ), oct_str_0 );
        }
    }
}

} // namespace popart

