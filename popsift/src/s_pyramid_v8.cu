#include "hip/hip_runtime.h"
#include "s_pyramid.h"

#include "gauss_filter.h"
#include "clamp.h"
#include "debug_macros.h"
#include "assist.h"

/*************************************************************
 * V8: device side
 *************************************************************/

#define V8_EDGE_LEN    32
#define V8_RANGE    4 // RANGES from 1 to 12 are possible
#define V8_GAUSS_BASE   ( GAUSS_ONE_SIDE_RANGE - V8_RANGE )
#define V8_FILTERSIZE   ( V8_RANGE + 1        + V8_RANGE )
#define V8_LEVELS       _levels

namespace popart {

__global__
void filter_gauss_horiz_v8( Plane2D_float src_data,
                            Plane2D_float dst_data )
{
    __shared__ float loaddata[V8_EDGE_LEN][V8_RANGE + V8_EDGE_LEN + V8_RANGE];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int block_x = blockIdx.x * V8_EDGE_LEN;
    int block_y = blockIdx.y * V8_EDGE_LEN;
    int idx     = threadIdx.x;
    int idy     = threadIdx.y;
    for( ; idx < V8_EDGE_LEN+2*V8_RANGE; idx += V8_EDGE_LEN) {
        int read_x = clamp( block_x + idx - V8_RANGE, src_w );
        int read_y = clamp( block_y + idy,            src_h );
        loaddata[idy][idx] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = V8_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = loaddata[threadIdx.y][idx+V8_RANGE];
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = loaddata[threadIdx.y][idx+V8_RANGE];
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = loaddata[threadIdx.y][idx+V8_RANGE];
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    if( idx >= src_w ) return;
    if( idy >= src_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__device__
void filter_gauss_vert_v8_sub( Plane2D_float&  src_data,
                               Plane2D_float&  dst_data )
{
    // does not work on Mac !
    // assert( blockDim.x == blockDim.y );
    const int src_w   = src_data.getWidth();
    const int src_h   = src_data.getHeight();

    /* loaddata is transposed with respect to the src plane */
    __shared__ float loaddata[V8_EDGE_LEN][V8_RANGE + V8_EDGE_LEN + V8_RANGE];

    int block_x = blockIdx.x * V8_EDGE_LEN;
    int block_y = blockIdx.y * V8_EDGE_LEN;
    int idx     = threadIdx.x;
    int idy     = threadIdx.y;
    for( ; idy < V8_EDGE_LEN+2*V8_RANGE; idy += V8_EDGE_LEN) {
        int read_x = clamp( block_x + idx,            src_w );
        int read_y = clamp( block_y + idy - V8_RANGE, src_h );
        loaddata[idx][idy] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = V8_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = loaddata[threadIdx.y][idx+V8_RANGE];
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = loaddata[threadIdx.y][idx+V8_RANGE];
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = loaddata[threadIdx.y][idx+V8_RANGE];
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    if( idx >= src_w ) return;
    if( idy >= src_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void filter_gauss_vert_v8( Plane2D_float   src_data,
                           Plane2D_float   dst_data )
{
    filter_gauss_vert_v8_sub( src_data, dst_data );
}

__global__
void filter_gauss_vert_v8_and_dog( Plane2D_float   src_data,
                                   Plane2D_float   dst_data,
                                   Plane2D_float   higher_level_data,
                                   Plane2D_float   dog_data )
{
    filter_gauss_vert_v8_sub( src_data, dst_data );

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y;

    const int width  = src_data.getWidth();
    const int height = src_data.getHeight();

    if( idx >= width ) return;
    if( idy >= height ) return;

    float a, b;
    a = dst_data.ptr(idy)[idx];
    b = higher_level_data.ptr(idy)[idx];
    a = fabs( a - b );
    dog_data.ptr(idy)[idx] = a;
}

__global__
void filter_gauss_horiz_v8_by_2( Plane2D_float   src_data,
                                 Plane2D_float   dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y;

    const int src_w   = src_data.getWidth();
    const int src_h   = src_data.getHeight();
    int       src_idx;
    const int src_idy = clamp( 2 * block_y, src_h );
    const int dst_w   = dst_data.getWidth();
    const int dst_h   = dst_data.getHeight();
    const int dst_idx = block_x + threadIdx.x;
    const int dst_idy = block_y;

    if( dst_idx >= dst_w ) return;
    if( dst_idy >= dst_h ) return;

    float g;
    float val;
    float out = 0;

    for( int offset = V8_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        src_idx = clamp( 2 * ( dst_idx - offset ), src_w );
        val = src_data.ptr(src_idy)[src_idx];
        out += ( val * g );

        src_idx = clamp( 2 * ( dst_idx + offset ), src_w );
        val = src_data.ptr(src_idy)[src_idx];
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    src_idx = clamp( 2 * dst_idx, src_w );
    val = src_data.ptr(src_idy)[src_idx];
    out += ( val * g );

    dst_data.ptr(dst_idy)[dst_idx] = out;
}

/*************************************************************
 * V8: host side
 *************************************************************/
__host__
void Pyramid::build_v8( Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#else
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    for( int octave=0; octave<_num_octaves; octave++ ) {
        dim3 block;
        block.x = V8_EDGE_LEN;
        block.y = V8_EDGE_LEN;

        dim3 grid;
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        grid.x = grid_divide( width,  V8_EDGE_LEN );
        grid.y = grid_divide( height, V8_EDGE_LEN );

        for( int level=0; level<V8_LEVELS; level++ ) {
#if 0
        cerr << "Configuration for octave " << octave << endl
             << "  Horiz: layer size: "
             << _octaves[octave].getData(level).getWidth() << "x" << _octaves[octave].getData(level).getHeight() << endl
             << "  Vert: layer size: "
             << _octaves[octave].getIntermediateData().getWidth() << "x" << _octaves[octave].getIntermediateData().getHeight() << endl
             << "  grid: "
             << "(" << grid.x << "," << grid.y << "," << grid.z << ")"
             << " block: "
             << "(" << block.x << "," << block.y << "," << block.z << ")" << endl;
#endif


            if( level == 0 ) {
                if( octave == 0 ) {
                    filter_gauss_horiz_v8
                        <<<grid,block,0,_stream>>>
                        ( base->array,
                          _octaves[octave].getIntermediateData( ) );
                } else {
                    filter_gauss_horiz_v8_by_2
                        <<<grid,block,0,_stream>>>
                        ( _octaves[octave-1].getData( V8_LEVELS-3 ),
                          _octaves[octave].getIntermediateData( ) );
                }
            } else {
                filter_gauss_horiz_v8
                    <<<grid,block,0,_stream>>>
                    ( _octaves[octave].getData( level-1 ),
                      _octaves[octave].getIntermediateData( ) );
            }
            // hipStreamSynchronize( _stream );
            // hipError_t err = hipGetLastError();
            // POP_CUDA_FATAL_TEST( err, "filter_gauss_horiz_v8 failed: " );

            if( level == 0 ) {
                filter_gauss_vert_v8
                    <<<grid,block,0,_stream>>>
                    ( _octaves[octave].getIntermediateData( ),
                      _octaves[octave].getData( level ) );
            } else {
                filter_gauss_vert_v8_and_dog
                    <<<grid,block,0,_stream>>>
                    ( _octaves[octave].getIntermediateData( ),
                      _octaves[octave].getData( level ),
                      _octaves[octave].getData( level-1 ),
                      _octaves[octave].getDogData( level-1 ) );
            }
            // hipStreamSynchronize( _stream );
            // err = hipGetLastError();
            // POP_CUDA_FATAL_TEST( err, "filter_gauss_horiz_v8 failed: " );
        }
    }
    hipStreamSynchronize( _stream );
    hipError_t err = hipGetLastError();
    POP_CUDA_FATAL_TEST( err, "filter_gauss_horiz_v8 failed: " );
}

} // namespace popart

