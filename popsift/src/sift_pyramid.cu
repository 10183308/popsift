#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <sys/stat.h>

#include "sift_pyramid.h"
#include "debug_macros.h"

#define PYRAMID_PRINT_DEBUG 0

using namespace std;

namespace popart {

/*************************************************************
 * CUDA device functions for printing debug information
 *************************************************************/

__global__
void py_print_corner_float( float* img, uint32_t pitch, uint32_t height, uint32_t level )
{
    const int xbase = 0;
    const int ybase = level * height + 0;
    for( int i=0; i<10; i++ ) {
        for( int j=0; j<10; j++ ) {
            printf("%3.3f ", img[(ybase+i)*pitch+xbase+j] );
        }
        printf("\n");
    }
    printf("\n");
}

__global__
void py_print_corner_float_transposed( float* img, uint32_t pitch, uint32_t height, uint32_t level )
{
    const int xbase = 0;
    const int ybase = level * height + 0;
    for( int i=0; i<10; i++ ) {
        for( int j=0; j<10; j++ ) {
            printf("%3.3f ", img[(ybase+j)*pitch+xbase+i] );
        }
        printf("\n");
    }
    printf("\n");
}

/*************************************************************
 * Callers for CUDA device functions that print debug information
 *************************************************************/

void Pyramid::debug_out_floats( float* data, uint32_t pitch, uint32_t height )
{
    py_print_corner_float
        <<<1,1>>>
        ( data,
          pitch,
          height,
          0 );

    test_last_error( __LINE__ );
}

void Pyramid::debug_out_floats_t( float* data, uint32_t pitch, uint32_t height )
{
    py_print_corner_float_transposed
        <<<1,1>>>
        ( data,
          pitch,
          height,
          0 );

    test_last_error( __LINE__ );
}

/*************************************************************
 * Host-sided debug function
 *************************************************************/

void Pyramid::test_last_error( int line )
{
    hipError_t err;
    hipDeviceSynchronize( );
    err = hipGetLastError();
    if( err != hipSuccess ) {
        printf("A problem in line %d, %s\n", line, hipGetErrorString(err) );
        exit( -__LINE__ );
    }
}

/*************************************************************
 * Debug output: write an octave/level to disk as PGM
 *************************************************************/

void Pyramid::download_and_save_array( const char* basename, uint32_t octave, uint32_t level )
{
    if( octave < _num_octaves ) {
        _octaves[octave].download_and_save_array( basename, octave, level );
    } else {
        cerr << "Octave " << octave << " does not exist" << endl;
        return;
    }
}

void Pyramid::download_descriptors( uint32_t octave )
{
    _octaves[octave].downloadDescriptor( );
}

void Pyramid::save_descriptors( const char* basename, uint32_t octave, int downscale_factor )
{
    struct stat st = {0};
    if (stat("dir-desc", &st) == -1) {
        mkdir("dir-desc", 0700);
    }
    ostringstream ostr;
    ostr << "dir-desc/desc-" << basename << "-o-" << octave << ".txt";
    ofstream of( ostr.str().c_str() );
    _octaves[octave].writeDescriptor( of, downscale_factor );
}

/*************************************************************
 * Pyramid constructor
 *************************************************************/

Pyramid::Pyramid( Config& config,
                  Image* base,
                  int width,
                  int height )
    : _num_octaves( config.octaves )
    , _levels( config.levels + 3 )
    , _scaling_mode( config.scaling_mode )
    , _gauss_group( config.gauss_group_size )
{
    // cerr << "Entering " << __FUNCTION__ << endl;

    _octaves = new Octave[_num_octaves];

    int w = width;
    int h = height;

    cout << "Size of the first octave's images: " << w << "X" << h << endl;

    for( int o=0; o<_num_octaves; o++ ) {
#if (PYRAMID_PRINT_DEBUG==1)
        printf("Allocating octave %u with width %u and height %u (%u levels)\n", o, w, h, _levels );
#endif // (PYRAMID_PRINT_DEBUG==1)
        _octaves[o].debugSetOctave( o );
        _octaves[o].alloc( w, h, _levels, 10000, _gauss_group );
        w = ceilf( w / 2.0f );
        h = ceilf( h / 2.0f );
    }
}

/*************************************************************
 * Pyramid destructor
 *************************************************************/

Pyramid::~Pyramid( )
{
    delete [] _octaves;
}

/*************************************************************
 * Build the pyramid in all levels, one octave
 *************************************************************/

void Pyramid::build( Image* base )
{
    build_v11( base );
}

void Pyramid::find_extrema( float edgeLimit, float threshold )
{
    find_extrema_v6( edgeLimit, threshold );

    orientation_v1();

    descriptors_v1( );
}

} // namespace popart

