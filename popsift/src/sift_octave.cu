#include "hip/hip_runtime.h"
#include <sstream>
#include <sys/stat.h>
#include <new> // for placement new

#include "sift_pyramid.h"
#include "sift_constants.h"
#include "debug_macros.h"
#include "clamp.h"
#include "write_plane_2d.h"
#include "sift_octave.h"

// #define PYRAMID_PRINT_DEBUG 0

using namespace std;

namespace popart {

/*************************************************************
 * Octave
 *************************************************************/

Octave::Octave( )
    : _data(0)
    , _h_extrema_mgmt(0)
    , _d_extrema_mgmt(0)
    , _h_extrema(0)
    , _d_extrema(0)
    , _d_desc(0)
    , _h_desc(0)
{ }


void Octave::alloc( int width, int height, int levels, int gauss_group )
{
    _w           = width;
    _h           = height;
    _levels      = levels;
    _gauss_group = gauss_group;

#if (PYRAMID_PRINT_DEBUG==1)
    printf("    correcting to width %u, height %u\n", _width, _height );
#endif // (PYRAMID_PRINT_DEBUG==1)

    alloc_data_planes( );
    alloc_data_tex( );

    alloc_interm_plane( );
    alloc_interm_tex( );

    alloc_dog_array( );
    alloc_dog_tex( );

    alloc_extrema_mgmt( );
    alloc_extrema( );

    _streams = new hipStream_t[_levels];
    _gauss_done = new hipEvent_t[_levels];
    for( int i=0; i<_levels; i++ ) {
        _streams[i]    = popart::cuda::stream_create( __FILE__, __LINE__ );
        _gauss_done[i] = popart::cuda::event_create( __FILE__, __LINE__ );
    }

    _d_desc = new Descriptor*[_levels];
    _h_desc = new Descriptor*[_levels];

    for( int l=0; l<_levels; l++ ) {
        int sz = h_max_orientations;
        if( sz == 0 ) {
            _d_desc[l] = 0;
            _h_desc[l] = 0;
        } else {
            _d_desc[l] = popart::cuda::malloc_devT<Descriptor>( sz, __FILE__, __LINE__ );
            _h_desc[l] = popart::cuda::malloc_hstT<Descriptor>( sz, __FILE__, __LINE__ );
        }
    }
}

void Octave::free( )
{
    for( int i=0; i<_levels; i++ ) {
        if( _h_desc && _h_desc[i] ) hipHostFree( _h_desc[i] );
        if( _d_desc && _d_desc[i] ) hipFree(     _d_desc[i] );
    }
    delete [] _d_desc;
    delete [] _h_desc;

    for( int i=0; i<_levels; i++ ) {
        popart::cuda::stream_destroy( _streams[i], __FILE__, __LINE__ );
        popart::cuda::event_destroy( _gauss_done[i], __FILE__, __LINE__ );
    }
    delete [] _streams;
    delete [] _gauss_done;

    free_extrema( );
    free_extrema_mgmt( );

    free_dog_tex( );
    free_dog_array( );

    free_interm_tex( );
    free_interm_plane( );

    free_data_tex( );
    free_data_planes( );
}

#if 0
void Octave::resetExtremaCount( )
{
    for( uint32_t i=1; i<_levels-1; i++ ) {
        _h_extrema_mgmt[i].resetCounter();
    }
    POP_CUDA_MEMCPY_ASYNC( _d_extrema_mgmt,
                           _h_extrema_mgmt,
                           _levels * sizeof(ExtremaMgmt),
                           hipMemcpyHostToDevice,
                           0,
                           true );
}
#endif

void Octave::readExtremaCount( )
{
    assert( _h_extrema_mgmt );
    assert( _d_extrema_mgmt );
    popcuda_memcpy_async( _h_extrema_mgmt,
                          _d_extrema_mgmt,
                          _levels * sizeof(ExtremaMgmt),
                          hipMemcpyDeviceToHost,
                          _streams[0] );
}

int Octave::getExtremaCount( ) const
{
    int ct = 0;
    for( uint32_t i=1; i<_levels-1; i++ ) {
        ct += _h_extrema_mgmt[i].getCounter();
    }
    return ct;
}

int Octave::getExtremaCount( uint32_t level ) const
{
    if( level < 1 )         return 0;
    if( level > _levels-2 ) return 0;
    return _h_extrema_mgmt[level].getCounter();
}

void Octave::downloadDescriptor( )
{
    for( uint32_t l=0; l<_levels; l++ ) {
        int sz = _h_extrema_mgmt[l].getCounter();
        if( sz != 0 ) {
            if( _h_extrema[l] == 0 ) continue;

            popcuda_memcpy_async( _h_desc[l],
                                  _d_desc[l],
                                  sz * sizeof(Descriptor),
                                  hipMemcpyDeviceToHost,
                                  0 );
            popcuda_memcpy_async( _h_extrema[l],
                                  _d_extrema[l],
                                  sz * sizeof(Extremum),
                                  hipMemcpyDeviceToHost,
                                  0 );
        }
    }

    hipDeviceSynchronize( );
}

void Octave::writeDescriptor( ostream& ostr, float downsampling_factor )
{
    for( uint32_t l=0; l<_levels; l++ ) {
        if( _h_extrema[l] == 0 ) continue;

        Extremum* cand = _h_extrema[l];

        Descriptor* desc = _h_desc[l];
        int sz = _h_extrema_mgmt[l].getCounter();
        for( int s=0; s<sz; s++ ) {
            const float reduce = downsampling_factor;

            ostr << setprecision(5)
                 << ( cand[s].xpos - 0.0f ) * pow( 2.0, _debug_octave_id + reduce ) << " "
                 << ( cand[s].ypos - 0.0f ) * pow( 2.0, _debug_octave_id + reduce ) << " "
                 << cand[s].sigma * pow( 2.0, _debug_octave_id + reduce ) << " "
                 << cand[s].orientation << " ";
            for( int i=0; i<128; i++ ) {
                ostr << setprecision(3) << desc[s].features[i] << " ";
            }
            ostr << endl;
        }
    }
}

Descriptor* Octave::getDescriptors( uint32_t level )
{
    return _d_desc[level];
}

/*************************************************************
 * Debug output: write an octave/level to disk as PGM
 *************************************************************/

void Octave::download_and_save_array( const char* basename, uint32_t octave, uint32_t level )
{
    // cerr << "Calling " << __FUNCTION__ << " for octave " << octave << endl;

    if( level >= _levels ) {
        // cerr << "Level " << level << " does not exist in Octave " << octave << endl;
        return;
    }

    struct stat st = {0};

#if 1
    {
        if (stat("dir-octave", &st) == -1) {
            mkdir("dir-octave", 0700);
        }

        ostringstream ostr;
        ostr << "dir-octave/" << basename << "-o-" << octave << "-l-" << level << ".pgm";
        // cerr << "Writing " << ostr.str() << endl;
        popart::write_plane2D( ostr.str().c_str(), true, getData(level) );

        if( level == 0 ) {
            int width  = getData(level).getWidth();
            int height = getData(level).getHeight();

            Plane2D_float hostPlane_f;
            hostPlane_f.allocHost( width, height, CudaAllocated );
            hostPlane_f.memcpyFromDevice( getData(level) );

            uint32_t total_ct = 0;

            readExtremaCount( );
            hipDeviceSynchronize( );
            for( uint32_t l=0; l<_levels; l++ ) {
                uint32_t ct = getExtremaCount( l );
                if( ct > 0 ) {
                    total_ct += ct;

                    Extremum* cand = new Extremum[ct];

                    popcuda_memcpy_sync( cand,
                                         _d_extrema[l],
                                         ct * sizeof(Extremum),
                                         hipMemcpyDeviceToHost );
                    for( uint32_t i=0; i<ct; i++ ) {
                        int32_t x = roundf( cand[i].xpos );
                        int32_t y = roundf( cand[i].ypos );
                        // cerr << "(" << x << "," << y << ") scale " << cand[i].sigma << " orient " << cand[i].orientation << endl;
                        for( int32_t j=-4; j<=4; j++ ) {
                            hostPlane_f.ptr( clamp(y+j,height) )[ clamp(x,  width) ] = 255;
                            hostPlane_f.ptr( clamp(y,  height) )[ clamp(x+j,width) ] = 255;
                        }
                    }

                    delete [] cand;
                }
            }

            if( total_ct > 0 ) {
                if (stat("dir-feat", &st) == -1) {
                    mkdir("dir-feat", 0700);
                }

                if (stat("dir-feat-txt", &st) == -1) {
                    mkdir("dir-feat-txt", 0700);
                }


                ostringstream ostr;
                ostr << "dir-feat/" << basename << "-o-" << octave << "-l-" << level << ".pgm";
                ostringstream ostr2;
                ostr2 << "dir-feat-txt/" << basename << "-o-" << octave << "-l-" << level << ".txt";
        #if 0
                ofstream of( ostr.str().c_str() );
                // cerr << "Writing " << ostr.str() << endl;
                of << "P5" << endl
                   << width << " " << height << endl
                   << "255" << endl;
                of.write( (char*)hostPlane_c.data, hostPlane_c.getByteSize() );
                of.close();
        #endif

                popart::write_plane2D( ostr.str().c_str(), false, hostPlane_f );
                popart::write_plane2Dunscaled( ostr2.str().c_str(), false, hostPlane_f );
            }

            hostPlane_f.freeHost( CudaAllocated );
        }
    }
#endif
#if 1
    if( level == _levels-1 ) {
        hipError_t err;
        int width  = getData(0).getWidth();
        int height = getData(0).getHeight();

        if (stat("dir-dog", &st) == -1) {
            mkdir("dir-dog", 0700);
        }

        if (stat("dir-dog-txt", &st) == -1) {
            mkdir("dir-dog-txt", 0700);
        }

        float* array;
        POP_CUDA_MALLOC_HOST( &array, width * height * (_levels-1) * sizeof(float) );

        hipMemcpy3DParms s = { 0 };
        s.srcArray = _dog_3d;
        s.dstPtr = make_hipPitchedPtr( array, width*sizeof(float), width, height );
        s.extent = make_hipExtent( width, height, _levels-1 );
        s.kind = hipMemcpyDeviceToHost;
        err = hipMemcpy3D( &s );
        POP_CUDA_FATAL_TEST( err, "hipMemcpy3D failed: " ); \

        for( int l=0; l<_levels-1; l++ ) {
            Plane2D_float p( width, height, &array[l*width*height], width*sizeof(float) );

            ostringstream ostr;
            ostr << "dir-dog/d-" << basename << "-o-" << octave << "-l-" << l << ".pgm";
            // cerr << "Writing " << ostr.str() << endl;
            popart::write_plane2D( ostr.str().c_str(), true, p );

            ostringstream ostr2;
            ostr2 << "dir-dog-txt/d-" << basename << "-o-" << octave << "-l-" << l << ".txt";
            popart::write_plane2Dunscaled( ostr2.str().c_str(), true, p );
        }

        POP_CUDA_FREE_HOST( array );
    }
#endif
}

void Octave::alloc_data_planes( )
{
    hipError_t err;
    void*       ptr;
    size_t      pitch;

    _data = new Plane2D_float[_levels];

    err = hipMallocPitch( &ptr, &pitch, _w * sizeof(float), _h * _levels );
    POP_CUDA_FATAL_TEST( err, "Cannot allocate data CUDA memory: " );
    for( int i=0; i<_levels; i++ ) {
        _data[i] = Plane2D_float( _w,
                                  _h,
                                  (float*)( (intptr_t)ptr + i*(pitch*_h) ),
                                  pitch );
    }
}

void Octave::free_data_planes( )
{
    POP_CUDA_FREE( _data[0].data );
    delete [] _data;
}

void Octave::alloc_data_tex( )
{
    hipError_t err;

    _data_tex = new hipTextureObject_t[_levels];

    hipTextureDesc      data_tex_desc;
    hipResourceDesc     data_res_desc;

    memset( &data_tex_desc, 0, sizeof(hipTextureDesc) );
    data_tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    data_tex_desc.addressMode[0]   = hipAddressModeClamp;
    data_tex_desc.addressMode[1]   = hipAddressModeClamp;
    data_tex_desc.addressMode[2]   = hipAddressModeClamp;
    data_tex_desc.readMode         = hipReadModeElementType; // read as float
    // data_tex_desc.filterMode       = hipFilterModePoint; // no interpolation
    data_tex_desc.filterMode       = hipFilterModeLinear; // bilinear interpolation

    memset( &data_res_desc, 0, sizeof(hipResourceDesc) );
    data_res_desc.resType                  = hipResourceTypePitch2D;
    data_res_desc.res.pitch2D.desc.f       = hipChannelFormatKindFloat;
    data_res_desc.res.pitch2D.desc.x       = 32;
    data_res_desc.res.pitch2D.desc.y       = 0;
    data_res_desc.res.pitch2D.desc.z       = 0;
    data_res_desc.res.pitch2D.desc.w       = 0;
    for( int i=0; i<_levels; i++ ) {
        data_res_desc.res.pitch2D.devPtr       = _data[i].data;
        data_res_desc.res.pitch2D.pitchInBytes = _data[i].step;
        data_res_desc.res.pitch2D.width        = _data[i].getCols();
        data_res_desc.res.pitch2D.height       = _data[i].getRows();

        err = hipCreateTextureObject( &_data_tex[i],
                                       &data_res_desc,
                                       &data_tex_desc, 0 );
        POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
    }
}

void Octave::free_data_tex( )
{
    hipError_t err;

    for( int i=0; i<_levels; i++ ) {
        err = hipDestroyTextureObject( _data_tex[i] );
        POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );
    }

    delete [] _data_tex;
}

void Octave::alloc_interm_plane( )
{
    /* Usually we alloc only one plane's worth of floats.
     * When we group gauss filters, we need #groupsize intermediate
     * planes. For efficiency, we use only a single allocation,
     * but if we use interpolation, we should better have a buffer
     * filled with zeros between the sections of the plane.
     * We give this buffer 4 rows.
     */
    _intermediate_data.allocDev( _w, _gauss_group * ( _h + 4 ) );
}

void Octave::free_interm_plane( )
{
    _intermediate_data.freeDev( );
}

void Octave::alloc_interm_tex( )
{
    hipError_t err;

    hipTextureDesc      data_tex_desc;
    hipResourceDesc     data_res_desc;

    memset( &data_tex_desc, 0, sizeof(hipTextureDesc) );
    data_tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    data_tex_desc.addressMode[0]   = hipAddressModeClamp;
    data_tex_desc.addressMode[1]   = hipAddressModeClamp;
    data_tex_desc.addressMode[2]   = hipAddressModeClamp;
    data_tex_desc.readMode         = hipReadModeElementType; // read as float
#ifdef GAUSS_INTERM_FILTER_MODE_POINT
    data_tex_desc.filterMode       = hipFilterModePoint; // no interpolation
#else // not GAUSS_INTERM_FILTER_MODE_POINT
    data_tex_desc.filterMode       = hipFilterModeLinear; // bilinear interpolation
#endif // not GAUSS_INTERM_FILTER_MODE_POINT

    memset( &data_res_desc, 0, sizeof(hipResourceDesc) );
    data_res_desc.resType                  = hipResourceTypePitch2D;
    data_res_desc.res.pitch2D.desc.f       = hipChannelFormatKindFloat;
    data_res_desc.res.pitch2D.desc.x       = 32;
    data_res_desc.res.pitch2D.desc.y       = 0;
    data_res_desc.res.pitch2D.desc.z       = 0;
    data_res_desc.res.pitch2D.desc.w       = 0;

    data_res_desc.res.pitch2D.devPtr       = _intermediate_data.data;
    data_res_desc.res.pitch2D.pitchInBytes = _intermediate_data.step;
    data_res_desc.res.pitch2D.width        = _intermediate_data.getCols();
    data_res_desc.res.pitch2D.height       = _intermediate_data.getRows();

    err = hipCreateTextureObject( &_interm_data_tex,
                                   &data_res_desc,
                                   &data_tex_desc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

void Octave::free_interm_tex( )
{
    hipError_t err;

    err = hipDestroyTextureObject( _interm_data_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );
}

void Octave::alloc_dog_array( )
{
    hipError_t err;

    _dog_3d_desc.f = hipChannelFormatKindFloat;
    _dog_3d_desc.x = 32;
    _dog_3d_desc.y = 0;
    _dog_3d_desc.z = 0;
    _dog_3d_desc.w = 0;

    _dog_3d_ext.width  = _w; // for hipMalloc3DArray, width in elements
    _dog_3d_ext.height = _h;
    _dog_3d_ext.depth  = _levels - 1;

    err = hipMalloc3DArray( &_dog_3d,
                             &_dog_3d_desc,
                             _dog_3d_ext,
                             hipArrayLayered | hipArraySurfaceLoadStore );
    POP_CUDA_FATAL_TEST( err, "Could not allocate 3D DoG array: " );
}

void Octave::free_dog_array( )
{
    hipError_t err;

    err = hipFreeArray( _dog_3d );
    POP_CUDA_FATAL_TEST( err, "Could not free 3D DoG array: " );
}

void Octave::alloc_dog_tex( )
{
    hipError_t err;

    hipResourceDesc dog_res_desc;
    dog_res_desc.resType         = hipResourceTypeArray;
    dog_res_desc.res.array.array = _dog_3d;

    err = hipCreateSurfaceObject( &_dog_3d_surf, &dog_res_desc );
    POP_CUDA_FATAL_TEST( err, "Could not create DoG surface: " );

    hipTextureDesc      dog_tex_desc;
    memset( &dog_tex_desc, 0, sizeof(hipTextureDesc) );
    dog_tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    dog_tex_desc.addressMode[0]   = hipAddressModeClamp;
    dog_tex_desc.addressMode[1]   = hipAddressModeClamp;
    dog_tex_desc.addressMode[2]   = hipAddressModeClamp;
    dog_tex_desc.readMode         = hipReadModeElementType; // read as float
    dog_tex_desc.filterMode       = hipFilterModePoint; // no interpolation

    // cudaResourceView dog_tex_view;
    // memset( &dog_tex_view, 0, sizeof(cudaResourceView) );
    // dog_tex_view.format     = hipResViewFormatFloat1;
    // dog_tex_view.width      = width;
    // dog_tex_view.height     = height;
    // dog_tex_view.depth      = 1;
    // dog_tex_view.firstLayer = 0;
    // dog_tex_view.lastLayer  = _levels - 1;

    err = hipCreateTextureObject( &_dog_3d_tex, &dog_res_desc, &dog_tex_desc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create DoG texture: " );
}

void Octave::free_dog_tex( )
{
    hipError_t err;

    err = hipDestroyTextureObject( _dog_3d_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy DoG texture: " );

    err = hipDestroySurfaceObject( _dog_3d_surf );
    POP_CUDA_FATAL_TEST( err, "Could not destroy DoG surface: " );
}

void Octave::alloc_extrema_mgmt( )
{
    _h_extrema_mgmt = popart::cuda::malloc_hstT<ExtremaMgmt>( _levels, __FILE__, __LINE__ );
    for( uint32_t i=0; i<_levels; i++ ) {
        _h_extrema_mgmt[i].reset();
    }

    _d_extrema_mgmt = popart::cuda::malloc_devT<ExtremaMgmt>( _levels, __FILE__, __LINE__ );
    popcuda_memcpy_sync( _d_extrema_mgmt,
                         _h_extrema_mgmt,
                         _levels * sizeof(ExtremaMgmt),
                         hipMemcpyHostToDevice );
}

void Octave::free_extrema_mgmt( )
{
    hipFree( _d_extrema_mgmt );
    hipHostFree( _h_extrema_mgmt );
}

void Octave::alloc_extrema( )
{
    _d_extrema = new Extremum*[ _levels ];
    _h_extrema = new Extremum*[ _levels ];

    _h_extrema[0] = 0;
    _h_extrema[_levels-1] = 0;
    _d_extrema[0] = 0;
    _d_extrema[_levels-1] = 0;

    int objects_per_level = h_max_orientations;
    int levels            = _levels - 2;

    Extremum* d = popart::cuda::malloc_devT<Extremum>( levels * objects_per_level, __FILE__, __LINE__ );
    Extremum* h = popart::cuda::malloc_hstT<Extremum>( levels * objects_per_level, __FILE__, __LINE__ );

    for( uint32_t i=1; i<_levels-1; i++ ) {
        const int offset = i-1;
        _d_extrema[i] = &d[offset*objects_per_level];
        _h_extrema[i] = &h[offset*objects_per_level];
    }
}

void Octave::free_extrema( )
{
    hipHostFree( _h_extrema[1] );
    hipFree(     _d_extrema[1] );
    delete [] _d_extrema;
    delete [] _h_extrema;
}

} // namespace popart

