#include "hip/hip_runtime.h"
#include "sift_extrema_mgmt.h"
#include "debug_macros.h"

namespace popart
{

int h_max_extrema      = 0;
int h_max_orientations = 0;
__device__ __constant__ int d_max_extrema;
__device__ __constant__ int d_max_orientations;

void ExtremaMgmt::init( int max_extrema )
{
    hipError_t err;

    h_max_extrema      = max_extrema;
    h_max_orientations = max_extrema + max_extrema/4;

    err = hipMemcpyToSymbol(HIP_SYMBOL( d_max_extrema), &h_max_extrema,
                              sizeof(int), 0,
                              hipMemcpyHostToDevice );
    POP_CUDA_FATAL_TEST( err, "Failed to upload h_max_extrema to device: " );

    err = hipMemcpyToSymbol(HIP_SYMBOL( d_max_orientations), &h_max_orientations,
                              sizeof(int), 0,
                              hipMemcpyHostToDevice );
    POP_CUDA_FATAL_TEST( err, "Failed to upload h_max_orientations to device: " );
}

void ExtremaMgmt::reset( )
{
    _counter = 0;
}

__device__
int ExtremaMgmt::atomicAddCounter( int ct )
{
    int idx = atomicAdd( &_counter, ct );
    return idx;
}


} // namespace iopart

