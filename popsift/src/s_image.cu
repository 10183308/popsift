#include "s_image.h"
#include <iostream>
#include <fstream>
#include "debug_macros.h"
#include "align_macro.h"
#include "assist.h"
#include <stdio.h>
#include <assert.h>

using namespace std;

namespace popart {


void Image::test_last_error( const char* file, int line )
{
    hipError_t err;
    err = hipGetLastError();
    if( err != hipSuccess ) {
        printf("Error in %s:%d\n     CUDA failed: %s\n", file, line, hipGetErrorString(err) );
        exit( -__LINE__ );
    }
}

#if 0
void Image::download_and_save_array( const char* filename )
{
    test_last_error( __FILE__, __LINE__ );

    cerr << "Downloading image from GPU to CPU and writing to file " << filename << endl;

    Plane2D_float f;
    f.allocHost( this->_upscaled_image_d.getCols(), this->_upscaled_image_d.getRows(), PageAligned );

    f.memcpyFromDevice( _upscaled_image_d );

    unsigned char* c = new unsigned char[ f.getCols() * f.getRows() ];
    for( int y=0; y<f.getRows(); y++ ) {
        for( int x=0; x<f.getCols(); x++ ) {
            c[y*f.getCols()+x] = (unsigned char)(f.ptr(y)[x]);
        }
    }
    ofstream of( filename );
    of << "P5" << endl
       << f.getCols() << " " << f.getRows() << endl
       << "255" << endl;
    of.write( (char*)c, f.getCols() * f.getRows() );
    delete [] c;

    f.freeHost( PageAligned );
}
#endif

Image::Image( size_t w, size_t h )
    : _w(w), _h(h)
{
    hipError_t err;

    _input_image_h.allocHost( w, h, popart::CudaAllocated );

    _input_image_d.allocDev( w, h );

    _upscaled_image_d.allocDev( 2 * w, 2 * h );

    cout << "Upscaled size of the input image: " << 2*w << "X" << 2*h << endl;

    /* initializing texture for upscaling
     */
    memset( &_input_image_texDesc, 0, sizeof(hipTextureDesc) );
    _input_image_texDesc.normalizedCoords = 1; // address 0..1 instead of 0..width/height
    _input_image_texDesc.addressMode[0]   = hipAddressModeClamp;
    _input_image_texDesc.addressMode[1]   = hipAddressModeClamp;
    _input_image_texDesc.addressMode[2]   = hipAddressModeClamp;
    _input_image_texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _input_image_texDesc.filterMode       = hipFilterModeLinear; // bilinear interpolation

    memset( &_input_image_resDesc, 0, sizeof(hipResourceDesc) );
    _input_image_resDesc.resType                  = hipResourceTypePitch2D;
    _input_image_resDesc.res.pitch2D.devPtr       = _input_image_d.data;
    _input_image_resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _input_image_resDesc.res.pitch2D.desc.x       = 8; // sizeof(uint8_t)*8
    _input_image_resDesc.res.pitch2D.desc.y       = 0;
    _input_image_resDesc.res.pitch2D.desc.z       = 0;
    _input_image_resDesc.res.pitch2D.desc.w       = 0;
    assert( _input_image_d.elemSize() == 1 );
    _input_image_resDesc.res.pitch2D.pitchInBytes = _input_image_d.step;
    _input_image_resDesc.res.pitch2D.width        = _input_image_d.getCols();
    _input_image_resDesc.res.pitch2D.height       = _input_image_d.getRows();

    err = hipCreateTextureObject( &_input_image_tex, &_input_image_resDesc, &_input_image_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );

    /* initializing texture for access by octaves
     */
    memset( &_upscaled_image_texDesc, 0, sizeof(hipTextureDesc) );
    _upscaled_image_texDesc.normalizedCoords = 1; // address 0..1 instead of 0..width/height
    _upscaled_image_texDesc.addressMode[0]   = hipAddressModeClamp;
    _upscaled_image_texDesc.addressMode[1]   = hipAddressModeClamp;
    _upscaled_image_texDesc.addressMode[2]   = hipAddressModeClamp;
    _upscaled_image_texDesc.readMode         = hipReadModeElementType; // no conversion, this is float
    _upscaled_image_texDesc.filterMode       = hipFilterModeLinear; // bilinear interpolation

    memset( &_upscaled_image_resDesc, 0, sizeof(hipResourceDesc) );
    _upscaled_image_resDesc.resType                  = hipResourceTypePitch2D;
    _upscaled_image_resDesc.res.pitch2D.devPtr       = _upscaled_image_d.data;
    _upscaled_image_resDesc.res.pitch2D.desc.f       = hipChannelFormatKindFloat; // float
    _upscaled_image_resDesc.res.pitch2D.desc.x       = 32; // sizeof(float)*8
    _upscaled_image_resDesc.res.pitch2D.desc.y       = 0;
    _upscaled_image_resDesc.res.pitch2D.desc.z       = 0;
    _upscaled_image_resDesc.res.pitch2D.desc.w       = 0;
    assert( _upscaled_image_d.elemSize() == 4 );
    _upscaled_image_resDesc.res.pitch2D.pitchInBytes = _upscaled_image_d.step;
    _upscaled_image_resDesc.res.pitch2D.width        = _upscaled_image_d.getCols();
    _upscaled_image_resDesc.res.pitch2D.height       = _upscaled_image_d.getRows();

    err = hipCreateTextureObject( &_upscaled_image_tex, &_upscaled_image_resDesc, &_upscaled_image_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

Image::~Image( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _input_image_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );

    err = hipDestroyTextureObject( _upscaled_image_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );

    _upscaled_image_d.freeDev( );
    _input_image_d   .freeDev( );
    _input_image_h   .freeHost( popart::CudaAllocated );
}

void Image::load( imgStream inp )
{
    memcpy( _input_image_h.data, inp.data_r, _w*_h );
    _input_image_h.memcpyToDevice( _input_image_d );
    upscale_v5( _input_image_tex );
}

} // namespace popart

