#include "s_image.h"
#include <iostream>
#include <fstream>
#include "debug_macros.h"
#include "align_macro.h"
#include "assist.h"
#include <stdio.h>
#include <assert.h>

using namespace std;

namespace popart {

void Image::test_last_error( const char* file, int line )
{
    hipError_t err;
    err = hipGetLastError();
    if( err != hipSuccess ) {
        printf("Error in %s:%d\n     CUDA failed: %s\n", file, line, hipGetErrorString(err) );
        exit( -__LINE__ );
    }
}

Image::Image( size_t w, size_t h )
    : _w(w), _h(h)
{
    hipError_t err;

    _input_image_h.allocHost( w, h, popart::CudaAllocated );

    _input_image_d.allocDev( w, h );

    _upscaled_image_d.allocDev( 2 * w, 2 * h );

    cout << "Upscaled size of the input image: " << 2*w << "X" << 2*h << endl;

    /* initializing texture for upscaling
     */
    memset( &_input_image_texDesc, 0, sizeof(hipTextureDesc) );
    _input_image_texDesc.normalizedCoords = 1; // address 0..1 instead of 0..width/height
    _input_image_texDesc.addressMode[0]   = hipAddressModeClamp;
    _input_image_texDesc.addressMode[1]   = hipAddressModeClamp;
    _input_image_texDesc.addressMode[2]   = hipAddressModeClamp;
    _input_image_texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _input_image_texDesc.filterMode       = hipFilterModeLinear; // bilinear interpolation

    memset( &_input_image_resDesc, 0, sizeof(hipResourceDesc) );
    _input_image_resDesc.resType                  = hipResourceTypePitch2D;
    _input_image_resDesc.res.pitch2D.devPtr       = _input_image_d.data;
    _input_image_resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _input_image_resDesc.res.pitch2D.desc.x       = 8; // sizeof(uint8_t)*8
    _input_image_resDesc.res.pitch2D.desc.y       = 0;
    _input_image_resDesc.res.pitch2D.desc.z       = 0;
    _input_image_resDesc.res.pitch2D.desc.w       = 0;
    assert( _input_image_d.elemSize() == 1 );
    _input_image_resDesc.res.pitch2D.pitchInBytes = _input_image_d.step;
    _input_image_resDesc.res.pitch2D.width        = _input_image_d.getCols();
    _input_image_resDesc.res.pitch2D.height       = _input_image_d.getRows();

    err = hipCreateTextureObject( &_input_image_tex, &_input_image_resDesc, &_input_image_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );

    /* initializing texture for access by octaves
     */
    memset( &_upscaled_image_texDesc, 0, sizeof(hipTextureDesc) );
    _upscaled_image_texDesc.normalizedCoords = 1; // address 0..1 instead of 0..width/height
    _upscaled_image_texDesc.addressMode[0]   = hipAddressModeClamp;
    _upscaled_image_texDesc.addressMode[1]   = hipAddressModeClamp;
    _upscaled_image_texDesc.addressMode[2]   = hipAddressModeClamp;
    _upscaled_image_texDesc.readMode         = hipReadModeElementType; // no conversion, this is float
    _upscaled_image_texDesc.filterMode       = hipFilterModeLinear; // bilinear interpolation

    memset( &_upscaled_image_resDesc, 0, sizeof(hipResourceDesc) );
    _upscaled_image_resDesc.resType                  = hipResourceTypePitch2D;
    _upscaled_image_resDesc.res.pitch2D.devPtr       = _upscaled_image_d.data;
    _upscaled_image_resDesc.res.pitch2D.desc.f       = hipChannelFormatKindFloat; // float
    _upscaled_image_resDesc.res.pitch2D.desc.x       = 32; // sizeof(float)*8
    _upscaled_image_resDesc.res.pitch2D.desc.y       = 0;
    _upscaled_image_resDesc.res.pitch2D.desc.z       = 0;
    _upscaled_image_resDesc.res.pitch2D.desc.w       = 0;
    assert( _upscaled_image_d.elemSize() == 4 );
    _upscaled_image_resDesc.res.pitch2D.pitchInBytes = _upscaled_image_d.step;
    _upscaled_image_resDesc.res.pitch2D.width        = _upscaled_image_d.getCols();
    _upscaled_image_resDesc.res.pitch2D.height       = _upscaled_image_d.getRows();

    err = hipCreateTextureObject( &_upscaled_image_tex, &_upscaled_image_resDesc, &_upscaled_image_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

Image::~Image( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _input_image_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );

    err = hipDestroyTextureObject( _upscaled_image_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );

    _upscaled_image_d.freeDev( );
    _input_image_d   .freeDev( );
    _input_image_h   .freeHost( popart::CudaAllocated );
}

void Image::load( imgStream inp )
{
    memcpy( _input_image_h.data, inp.data_r, _w*_h );
    _input_image_h.memcpyToDevice( _input_image_d );
    upscale_v5( _input_image_tex );
}

} // namespace popart

