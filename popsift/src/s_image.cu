#include "s_image.h"
#include <iostream>
#include <fstream>
#include "debug_macros.h"
#include "align_macro.h"
#include "assist.h"
#include <stdio.h>
#include <assert.h>

using namespace std;

namespace popart {

__host__
void Image::upscale( Plane2D_uint8 & src, hipTextureObject_t & tex, size_t scalefactor )
{
    if( scalefactor != 2 ) {
        cerr << "Scale factor is " << scalefactor << endl;
        cerr << "Currently only 2 is supported" << endl;
        exit( -__LINE__ );
    }

    if( false ) upscale_v1( src );
    if( false ) upscale_v2( src );
    if( false ) upscale_v3( src );
    if( false ) upscale_v4( src );
    if( true  ) upscale_v5( tex );
}

void Image::test_last_error( const char* file, int line )
{
    hipError_t err;
    err = hipGetLastError();
    if( err != hipSuccess ) {
        printf("Error in %s:%d\n     CUDA failed: %s\n", file, line, hipGetErrorString(err) );
        exit( -__LINE__ );
    }
}

void Image::download_and_save_array( const char* filename )
{
    test_last_error( __FILE__, __LINE__ );

    cerr << "Downloading image from GPU to CPU and writing to file " << filename << endl;

    Plane2D_float f;
    f.allocHost( this->array.getCols(), this->array.getRows(), PageAligned );

    f.memcpyFromDevice( array );

    unsigned char* c = new unsigned char[ f.getCols() * f.getRows() ];
    for( int y=0; y<f.getRows(); y++ ) {
        for( int x=0; x<f.getCols(); x++ ) {
            c[y*f.getCols()+x] = (unsigned char)(f.ptr(y)[x]);
        }
    }
    ofstream of( filename );
    of << "P5" << endl
       << f.getCols() << " " << f.getRows() << endl
       << "255" << endl;
    of.write( (char*)c, f.getCols() * f.getRows() );
    delete [] c;

    f.freeHost( PageAligned );
}

Image::Image( size_t w, size_t h )
{
    array.allocDev( w, h );
}

Image::~Image( )
{
    array.freeDev( );
}

} // namespace popart

