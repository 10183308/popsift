#include "hip/hip_runtime.h"
#include "s_image.h"
#include "clamp.h"
#include "assist.h"

#include <iostream>
#include <sstream>
#include <map>

#undef FIND_BLOCK_SIZE

using namespace std;

namespace popart {

__global__
void p_upscale_5( Plane2D_float dst, hipTextureObject_t src )
{
    int idx  = blockIdx.x * blockDim.x + threadIdx.x;
    int idy  = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= dst.getCols() ) return;
    if( idy >= dst.getRows() ) return;
    // const float src_x = (float(idx))/float(dst.getCols());
    // const float src_y = (float(idy))/float(dst.getRows());
    const float src_x = (float(idx)+0.5f)/float(dst.getCols());
    const float src_y = (float(idy)+0.5f)/float(dst.getRows());
    // const float src_x = (float(idx)-0.5f)/float(dst.getCols());
    // const float src_y = (float(idy)-0.5f)/float(dst.getRows());
    float d = tex2D<float>( src, src_x, src_y );
    dst.ptr(idy)[idx] = d * 255.0f;
}

#ifdef FIND_BLOCK_SIZE
int condition[][2] = {
    // { 1, 1 }, { 8, 1 },
    { 32, 1 },
    { 64, 1 },
    { 128, 1 }, // this is the winner for GeForce GT 650M, CC 3.0 (MAC)
    { 256, 1 },
    { 1024, 1 },
    // { 1, 2 }, { 8, 2 },
    { 32, 2 }, { 64, 2 }, { 128, 2 }, { 256, 2 }, { 512, 2 },
    { 8, 8 }, { 32, 8 }, { 64, 8 }, { 128, 8 },
    { 32, 32 },
    { 0, 0 } };
__host__
void Image::upscale_v5( hipTextureObject_t & tex )
{
    std::map<float,string> logtimes;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    for( int cond=0; condition[cond][0]!=0; cond++ ) {
        int blockx = condition[cond][0];
        int blocky = condition[cond][1];

        int loops  = 100;

        hipEventRecord( start, 0 );
        for( int i=0; i<loops; i++ ) {
            int gridx = grid_divide( this->_upscaled_image_d.getCols(), blockx );
            int gridy = grid_divide( this->_upscaled_image_d.getRows(), blocky );
            dim3 grid( gridx, gridy );
            dim3 block( blockx, blocky );

            p_upscale_5<<<grid,block>>> ( this->_upscaled_image_d, tex );
        }
        hipEventRecord( stop, 0 );
        hipDeviceSynchronize( );
        float diff;
        hipEventElapsedTime( &diff, start, stop );

        std::pair<float,string> datum;
        ostringstream ostr;
        ostr << "(" << blockx << ", " << blocky << ")";
        datum.first  = diff/loops;
        datum.second = ostr.str();
        logtimes.insert( datum );
    }

    std::map<float,string>::const_iterator it  = logtimes.begin();
    std::map<float,string>::const_iterator end = logtimes.end();
    for( ; it != end; it++ ) {
        const std::pair<float,string>& g = *it;
        cerr << "avg times: " << g.first
             << ": " << g.second
             << endl;
    }
    hipEventDestroy( start );
    hipEventDestroy( stop );
}
#else // not FIND_BLOCK_SIZE
__host__
void Image::upscale_v5( hipTextureObject_t & tex )
{
    dim3 block( 64, 2 );
    int gridx = grid_divide( this->_upscaled_image_d.getCols(), block.x );
    int gridy = grid_divide( this->_upscaled_image_d.getRows(), block.y );
    dim3 grid( gridx, gridy );

    p_upscale_5
        <<<grid,block>>>
        ( this->_upscaled_image_d,
          tex );

    test_last_error( __FILE__,  __LINE__ );
}
#endif // not FIND_BLOCK_SIZE

} // namespace popart

