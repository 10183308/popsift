#include "hip/hip_runtime.h"
#include "s_extrema.v4.h"
#include "debug_macros.hpp"

#define DEBUG_MODE 1

#define DEBUG_

/*************************************************************
 * V5: device side
 *************************************************************/
__device__ __constant__ float d_sigma0;
__device__ __constant__ float d_sigma_k;

__device__
inline void extremum_cmp_v4( float val, float f, uint32_t& gt, uint32_t& lt, uint32_t mask )
{
    gt |= ( ( val > f ) ? mask : 0 );
    lt |= ( ( val < f ) ? mask : 0 );
}

__device__
inline uint32_t extrema_count_v4( uint32_t indicator, ExtremaMgmt* mgmt )
{
    uint32_t mask = __ballot( indicator ); // bitfield of warps with results

    uint32_t ct = __popc( mask );          // horizontal reduce

    uint32_t leader = __ffs(mask) - 1;     // the highest thread id with indicator==true

    uint32_t write_index;
    if( threadIdx.x == leader ) {
        // atomicAdd returns the old value, we consider this the based
        // index for this thread's write operation
        write_index = atomicAdd( &mgmt->counter, ct );
    }
    // broadcast from leader thread to all threads in warp
    write_index = __shfl( write_index, leader );

    // this thread's offset: count only bits below the bit of the own
    // thread index; this provides the 0 result and every result up to ct
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) );

    return write_index;
}

__device__
inline bool is_extremum_v4( float* dog[3],
                            uint32_t y0, uint32_t y1, uint32_t y2,
                            uint32_t x0, uint32_t x1, uint32_t x2 )
{
    // somewhat annoying: to read center val, x1==31 requires a second 128-byte read
    // so: read left value first (one 128-byte read)
    //     read right value after (30 floats from cache, 2 from next 128-byte read)
    //     finally, read center value (from cache)
    uint32_t gt = 0;
    uint32_t lt = 0;

    float val0 = dog[1][y1+x0];
    float val2 = dog[1][y1+x2];
    float val  = dog[1][y1+x1];

    // bit indeces for neighbours:
    //     7 0 1    0x80 0x01 0x02
    //     6   2 -> 0x40      0x04
    //     5 4 3    0x20 0x10 0x08
    // upper layer << 24 ; own layer << 16 ; lower layer << 8
    // 1st group: left and right neigbhour
    extremum_cmp_v4( val, val0, gt, lt, 0x00400000 ); // ( 0x01<<6 ) << 16
    extremum_cmp_v4( val, val2, gt, lt, 0x00040000 ); // ( 0x01<<2 ) << 16

    if( ( gt != 0x00440000 ) && ( lt != 0x00440000 ) ) return false;

    // 2nd group: requires a total of 8 128-byte reads
    extremum_cmp_v4( val, dog[1][y0+x0], gt, lt, 0x00800000 ); // ( 0x01<<7 ) << 16
    extremum_cmp_v4( val, dog[1][y2+x0], gt, lt, 0x00200000 ); // ( 0x01<<5 ) << 16
    extremum_cmp_v4( val, dog[0][y0+x0], gt, lt, 0x80000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp_v4( val, dog[0][y2+x0], gt, lt, 0x40000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp_v4( val, dog[0][y1+x0], gt, lt, 0x20000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp_v4( val, dog[2][y0+x0], gt, lt, 0x00008000 ); // ( 0x01<<6 ) <<  8
    extremum_cmp_v4( val, dog[2][y1+x0], gt, lt, 0x00004000 ); // ( 0x01<<6 ) <<  8
    extremum_cmp_v4( val, dog[2][y2+x0], gt, lt, 0x00002000 ); // ( 0x01<<6 ) <<  8

    if( ( gt != 0xe0e4e000 ) && ( lt != 0xe0e4e000 ) ) return false;

    // 3rd group: remaining 2 cache misses in own layer
    extremum_cmp_v4( val, dog[1][y0+x1], gt, lt, 0x00010000 ); // ( 0x01<<0 ) << 16
    extremum_cmp_v4( val, dog[1][y0+x2], gt, lt, 0x00020000 ); // ( 0x01<<1 ) << 16
    extremum_cmp_v4( val, dog[1][y2+x1], gt, lt, 0x00100000 ); // ( 0x01<<4 ) << 16
    extremum_cmp_v4( val, dog[1][y2+x2], gt, lt, 0x00080000 ); // ( 0x01<<3 ) << 16

    if( ( gt != 0xe0ffe000 ) && ( lt != 0xe0ffe000 ) ) return false;

    // 4th group: 3 cache misses higher layer
    extremum_cmp_v4( val, dog[0][y0+x1], gt, lt, 0x01000000 ); // ( 0x01<<0 ) << 24
    extremum_cmp_v4( val, dog[0][y0+x2], gt, lt, 0x02000000 ); // ( 0x01<<1 ) << 24
    extremum_cmp_v4( val, dog[0][y1+x1], gt, lt, 0x00000004 ); // ( 0x01<<2 )
    extremum_cmp_v4( val, dog[0][y1+x2], gt, lt, 0x04000000 ); // ( 0x01<<2 ) << 24
    extremum_cmp_v4( val, dog[0][y2+x1], gt, lt, 0x10000000 ); // ( 0x01<<4 ) << 24
    extremum_cmp_v4( val, dog[0][y2+x2], gt, lt, 0x08000000 ); // ( 0x01<<3 ) << 24

    if( ( gt != 0xffffe004 ) && ( lt != 0xffffe004 ) ) return false;

    // 5th group: 3 cache misses lower layer
    extremum_cmp_v4( val, dog[2][y0+x1], gt, lt, 0x00000100 ); // ( 0x01<<0 ) <<  8
    extremum_cmp_v4( val, dog[2][y0+x2], gt, lt, 0x00000200 ); // ( 0x01<<1 ) <<  8
    extremum_cmp_v4( val, dog[2][y1+x1], gt, lt, 0x00000001 ); // ( 0x01<<0 )
    extremum_cmp_v4( val, dog[2][y1+x2], gt, lt, 0x00000400 ); // ( 0x01<<2 ) <<  8
    extremum_cmp_v4( val, dog[2][y2+x1], gt, lt, 0x00001000 ); // ( 0x01<<4 ) <<  8
    extremum_cmp_v4( val, dog[2][y2+x2], gt, lt, 0x00000800 ); // ( 0x01<<3 ) <<  8

    if( ( gt != 0xffffff05 ) && ( lt != 0xffffff05 ) ) return false;
    
    return true;
}

__device__ bool solve( float A[3][3], float b[3] )
{
    // Gauss elimination
    for( int j = 0 ; j < 3 ; j++ ) {
            // look for leading pivot
            float maxa    = 0;
            float maxabsa = 0;
            int   maxi    = -1;
            for( int i = j ; i < 3 ; i++ ) {
                float a    = A[j][i];
                float absa = fabs( a );
                if ( absa > maxabsa ) {
                    maxa    = a;
                    maxabsa = absa;
                    maxi    = i;
                }
            }

            // singular?
            if( maxabsa < 1e-15 ) {
                return false;
            }

            int i = maxi;

            // swap j-th row with i-th row and
            // normalize j-th row
            for(int jj = j ; jj < 3 ; ++jj) {
                float tmp = A[jj][j];
                A[jj][j]  = A[jj][i];
                A[jj][i]  = tmp;
                A[jj][j] /= maxa;
            }
            float tmp = b[j];
            b[j]  = b[i];
            b[i]  = tmp;
            b[j] /= maxa;

            // elimination
            for(int ii = j+1 ; ii < 3 ; ++ii) {
                float x = A[j][ii];
                for( int jj = j ; jj < 3 ; jj++ ) {
                    A[jj][ii] -= x * A[jj][j];
                }
                b[ii] -= x * b[j] ;
            }
    }

    // backward substitution
    for( int i = 2 ; i > 0 ; i-- ) {
            float x = b[i] ;
            for( int ii = i-1 ; ii >= 0 ; ii-- ) {
                b[ii] -= x * A[i][ii];
            }
    }
    return true;
}

__device__ bool solve2( float i[3][3], float b[3] )
{
    float det0b = - i[1][2] * i[1][2];
    float det0a =   i[1][1] * i[2][2];
    float det0 = det0b + det0a;

    float det1b = - i[0][1] * i[2][2];
    float det1a =   i[1][2] * i[0][2];
    float det1 = det1b + det1a;

    float det2b = - i[1][1] * i[0][2];
    float det2a =   i[0][1] * i[1][2];
    float det2 = det2b + det2a;

    float det3b = - i[0][2] * i[0][2];
    float det3a =   i[0][0] * i[2][2];
    float det3 = det3b + det3a;

    float det4b = - i[0][0] * i[1][2];
    float det4a =   i[0][1] * i[0][2];
    float det4 = det4b + det4a;

    float det5b = - i[0][1] * i[0][1];
    float det5a =   i[0][0] * i[1][1];
    float det5 = det5b + det5a;

    float det;
    det  = ( i[0][0] * det0 );
    det += ( i[0][1] * det1 );
    det += ( i[0][2] * det2 );

    // float rsd = 1.0 / det;
    float rsd = __frcp_rn( det );

    i[0][0] = det0 * rsd;
    i[1][0] = det1 * rsd;
    i[2][0] = det2 * rsd;
    i[1][1] = det3 * rsd;
    i[1][2] = det4 * rsd;
    i[2][2] = det5 * rsd;
    i[0][1] = i[1][0];
    i[0][2] = i[2][0];
    i[2][1] = i[1][2];

    float vout[3];
    vout[0] = vout[1] = vout[2] = 0;
    for (   int y = 0;  y < 3;  y ++ ) {
        for ( int x = 0;  x < 3;  x ++ ) {
            vout[y] += ( i[y][x] * b[x] );
        }
    }
    b[0] = vout[0];
    b[1] = vout[1];
    b[2] = vout[2];

    return true;
}

__device__
bool find_extrema_in_dog_v4_bemap( float*             dog[3], // level-1, level, level+1
                                   float              edge_limit,
                                   float              threshold,
                                   const uint32_t     width,
                                   const uint32_t     pitch,
                                   const uint32_t     height,
                                   const uint32_t     level,
                                   const uint32_t     maxlevel,
                                   ExtremaMgmt*       d_extrema_mgmt,
                                   ExtremumCandidate* d_extrema )
{
    /*
     * First consideration: extrema cannot be found on any outermost edge,
     * one pixel on the left, right, upper, lower edge will never qualify.
     * Also, the upper and lower DoG layer will never qualify. So there is
     * no reason for selecting any of those pixel for the center of a 3x3x3
     * region.
     * Instead, I use groups of 32xHEIGHT threads that read from a 34x34x3 area,
     * but implicitly, they fetch * 64xHEIGHT+2x3 floats (bad luck).
     * To find maxima, compare first on the left edge of the 3x3x3 cube, ie.
     * a 1x3x3 area. If the rightmost 2 threads of a warp (x==30 and 3==31)
     * are not extreme w.r.t. to the left slice, 8 fetch operations.
     */
    int32_t block_x = blockIdx.x * 32;
    int32_t block_y = blockIdx.y * blockDim.y;
    int32_t y       = block_y + threadIdx.y;
    int32_t x       = block_x + threadIdx.x;
    // int32_t z       = 0;

    if ( x+2 >= width ) {
        // atomicAdd( &debug_r.too_wide, 1 );
        return false;
    }
    if ( y+2 >= height ) {
        // atomicAdd( &debug_r.too_high, 1 );
        return false;
    }

    int32_t x0      = x;
    int32_t x1      = x+1;
    int32_t x2      = x+2;
    int32_t y0w     = y * pitch;
    int32_t y1w     = (y+1) * pitch;
    int32_t y2w     = (y+2) * pitch;

    float val = dog[1][y1w+x1];

    if( fabs( val ) < threshold ) {
        // atomicAdd( &debug_r.under_threshold, 1 );
        return false;
    }

    if( not is_extremum_v4( dog, y0w, y1w, y2w, x0, x1, x2 ) ) {
        // atomicAdd( &debug_r.not_extremum, 1 );
        return false;
    }

    // based on Bemap
    float Dx  = 0.0f;
    float Dy  = 0.0f;
    float Ds  = 0.0f;
    float Dxx = 0.0f;
    float Dyy = 0.0f;
    float Dss = 0.0f;
    float Dxy = 0.0f;
    float Dxs = 0.0f;
    float Dys = 0.0f;
    float dx  = 0.0f;
    float dy  = 0.0f;
    float ds  = 0.0f;

    float v = val;

    int32_t ni = y+1; // y1w;
    int32_t nj = x1;
    int32_t ns = level;

    int32_t tx = 0;
    int32_t ty = 0;
    int32_t ts = 0;

    int32_t iter;

    /* must be execute at least once */
    for ( iter = 0; iter < 5; iter++) {
        /* compute gradient */
        Dx = 0.5 * ( dog[1][y1w+x2] - dog[1][y1w+x0] );
        Dy = 0.5 * ( dog[1][y2w+x1] - dog[1][y2w+x1] );
        Ds = 0.5 * ( dog[2][y1w+x1] - dog[0][y1w+x1] );

        /* compute Hessian */
        Dxx = dog[1][y1w+x2] + dog[1][y1w+x0] - 2.0 * dog[1][y1w+x1];
        Dyy = dog[1][y2w+x1] + dog[1][y0w+x1] - 2.0 * dog[1][y1w+x1];
        Dss = dog[2][y1w+x1] + dog[0][y1w+x1] - 2.0 * dog[1][y1w+x1];

        Dxy = 0.25f * ( dog[1][y2w+x2] + dog[1][y0w+x0]
                      - dog[1][y2w+x0] - dog[1][y0w+x2] );
        Dxs = 0.25f * ( dog[2][y1w+x2] + dog[0][y1w+x0]
                      - dog[2][y1w+x0] - dog[0][y1w+x2] );
        Dys = 0.25f * ( dog[2][y2w+x1] + dog[0][y0w+x1]
                      - dog[0][y2w+x1] - dog[2][y0w+x1] );

        float b[3];
        float A[3][3];

        /* Solve linear system. */
        A[0][0] = Dxx;
        A[1][1] = Dyy;
        A[2][2] = Dss;
        A[1][0] = A[0][1] = Dxy;
        A[2][0] = A[0][2] = Dxs;
        A[2][1] = A[1][2] = Dys;

        b[0] = -Dx;
        b[1] = -Dy;
        b[2] = -Ds;

#if 0
        if( solve( A, b ) == false ) {
            dx = 0;
            dy = 0;
            ds = 0;
            break ;
        }
#else
        if( solve2( A, b ) == false ) {
            dx = 0;
            dy = 0;
            ds = 0;
            break ;
        }
#endif

        dx = b[0];
        dy = b[1];
        ds = b[2];

        /* If the translation of the keypoint is big, move the keypoint
         * and re-iterate the computation. Otherwise we are all set.
         */
        if( fabs(ds) < 0.5f && fabs(dy) < 0.5f && fabs(dx) < 0.5f) break;

        tx = ((dx >= 0.5f && nj < width-2) ?  1 : 0 )
           + ((dx <= -0.5f && nj > 1)? -1 : 0 );

        ty = ((dy >= 0.5f && ni < height-2)  ?  1 : 0 )
           + ((dy <= -0.5f && ni > 1) ? -1 : 0 );

        ts = ((ds >= 0.5f && ns < maxlevel-1)  ?  1 : 0 )
           + ((ds <= -0.5f && ns > 1) ? -1 : 0 );

        ni += ty;
        nj += tx;
        ns += ts;
    } /* go to next iter */

    /* ensure convergence of interpolation */
    if (iter >= 5) {
        // atomicAdd( &debug_r.convergence_failure, 1 );
        return false;
    }

    float contr   = v + 0.5f * (Dx * dx + Dy * dy + Ds * ds);
    float tr      = Dxx + Dyy;
    float det     = Dxx * Dyy - Dxy * Dxy;
    float edgeval = tr * tr / det;
    float xn      = nj + dx;
    float yn      = ni + dy;
    float sn      = ns + ds;

    /* negative determinant => curvatures have different signs -> reject it */
    if (det <= 0.0) {
        // atomicAdd( &debug_r.determinant_zero, 1 );
        return false;
    }

    /* accept-reject extremum */
    if( fabs(contr) < (threshold*2.0f) ) {
        // atomicAdd( &debug_r.thresh_exceeded, 1 );
        return false;
    }

    /* reject condition: tr(H)^2/det(H) < (r+1)^2/r */
    if( edgeval > (edge_limit+1.0f)*(edge_limit+1.0f)/edge_limit ) {
        // atomicAdd( &debug_r.edge_exceeded, 1 );
        return false;
    }

    uint32_t write_index = extrema_count_v4( true, d_extrema_mgmt );

    if( write_index >= d_extrema_mgmt->max1 ) {
        // atomicAdd( &debug_r.max_exceeded, 1 );
        return false;
    }
    // atomicAdd( &debug_r.continuing, 1 );
    // __syncthreads();

    ExtremumCandidate ec;
    ec.xpos    = xn;
    ec.ypos    = yn;
    ec.sigma   = d_sigma0 * pow(d_sigma_k, sn);
            // key_candidate->sigma = sigma0 * pow(sigma_k, sn);
        // ec.value   = 0;
        // ec.edge    = 0;
    ec.angle_from_bemap = 0;
    ec.not_a_keypoint   = 0;
    d_extrema[write_index] = ec;

    return true;
}

__global__
void find_extrema_in_dog_v4( float*             dog_upper,
                             float*             dog_here,
                             float*             dog_lower,
                             float              edge_limit,
                             float              threshold,
                             const uint32_t     width,
                             const uint32_t     pitch,
                             const uint32_t     height,
                             const uint32_t     level,
                             const uint32_t     maxlevel,
                             ExtremaMgmt*       mgmt_array,
                             ExtremumCandidate* d_extrema )
{
    float* dog_array[3];
    dog_array[0] = dog_upper;
    dog_array[1] = dog_here;
    dog_array[2] = dog_lower;

    ExtremaMgmt* mgmt = &mgmt_array[level];

    uint32_t indicator = find_extrema_in_dog_v4_bemap( dog_array, edge_limit, threshold, width, pitch, height, level, maxlevel, mgmt, d_extrema );
}

__global__
void fix_extrema_count_v4( ExtremaMgmt* mgmt_array, uint32_t mgmt_level )
{
    ExtremaMgmt* mgmt = &mgmt_array[mgmt_level];

    mgmt->counter = min( mgmt->counter, mgmt->max1 );
    // printf("%s>%d - %d\n", __FILE__, __LINE__, mgmt->counter );
}

#if 0
__global__
void start_orientation_v4( ExtremumCandidate* extrema,
                           ExtremaMgmt*       mgmt,
                           const float*       layer,
                           int                layer_pitch,
                           int                layer_height )
{
    mgmt->counter = min( mgmt->counter, mgmt->max1 );

    compute_keypoint_orientations_v2
        <<<mgmt->counter,16>>>
        ( extrema,
          mgmt,
          layer,
          layer_pitch,
          layer_height );
}
#endif

/*************************************************************
 * V4: host side
 *************************************************************/
__host__
void Pyramid::find_extrema_v4( uint32_t height, float edgeLimit, float threshold )
{
    cerr << "Entering " << __FUNCTION__ << " - bitfield, 32x" << height << " kernels" << endl;

#if 0
    hipDeviceSynchronize();
    ReturnReasons a;
    a.too_wide = 0;
    a.too_high = 0;
    a.under_threshold = 0;
    a.not_extremum = 0;
    a.convergence_failure = 0;
    a.determinant_zero = 0;
    a.thresh_exceeded = 0;
    a.edge_exceeded = 0;
    a.max_exceeded = 0;
    a.continuing = 0;
    hipMemcpyToSymbol(HIP_SYMBOL( debug_r), &a, sizeof(ReturnReasons), 0, hipMemcpyHostToDevice );
#endif

    _keep_time_extrema_v4.start();

    for( int octave=0; octave<_num_octaves; octave++ ) {
        for( int level=1; level<_levels-1; level++ ) {
            dim3 block;
            dim3 grid;
            grid.x  = _octaves[octave].getPitch()  / 32;
            grid.y  = _octaves[octave].getHeight() / height;
            block.x = 32;
            block.y = height;

            find_extrema_in_dog_v4
                <<<grid,block,0,_stream>>>
                ( _octaves[octave].getDogData( level-1 ),
                  _octaves[octave].getDogData( level ),
                  _octaves[octave].getDogData( level+1 ),
                  edgeLimit,
                  threshold,
                  _octaves[octave].getWidth( ),
                  _octaves[octave].getPitch( ),
                  _octaves[octave].getHeight( ),
                  level,
                  _levels,
                  _octaves[octave].getExtremaMgmtD( ),
                  _octaves[octave].getExtrema( level ) );
#if 1
            fix_extrema_count_v4
                <<<1,1,0,_stream>>>
                ( _octaves[octave].getExtremaMgmtD( ),
                  level );
#else
    // this does not work yet: I have no idea how to link with CUDA
    // and still achieve dynamic parallelism
            start_orientation_v4
                <<<1,1,0,_stream>>>
                ( _octaves[octave].getExtrema( level ),
                  _octaves[octave].getExtremaMgmtD( level ),
                  _octaves[octave].getDogData( level ),
                  _octaves[octave].getPitch( ),
                  _octaves[octave].getHeight( ) );
#endif
        }
    }
    hipError_t err = hipGetLastError();
    POP_CUDA_FATAL_TEST( err, "find_extrema_in_dog_v4 failed: " );

    _keep_time_extrema_v4.stop();

#if 0
    hipDeviceSynchronize();
    hipMemcpyFromSymbol( &a, HIP_SYMBOL(debug_r), sizeof(ReturnReasons), 0, hipMemcpyDeviceToHost );
    cerr << __FILE__ << ":" << __LINE__ << endl
         << "reasons for returning:" << endl
         << "  too wide: " << a.too_wide << endl
         << "  too high: " << a.too_high << endl
         << "  under threshold: " << a.under_threshold << endl
         << "  not extremum: " << a.not_extremum << endl
         << "  convergence failure: " << a.convergence_failure << endl
         << "  determinant zero: " << a.determinant_zero << endl
         << "  threshold exceeded: " << a.thresh_exceeded << endl
         << "  edge limit exceeded: " << a.edge_exceeded << endl
         << "  max exceeded: " << a.max_exceeded << endl
         << "  everything OK: " << a.continuing << endl
         << endl;
#endif
}

void Pyramid::init_sigma( float sigma0, uint32_t levels, hipStream_t stream )
{
    hipError_t err;

    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( d_sigma0), &sigma0,
                                   sizeof(float), 0,
                                   hipMemcpyHostToDevice,
                                   stream );
    POP_CUDA_FATAL_TEST( err, "Failed to upload sigma0 to device: " );

    const float sigma_k = powf(2.0f, 1.0f / levels );

    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( d_sigma_k), &sigma_k,
                                   sizeof(float), 0,
                                   hipMemcpyHostToDevice,
                                   stream );
    POP_CUDA_FATAL_TEST( err, "Failed to upload sigma_k to device: " );
}

