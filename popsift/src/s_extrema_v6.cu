#include "hip/hip_runtime.h"
#include "sift_pyramid.h"
#include "s_sigma.h"
#include "s_solve.h"
#include "debug_macros.h"
#include "assist.h"
#include "clamp.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace popart{

/*************************************************************
 * V6 (with dog array): device side
 *************************************************************/

template<int HEIGHT>
__device__
static
inline uint32_t extrema_count( int indicator, int* extrema_counter )
{
    uint32_t mask = __ballot( indicator ); // bitfield of warps with results

    uint32_t ct = __popc( mask );          // horizontal reduce

    int write_index;
    if( threadIdx.x == 0 ) {
        // atomicAdd returns the old value, we consider this the based
        // index for this thread's write operation
        write_index = atomicAdd( extrema_counter, ct );
    }
    // broadcast from thread 0 to all threads in warp
    write_index = __shfl( write_index, 0 );

    // this thread's offset: count only bits below the bit of the own
    // thread index; this provides the 0 result and every result up to ct
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) );

    return write_index;
}

__device__
static
inline void extremum_cmp( float val, float f, uint32_t& gt, uint32_t& lt, uint32_t mask )
{
    gt |= ( ( val > f ) ? mask : 0 );
    lt |= ( ( val < f ) ? mask : 0 );
}


#define TX(dx,dy,dz) tex2DLayered<float>( obj, x+dx, y+dy, z+dz )

__device__
static
inline bool is_extremum( hipTextureObject_t obj,
                         int x, int y, int z )
{
    uint32_t gt = 0;
    uint32_t lt = 0;

    float val0 = TX( 0, 1, 1 );
    float val2 = TX( 2, 1, 1 );
    float val  = TX( 1, 1, 1 );

    // bit indeces for neighbours:
    //     7 0 1    0x80 0x01 0x02
    //     6   2 -> 0x40      0x04
    //     5 4 3    0x20 0x10 0x08
    // upper layer << 24 ; own layer << 16 ; lower layer << 8
    // 1st group: left and right neigbhour
    extremum_cmp( val, val0, gt, lt, 0x00400000 ); // ( 0x01<<6 ) << 16
    extremum_cmp( val, val2, gt, lt, 0x00040000 ); // ( 0x01<<2 ) << 16

    if( ( gt != 0x00440000 ) && ( lt != 0x00440000 ) ) return false;

    // 2nd group: requires a total of 8 128-byte reads
    extremum_cmp( val, TX(0,0,1), gt, lt, 0x00800000 ); // ( 0x01<<7 ) << 16
    extremum_cmp( val, TX(0,2,1), gt, lt, 0x00200000 ); // ( 0x01<<5 ) << 16
    extremum_cmp( val, TX(0,0,0), gt, lt, 0x80000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp( val, TX(0,2,0), gt, lt, 0x40000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp( val, TX(0,1,0), gt, lt, 0x20000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp( val, TX(0,0,2), gt, lt, 0x00008000 ); // ( 0x01<<6 ) <<  8
    extremum_cmp( val, TX(0,1,2), gt, lt, 0x00004000 ); // ( 0x01<<6 ) <<  8
    extremum_cmp( val, TX(0,2,2), gt, lt, 0x00002000 ); // ( 0x01<<6 ) <<  8

    if( ( gt != 0xe0e4e000 ) && ( lt != 0xe0e4e000 ) ) return false;

    // 3rd group: remaining 2 cache misses in own layer
    extremum_cmp( val, TX(1,0,1), gt, lt, 0x00010000 ); // ( 0x01<<0 ) << 16
    extremum_cmp( val, TX(2,0,1), gt, lt, 0x00020000 ); // ( 0x01<<1 ) << 16
    extremum_cmp( val, TX(1,2,1), gt, lt, 0x00100000 ); // ( 0x01<<4 ) << 16
    extremum_cmp( val, TX(2,2,1), gt, lt, 0x00080000 ); // ( 0x01<<3 ) << 16

    if( ( gt != 0xe0ffe000 ) && ( lt != 0xe0ffe000 ) ) return false;

    // 4th group: 3 cache misses higher layer
    extremum_cmp( val, TX(1,0,0), gt, lt, 0x01000000 ); // ( 0x01<<0 ) << 24
    extremum_cmp( val, TX(2,0,0), gt, lt, 0x02000000 ); // ( 0x01<<1 ) << 24
    extremum_cmp( val, TX(1,1,0), gt, lt, 0x00000004 ); // ( 0x01<<2 )
    extremum_cmp( val, TX(2,1,0), gt, lt, 0x04000000 ); // ( 0x01<<2 ) << 24
    extremum_cmp( val, TX(1,2,0), gt, lt, 0x10000000 ); // ( 0x01<<4 ) << 24
    extremum_cmp( val, TX(2,2,0), gt, lt, 0x08000000 ); // ( 0x01<<3 ) << 24

    if( ( gt != 0xffffe004 ) && ( lt != 0xffffe004 ) ) return false;

    // 5th group: 3 cache misss lower layer
    extremum_cmp( val, TX(1,0,2), gt, lt, 0x00000100 ); // ( 0x01<<0 ) <<  8
    extremum_cmp( val, TX(2,0,2), gt, lt, 0x00000200 ); // ( 0x01<<1 ) <<  8
    extremum_cmp( val, TX(1,1,2), gt, lt, 0x00000001 ); // ( 0x01<<0 )
    extremum_cmp( val, TX(2,1,2), gt, lt, 0x00000400 ); // ( 0x01<<2 ) <<  8
    extremum_cmp( val, TX(1,2,2), gt, lt, 0x00001000 ); // ( 0x01<<4 ) <<  8
    extremum_cmp( val, TX(2,2,2), gt, lt, 0x00000800 ); // ( 0x01<<3 ) <<  8

    if( ( gt != 0xffffff05 ) && ( lt != 0xffffff05 ) ) return false;

    return true;
}

__device__
bool find_extrema_in_dog_v6_sub( hipTextureObject_t dog,
                                 int                 level,
                                 int                 width,
                                 int                 height,
                                 const uint32_t      maxlevel,
                                 Extremum&           ec )
{
    ec.xpos    = 0;
    ec.ypos    = 0;
    ec.sigma   = 0;
    ec.orientation = 0;

    /*
     * First consideration: extrema cannot be found on any outermost edge,
     * one pixel on the left, right, upper, lower edge will never qualify.
     * Also, the upper and lower DoG layer will never qualify. So there is
     * no reason for selecting any of those pixel for the center of a 3x3x3
     * region.
     * Instead, I use groups of 32xHEIGHT threads that read from a 34x34x3 area,
     * but implicitly, they fetch * 64xHEIGHT+2x3 floats (bad luck).
     * To find maxima, compare first on the left edge of the 3x3x3 cube, ie.
     * a 1x3x3 area. If the rightmost 2 threads of a warp (x==30 and 3==31)
     * are not extreme w.r.t. to the left slice, 8 fetch operations.
     */
    int32_t block_x = blockIdx.x * 32;
    int32_t block_y = blockIdx.y * blockDim.y;
    int32_t y       = block_y + threadIdx.y;
    int32_t x       = block_x + threadIdx.x;

    // int32_t x0 = x;
    // int32_t x1 = x+1;
    // int32_t x2 = x+2;
    // int32_t y0 = y;
    // int32_t y1 = y+1;
    // int32_t y2 = y+2;

    float val = tex2DLayered<float>( dog, x+1, y+1, level );

    if( fabs( val ) < d_threshold ) {
        return false;
    }

    if( not is_extremum( dog, x, y, level-1 ) ) {
        return false;
    }

    // based on Bemap
    float Dx  = 0.0f;
    float Dy  = 0.0f;
    float Ds  = 0.0f;
    float Dxx = 0.0f;
    float Dyy = 0.0f;
    float Dss = 0.0f;
    float Dxy = 0.0f;
    float Dxs = 0.0f;
    float Dys = 0.0f;
    float dx  = 0.0f;
    float dy  = 0.0f;
    float ds  = 0.0f;

    float v = val;

    int32_t ni = y+1; // y1w;
    int32_t nj = x+1;
    int32_t ns = level;

    int32_t tx = 0;
    int32_t ty = 0;
    int32_t ts = 0;

    int32_t iter;

    /* must be execute at least once */
    for ( iter = 0; iter < 5; iter++) {
        const int z = level - 1;
        /* compute gradient */
        const float x2y1z1 = tex2DLayered<float>( dog, x+2, y+1, z+1 );
        const float x0y1z1 = tex2DLayered<float>( dog, x+0, y+1, z+1 );
        const float x1y2z1 = tex2DLayered<float>( dog, x+1, y+2, z+1 );
        const float x1y0z1 = tex2DLayered<float>( dog, x+1, y+0, z+1 );
        const float x1y1z2 = tex2DLayered<float>( dog, x+1, y+1, z+2 );
        const float x1y1z0 = tex2DLayered<float>( dog, x+1, y+1, z+0 );
        Dx = 0.5 * ( x2y1z1 - x0y1z1 );
        Dy = 0.5 * ( x1y2z1 - x1y0z1 );
        Ds = 0.5 * ( x1y1z2 - x1y1z0 );

        /* compute Hessian */
        const float x1y1z1 = tex2DLayered<float>( dog, x+1, y+1, z+1 );
        Dxx = x2y1z1 + x0y1z1 - 2.0 * x1y1z1;
        Dyy = x1y2z1 + x1y0z1 - 2.0 * x1y1z1;
        Dss = x1y1z2 + x1y1z0 - 2.0 * x1y1z1;

        const float x0y0z1 = tex2DLayered<float>( dog, x+0, y+0, z+1 );
        const float x0y1z0 = tex2DLayered<float>( dog, x+0, y+1, z+0 );
        const float x0y1z2 = tex2DLayered<float>( dog, x+0, y+1, z+2 );
        const float x0y2z1 = tex2DLayered<float>( dog, x+0, y+2, z+1 );
        const float x1y0z0 = tex2DLayered<float>( dog, x+1, y+0, z+0 );
        const float x1y0z2 = tex2DLayered<float>( dog, x+1, y+0, z+2 );
        const float x1y2z0 = tex2DLayered<float>( dog, x+1, y+2, z+0 );
        const float x1y2z2 = tex2DLayered<float>( dog, x+1, y+2, z+2 );
        const float x2y0z1 = tex2DLayered<float>( dog, x+2, y+0, z+1 );
        const float x2y1z0 = tex2DLayered<float>( dog, x+2, y+1, z+0 );
        const float x2y1z2 = tex2DLayered<float>( dog, x+2, y+1, z+2 );
        const float x2y2z1 = tex2DLayered<float>( dog, x+2, y+2, z+1 );
        Dxy = 0.25f * ( x2y2z1 + x0y0z1 - x0y2z1 - x2y0z1 );
        Dxs = 0.25f * ( x2y1z2 + x0y1z0 - x0y1z2 - x2y1z0 );
        Dys = 0.25f * ( x1y2z2 + x1y0z0 - x1y2z0 - x1y0z2 );

        float b[3];
        float A[3][3];

        /* Solve linear system. */
        A[0][0] = Dxx;
        A[1][1] = Dyy;
        A[2][2] = Dss;
        A[1][0] = A[0][1] = Dxy;
        A[2][0] = A[0][2] = Dxs;
        A[2][1] = A[1][2] = Dys;

        b[0] = -Dx;
        b[1] = -Dy;
        b[2] = -Ds;

        if( solve( A, b ) == false ) {
            dx = 0;
            dy = 0;
            ds = 0;
            break ;
        }

        dx = b[0];
        dy = b[1];
        ds = b[2];

        /* If the translation of the keypoint is big, move the keypoint
         * and re-iterate the computation. Otherwise we are all set.
         */
        if( fabs(ds) < 0.5f && fabs(dy) < 0.5f && fabs(dx) < 0.5f) break;

        tx = ((dx >= 0.5f && nj < width-2) ?  1 : 0 )
             + ((dx <= -0.5f && nj > 1)? -1 : 0 );

        ty = ((dy >= 0.5f && ni < height-2)  ?  1 : 0 )
             + ((dy <= -0.5f && ni > 1) ? -1 : 0 );

        ts = ((ds >= 0.5f && ns < maxlevel-1)  ?  1 : 0 )
             + ((ds <= -0.5f && ns > 1) ? -1 : 0 );

        ni += ty;
        nj += tx;
        ns += ts;
    } /* go to next iter */

    /* ensure convergence of interpolation */
    if (iter >= 5) {
        return false;
    }

    float contr   = v + 0.5f * (Dx * dx + Dy * dy + Ds * ds);
    float tr      = Dxx + Dyy;
    float det     = Dxx * Dyy - Dxy * Dxy;
    float edgeval = tr * tr / det;
    float xn      = nj + dx;
    float yn      = ni + dy;
    float sn      = ns + ds;

    /* negative determinant => curvatures have different signs -> reject it */
    if (det <= 0.0) {
        return false;
    }

    /* accept-reject extremum */
    if( fabs(contr) < (d_threshold*2.0f) ) {
        return false;
    }

    /* reject condition: tr(H)^2/det(H) < (r+1)^2/r */
    if( edgeval >= (d_edge_limit+1.0f)*(d_edge_limit+1.0f)/d_edge_limit ) {
        return false;
    }

    ec.xpos    = xn;
    ec.ypos    = yn;
    ec.sigma   = d_sigma0 * pow(d_sigma_k, sn);
    // key_candidate->sigma = sigma0 * pow(sigma_k, sn);
    // ec.value   = 0;
    // ec.edge    = 0;
    ec.orientation = 0;

    return true;
}



template<int HEIGHT>
__global__
void find_extrema_in_dog_v6( hipTextureObject_t dog,
                             int                 level,
                             int                 width,
                             int                 height,
                             const uint32_t      maxlevel,
                             int*                extrema_counter,
                             Extremum*           d_extrema,
                             int*                d_number_of_blocks,
                             int                 number_of_blocks )
{
    Extremum ec;

    bool indicator = find_extrema_in_dog_v6_sub( dog, level, width, height, maxlevel, ec );

    uint32_t write_index = extrema_count<HEIGHT>( indicator, extrema_counter );

    if( indicator && write_index < d_max_extrema ) {
        d_extrema[write_index] = ec;
    }

    // without syncthreads, (0,0) threads may precede some calls to extrema_count()
    // in non-(0,0) threads and increase barrier count too early
    __syncthreads();

    // __threadfence(); probably not needed

    if( threadIdx.x == 0 && threadIdx.y == 0 ) {
        int ct = atomicAdd( d_number_of_blocks, 1 );
        if( ct >= number_of_blocks-1 ) {
            int num_ext = atomicMin( extrema_counter, d_max_extrema );
            // printf("counted to %d, num extrema %d\n", ct, num_ext );
            printf("Number of extrema at level %d: %d\n", level, num_ext );
        }
    }
}


/*************************************************************
 * V6: host side
 *************************************************************/
template<int HEIGHT>
__host__
void Pyramid::find_extrema_v6_sub( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        hipEvent_t  reset_done_ev  = oct_obj.getEventExtremaDone(0);

        int*  extrema_counters   = oct_obj.getExtremaMgmtD( );
        int*  extrema_num_blocks = oct_obj.getNumberOfBlocks( );

        for( int level=1; level<_levels-2; level++ ) {
            int cols = oct_obj.getWidth();
            int rows = oct_obj.getHeight();

            dim3 block( 32, HEIGHT );
            dim3 grid;
            grid.x  = grid_divide( cols, block.x );
            grid.y  = grid_divide( rows, block.y );

            hipStream_t oct_str = oct_obj.getStream(level+2);

            hipEvent_t  upp_ev  = oct_obj.getEventDogDone(level+0);
            hipEvent_t  mid_ev  = oct_obj.getEventDogDone(level+1);
            // hipEvent_t  low_ev  = oct_obj.getEventDogDone(level+2); - we are in the same stream

            int*  extrema_counter = &extrema_counters[level];
            int*  num_blocks      = &extrema_num_blocks[level];

            hipStreamWaitEvent( oct_str, reset_done_ev, 0 );
            hipStreamWaitEvent( oct_str, upp_ev, 0 );
            hipStreamWaitEvent( oct_str, mid_ev, 0 );
            // hipStreamWaitEvent( oct_str, low_ev, 0 ); - we are in the same stream

            find_extrema_in_dog_v6<HEIGHT>
                <<<grid,block,0,oct_str>>>
                ( oct_obj.getDogTexture( ),
                  level,
                  cols,
                  rows,
                  _levels,
                  extrema_counter,
                  oct_obj.getExtrema( level ),
                  num_blocks,
                  grid.x * grid.y );

            hipEvent_t  extrema_done_ev  = oct_obj.getEventExtremaDone(level+2);
            hipEventRecord( extrema_done_ev, oct_str );
        }
    }

    // hipDeviceSynchronize();
}

__host__
void Pyramid::find_extrema_v6( )
{
#define MANYLY(H) \
    find_extrema_v6_sub<H> ( );

    // MANYLY(1)
    // MANYLY(2)
    // MANYLY(3)
    MANYLY(4)
    // MANYLY(5)
    // MANYLY(6)
    // MANYLY(7)
    // MANYLY(8)
    // MANYLY(16)
    // fails // MANYLY(32)
}

} // namespace popart

