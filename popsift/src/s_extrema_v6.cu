#include "hip/hip_runtime.h"
#include "s_pyramid.h"
#include "s_sigma.h"
#include "s_solve.h"
#include "debug_macros.h"
#include "assist.h"
#include "clamp.h"
#include <hip/hip_runtime.h>

namespace popart{

/*************************************************************
 * V6 (with dog array): device side
 *************************************************************/

template<int HEIGHT>
__device__
static
inline uint32_t extrema_count( int indicator, ExtremaMgmt* mgmt )
{
    uint32_t mask = __ballot( indicator ); // bitfield of warps with results

    uint32_t ct = __popc( mask );          // horizontal reduce

    uint32_t write_index;
    if( threadIdx.x == 0 ) {
        // atomicAdd returns the old value, we consider this the based
        // index for this thread's write operation
        write_index = atomicAdd( &mgmt->counter, ct );
    }
    // broadcast from thread 0 to all threads in warp
    write_index = __shfl( write_index, 0 );

    // this thread's offset: count only bits below the bit of the own
    // thread index; this provides the 0 result and every result up to ct
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) );

    return write_index;
}

__device__
static
inline void extremum_cmp( float val, float f, uint32_t& gt, uint32_t& lt, uint32_t mask )
{
    gt |= ( ( val > f ) ? mask : 0 );
    lt |= ( ( val < f ) ? mask : 0 );
}


#define TX(dx,dy,dz) tex2DLayered<float>( obj, x+dx, y+dy, z+dz )

__device__
static
inline bool is_extremum( hipTextureObject_t obj,
                         int x, int y, int z )
{
    uint32_t gt = 0;
    uint32_t lt = 0;

    float val0 = TX( 0, 1, 1 );
    float val2 = TX( 2, 1, 1 );
    float val  = TX( 1, 1, 1 );

    // bit indeces for neighbours:
    //     7 0 1    0x80 0x01 0x02
    //     6   2 -> 0x40      0x04
    //     5 4 3    0x20 0x10 0x08
    // upper layer << 24 ; own layer << 16 ; lower layer << 8
    // 1st group: left and right neigbhour
    extremum_cmp( val, val0, gt, lt, 0x00400000 ); // ( 0x01<<6 ) << 16
    extremum_cmp( val, val2, gt, lt, 0x00040000 ); // ( 0x01<<2 ) << 16

    if( ( gt != 0x00440000 ) && ( lt != 0x00440000 ) ) return false;

    // 2nd group: requires a total of 8 128-byte reads
    extremum_cmp( val, TX(0,0,1), gt, lt, 0x00800000 ); // ( 0x01<<7 ) << 16
    extremum_cmp( val, TX(0,2,1), gt, lt, 0x00200000 ); // ( 0x01<<5 ) << 16
    extremum_cmp( val, TX(0,0,0), gt, lt, 0x80000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp( val, TX(0,2,0), gt, lt, 0x40000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp( val, TX(0,1,0), gt, lt, 0x20000000 ); // ( 0x01<<6 ) << 24
    extremum_cmp( val, TX(0,0,2), gt, lt, 0x00008000 ); // ( 0x01<<6 ) <<  8
    extremum_cmp( val, TX(0,1,2), gt, lt, 0x00004000 ); // ( 0x01<<6 ) <<  8
    extremum_cmp( val, TX(0,2,2), gt, lt, 0x00002000 ); // ( 0x01<<6 ) <<  8

    if( ( gt != 0xe0e4e000 ) && ( lt != 0xe0e4e000 ) ) return false;

    // 3rd group: remaining 2 cache misses in own layer
    extremum_cmp( val, TX(1,0,1), gt, lt, 0x00010000 ); // ( 0x01<<0 ) << 16
    extremum_cmp( val, TX(2,0,1), gt, lt, 0x00020000 ); // ( 0x01<<1 ) << 16
    extremum_cmp( val, TX(1,2,1), gt, lt, 0x00100000 ); // ( 0x01<<4 ) << 16
    extremum_cmp( val, TX(2,2,1), gt, lt, 0x00080000 ); // ( 0x01<<3 ) << 16

    if( ( gt != 0xe0ffe000 ) && ( lt != 0xe0ffe000 ) ) return false;

    // 4th group: 3 cache misses higher layer
    extremum_cmp( val, TX(1,0,0), gt, lt, 0x01000000 ); // ( 0x01<<0 ) << 24
    extremum_cmp( val, TX(2,0,0), gt, lt, 0x02000000 ); // ( 0x01<<1 ) << 24
    extremum_cmp( val, TX(1,1,0), gt, lt, 0x00000004 ); // ( 0x01<<2 )
    extremum_cmp( val, TX(2,1,0), gt, lt, 0x04000000 ); // ( 0x01<<2 ) << 24
    extremum_cmp( val, TX(1,2,0), gt, lt, 0x10000000 ); // ( 0x01<<4 ) << 24
    extremum_cmp( val, TX(2,2,0), gt, lt, 0x08000000 ); // ( 0x01<<3 ) << 24

    if( ( gt != 0xffffe004 ) && ( lt != 0xffffe004 ) ) return false;

    // 5th group: 3 cache misss lower layer
    extremum_cmp( val, TX(1,0,2), gt, lt, 0x00000100 ); // ( 0x01<<0 ) <<  8
    extremum_cmp( val, TX(2,0,2), gt, lt, 0x00000200 ); // ( 0x01<<1 ) <<  8
    extremum_cmp( val, TX(1,1,2), gt, lt, 0x00000001 ); // ( 0x01<<0 )
    extremum_cmp( val, TX(2,1,2), gt, lt, 0x00000400 ); // ( 0x01<<2 ) <<  8
    extremum_cmp( val, TX(1,2,2), gt, lt, 0x00001000 ); // ( 0x01<<4 ) <<  8
    extremum_cmp( val, TX(2,2,2), gt, lt, 0x00000800 ); // ( 0x01<<3 ) <<  8

    if( ( gt != 0xffffff05 ) && ( lt != 0xffffff05 ) ) return false;

    return true;
}

__device__
bool find_extrema_in_dog_v6_sub( hipTextureObject_t dog,
                                 int                 level,
                                 int                 width,
                                 int                 height,
                                 float               edge_limit,
                                 float               threshold,
                                 const uint32_t      maxlevel,
                                 ExtremumCandidate&  ec )
{
    ec.xpos    = 0;
    ec.ypos    = 0;
    ec.sigma   = 0;
    ec.orientation = 0;

    /*
     * First consideration: extrema cannot be found on any outermost edge,
     * one pixel on the left, right, upper, lower edge will never qualify.
     * Also, the upper and lower DoG layer will never qualify. So there is
     * no reason for selecting any of those pixel for the center of a 3x3x3
     * region.
     * Instead, I use groups of 32xHEIGHT threads that read from a 34x34x3 area,
     * but implicitly, they fetch * 64xHEIGHT+2x3 floats (bad luck).
     * To find maxima, compare first on the left edge of the 3x3x3 cube, ie.
     * a 1x3x3 area. If the rightmost 2 threads of a warp (x==30 and 3==31)
     * are not extreme w.r.t. to the left slice, 8 fetch operations.
     */
    int32_t block_x = blockIdx.x * 32;
    int32_t block_y = blockIdx.y * blockDim.y;
    int32_t y       = block_y + threadIdx.y;
    int32_t x       = block_x + threadIdx.x;

    // int32_t x0 = x;
    // int32_t x1 = x+1;
    // int32_t x2 = x+2;
    // int32_t y0 = y;
    // int32_t y1 = y+1;
    // int32_t y2 = y+2;

    float val = tex2DLayered<float>( dog, x+1, y+1, level );

    if( fabs( val ) < threshold ) {
        return false;
    }

    if( not is_extremum( dog, x, y, level-1 ) ) {
        return false;
    }

    // based on Bemap
    float Dx  = 0.0f;
    float Dy  = 0.0f;
    float Ds  = 0.0f;
    float Dxx = 0.0f;
    float Dyy = 0.0f;
    float Dss = 0.0f;
    float Dxy = 0.0f;
    float Dxs = 0.0f;
    float Dys = 0.0f;
    float dx  = 0.0f;
    float dy  = 0.0f;
    float ds  = 0.0f;

    float v = val;

    int32_t ni = y+1; // y1w;
    int32_t nj = x+1;
    int32_t ns = level;

    int32_t tx = 0;
    int32_t ty = 0;
    int32_t ts = 0;

    int32_t iter;

    /* must be execute at least once */
    for ( iter = 0; iter < 5; iter++) {
        const int z = level - 1;
        /* compute gradient */
        const float x2y1z1 = tex2DLayered<float>( dog, x+2, y+1, z+1 );
        const float x0y1z1 = tex2DLayered<float>( dog, x+0, y+1, z+1 );
        const float x1y2z1 = tex2DLayered<float>( dog, x+1, y+2, z+1 );
        const float x1y0z1 = tex2DLayered<float>( dog, x+1, y+0, z+1 );
        const float x1y1z2 = tex2DLayered<float>( dog, x+1, y+1, z+2 );
        const float x1y1z0 = tex2DLayered<float>( dog, x+1, y+1, z+0 );
        Dx = 0.5 * ( x2y1z1 - x0y1z1 );
        Dy = 0.5 * ( x1y2z1 - x1y0z1 );
        Ds = 0.5 * ( x1y1z2 - x1y1z0 );

        /* compute Hessian */
        const float x1y1z1 = tex2DLayered<float>( dog, x+1, y+1, z+1 );
        Dxx = x2y1z1 + x0y1z1 - 2.0 * x1y1z1;
        Dyy = x1y2z1 + x1y0z1 - 2.0 * x1y1z1;
        Dss = x1y1z2 + x1y1z0 - 2.0 * x1y1z1;

        const float x0y0z1 = tex2DLayered<float>( dog, x+0, y+0, z+1 );
        const float x0y1z0 = tex2DLayered<float>( dog, x+0, y+1, z+0 );
        const float x0y1z2 = tex2DLayered<float>( dog, x+0, y+1, z+2 );
        const float x0y2z1 = tex2DLayered<float>( dog, x+0, y+2, z+1 );
        const float x1y0z0 = tex2DLayered<float>( dog, x+1, y+0, z+0 );
        const float x1y0z2 = tex2DLayered<float>( dog, x+1, y+0, z+2 );
        const float x1y2z0 = tex2DLayered<float>( dog, x+1, y+2, z+0 );
        const float x1y2z2 = tex2DLayered<float>( dog, x+1, y+2, z+2 );
        const float x2y0z1 = tex2DLayered<float>( dog, x+2, y+0, z+1 );
        const float x2y1z0 = tex2DLayered<float>( dog, x+2, y+1, z+0 );
        const float x2y1z2 = tex2DLayered<float>( dog, x+2, y+1, z+2 );
        const float x2y2z1 = tex2DLayered<float>( dog, x+2, y+2, z+1 );
        Dxy = 0.25f * ( x2y2z1 + x0y0z1 - x0y2z1 - x2y0z1 );
        Dxs = 0.25f * ( x2y1z2 + x0y1z0 - x0y1z2 - x2y1z0 );
        Dys = 0.25f * ( x1y2z2 + x1y0z0 - x1y2z0 - x1y0z2 );

        float b[3];
        float A[3][3];

        /* Solve linear system. */
        A[0][0] = Dxx;
        A[1][1] = Dyy;
        A[2][2] = Dss;
        A[1][0] = A[0][1] = Dxy;
        A[2][0] = A[0][2] = Dxs;
        A[2][1] = A[1][2] = Dys;

        b[0] = -Dx;
        b[1] = -Dy;
        b[2] = -Ds;

        if( solve( A, b ) == false ) {
            dx = 0;
            dy = 0;
            ds = 0;
            break ;
        }

        dx = b[0];
        dy = b[1];
        ds = b[2];

        /* If the translation of the keypoint is big, move the keypoint
         * and re-iterate the computation. Otherwise we are all set.
         */
        if( fabs(ds) < 0.5f && fabs(dy) < 0.5f && fabs(dx) < 0.5f) break;

        tx = ((dx >= 0.5f && nj < width-2) ?  1 : 0 )
             + ((dx <= -0.5f && nj > 1)? -1 : 0 );

        ty = ((dy >= 0.5f && ni < height-2)  ?  1 : 0 )
             + ((dy <= -0.5f && ni > 1) ? -1 : 0 );

        ts = ((ds >= 0.5f && ns < maxlevel-1)  ?  1 : 0 )
             + ((ds <= -0.5f && ns > 1) ? -1 : 0 );

        ni += ty;
        nj += tx;
        ns += ts;
    } /* go to next iter */

    /* ensure convergence of interpolation */
    if (iter >= 5) {
        return false;
    }

    float contr   = v + 0.5f * (Dx * dx + Dy * dy + Ds * ds);
    float tr      = Dxx + Dyy;
    float det     = Dxx * Dyy - Dxy * Dxy;
    float edgeval = tr * tr / det;
    float xn      = nj + dx;
    float yn      = ni + dy;
    float sn      = ns + ds;

    /* negative determinant => curvatures have different signs -> reject it */
    if (det <= 0.0) {
        return false;
    }

    /* accept-reject extremum */
    if( fabs(contr) < (threshold*2.0f) ) {
        return false;
    }

    /* reject condition: tr(H)^2/det(H) < (r+1)^2/r */
    if( edgeval >= (edge_limit+1.0f)*(edge_limit+1.0f)/edge_limit ) {
        return false;
    }

    ec.xpos    = xn;
    ec.ypos    = yn;
    ec.sigma   = d_sigma0 * pow(d_sigma_k, sn);
    // key_candidate->sigma = sigma0 * pow(sigma_k, sn);
    // ec.value   = 0;
    // ec.edge    = 0;
    ec.orientation = 0;

    return true;
}



template<int HEIGHT>
__global__
void find_extrema_in_dog_v6( hipTextureObject_t dog,
                             int                 level,
                             int                 width,
                             int                 height,
                             float               edge_limit,
                             float               threshold,
                             const uint32_t      maxlevel,
                             ExtremaMgmt*        mgmt_array,
                             ExtremumCandidate*  d_extrema )
{
    ExtremaMgmt* mgmt = &mgmt_array[level];
    ExtremumCandidate ec;

    bool indicator = find_extrema_in_dog_v6_sub( dog, level, width, height, edge_limit, threshold, maxlevel, ec );

    uint32_t write_index = extrema_count<HEIGHT>( indicator, mgmt );

    if( indicator && write_index < mgmt->max1 ) {
        d_extrema[write_index] = ec;
    }
}


__global__
void reset_extrema_count_v6( ExtremaMgmt* mgmt_array, uint32_t mgmt_level )
{
    ExtremaMgmt* mgmt = &mgmt_array[mgmt_level];

    mgmt->counter = 0;
}

__global__
void fix_extrema_count_v6( ExtremaMgmt* mgmt_array, uint32_t mgmt_level )
{
    ExtremaMgmt* mgmt = &mgmt_array[mgmt_level];

    mgmt->counter = min( mgmt->counter, mgmt->max1 );
}

/*************************************************************
 * V6: host side
 *************************************************************/
template<int HEIGHT>
__host__
void Pyramid::find_extrema_v6_sub( float edgeLimit, float threshold )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        for( int level=1; level<_levels-2; level++ ) {
            int cols = _octaves[octave].getData(level).getCols();
            int rows = _octaves[octave].getData(level).getRows();
            dim3 block( 32, HEIGHT );
            dim3 grid;
            grid.x  = grid_divide( cols, block.x );
            grid.y  = grid_divide( rows, block.y );

            Octave&      oct_obj = _octaves[octave];
            hipStream_t oct_str = oct_obj.getStream(level);
            hipEvent_t  oct_ev  = oct_obj.getEventGaussDone(level+1);

            hipStreamWaitEvent( oct_str, oct_ev, 0 );

            reset_extrema_count_v6
                <<<1,1,0,oct_str>>>
                ( _octaves[octave].getExtremaMgmtD( ), level );

            find_extrema_in_dog_v6<HEIGHT>
                <<<grid,block,0,oct_str>>>
                ( _octaves[octave].getDogTexture( ),
                  level,
                  cols,
                  rows,
                  edgeLimit,
                  threshold,
                  _levels,
                  _octaves[octave].getExtremaMgmtD( ),
                  _octaves[octave].getExtrema( level ) );

#if 1
            fix_extrema_count_v6
                <<<1,1,0,oct_str>>>
                ( _octaves[octave].getExtremaMgmtD( ), level );
#else
            // this does not work yet: I have no idea how to link with CUDA
            // and still achieve dynamic parallelism
            start_orientation_v6
                <<<1,1>>>
                ( _octaves[octave].getExtrema( level ),
                  _octaves[octave].getExtremaMgmtD( level ),
                  d1,
                  _octaves[octave].getPitch( ),
                  _octaves[octave].getHeight( ) );
#endif
        }
    }
}

__host__
void Pyramid::find_extrema_v6( float edgeLimit, float threshold )
{
#define MANYLY(H) \
    find_extrema_v6_sub<H> ( edgeLimit, threshold );

    MANYLY(1)
    // MANYLY(2)
    // MANYLY(3)
    // MANYLY(4)
    // MANYLY(5)
    // MANYLY(6)
    // MANYLY(7)
    // MANYLY(8)
    // MANYLY(16)
    // fails // MANYLY(32)
}

} // namespace popart

