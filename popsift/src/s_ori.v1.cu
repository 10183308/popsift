#include "hip/hip_runtime.h"
#include "s_pyramid.h"
#include "s_gradiant.h"
#include "debug_macros.h"

#include <stdio.h>
#include <inttypes.h>

#define ORI_V1_NUM_THREADS 16
#define NBINS_V1           36
#define WINFACTOR_V1       1.5F

using namespace popart;

/*************************************************************
 * V1: device side
 *************************************************************/

__device__
inline float compute_angle( int bin, float hc, float hn, float hp )
{
    /* interpolate */
    float di = bin + 0.5f * (hn - hp) / (hc+hc-hn-hp);

    /* clamp */
    di = (di < 0) ? 
            (di + NBINS_V1) : 
            ((di >= NBINS_V1) ? (di - NBINS_V1) : (di));

    float th = ((M_PI2 * di) / NBINS_V1) - M_PI;
    // float th = ((M_PI2 * di) / NBINS_V1);
    return th;
}

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 */
__global__
void compute_keypoint_orientations_v1( ExtremumCandidate* extremum,
                                       ExtremaMgmt*       mgmt_array,
                                       uint32_t           mgmt_level,
                                       Plane2D_float      layer )
{
    uint32_t w   = layer.getWidth();
    uint32_t h   = layer.getHeight();

    ExtremaMgmt* mgmt = &mgmt_array[mgmt_level];

    // if( threadIdx.y >= mgmt->counter ) return;

    ExtremumCandidate* ext = &extremum[blockIdx.x];

    float hist[NBINS_V1];
    for (int i = 0; i < NBINS_V1; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = WINFACTOR_V1 * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = -0.5f / (sigw * sigw);
    int sq_thres  = rad * rad;
    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * exp(sq_dist * factor);

            // int bidx = (int)rintf(NBINS_V1 * (theta + M_PI) / M_PI2);
            int bidx = (int)roundf(NBINS_V1 * (theta + M_PI) / M_PI2);
            // int bidx = (int)roundf(NBINS_V1 * (theta + M_PI) / M_PI2 - 0.5f);

            if( bidx > NBINS_V1 ) {
                printf("Crashing: bin %d theta %f :-)\n", bidx, theta);
            }

            bidx = (bidx == NBINS_V1) ? 0 : bidx;

            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < NBINS_V1; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }


#define OLD_ORIENTATION

        if(threadIdx.x != 0) return;

    // for (int bin = 0; bin < NBINS_V1; bin++) {
        // printf( "%f %f %d %f\n", x, y, bin, hist[bin] );
    // }

#ifdef OLD_ORIENTATION
        for (int iter = 0; iter < 2; iter++) {
            float first = hist[0];
            float prev = hist[(NBINS_V1 - 1)];

            int bin;
            //0,35
            for (bin = 0; bin < NBINS_V1 - 1; bin++) {
                float temp = hist[bin];
                hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * hist[bin + 1];
                prev = temp;
            }

            hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * first;
            //z vprintf("val: %f, indx: %d\n", hist[bin], bin);
        }
	
        /* find histogram maximum */
        float maxh = NINF;
        int binh = 0;
        for (int bin = 0; bin < NBINS_V1; bin++) {
            // maxh = fmaxf(maxh, hist[bin]);
            if (hist[bin] > maxh) {
                maxh = hist[bin];
                binh = bin;
            }
        }

        {
            float hc = hist[binh];
            float hn = hist[((binh + 1 + NBINS_V1) % NBINS_V1)];
            float hp = hist[((binh - 1 + NBINS_V1) % NBINS_V1)];
            float th = compute_angle(binh, hc, hn, hp);

            if( isnan(th) ) {
                printf("NAN value in compute_angle\n");
            }

            ext->orientation = th;
        }

        /* find other peaks, boundary of 80% of max */
        int nangles = 1;

        for (int numloops = 1; numloops < NBINS_V1; numloops++) {
            int bin = (binh + numloops) % NBINS_V1;

            float hc = hist[bin];
            float hn = hist[((bin + 1 + NBINS_V1) % NBINS_V1)];
            float hp = hist[((bin - 1 + NBINS_V1) % NBINS_V1)];

            /* find if a peak */
            if (hc >= (0.8f * maxh) && hc > hn && hc > hp) {
                int idx = atomicAdd(&mgmt->counter, 1);
                if (idx >= mgmt->max2) break;

                float th = compute_angle(bin, hc, hn, hp);

                ext = &extremum[idx];
                ext->xpos = x;
                ext->ypos = y;
                ext->sigma = sig;
                ext->orientation = th;

                nangles++;
                if (nangles > 2) break;
            }
        }
#else // not OLD_ORIENTATION
        float xcoord[NBINS_V1];
        float yval[NBINS_V1];

        int   maxbin = 0;
        float y_max = 0;
        for(int bin = 0; bin < NBINS_V1; bin++) {
            int prev = bin - 1;
            if( prev < 0 ) prev = NBINS_V1 - 1;
            int next = bin + 1;
            if( next == NBINS_V1 ) next = 0;

            if( hist[bin] > max( hist[prev], hist[next] ) ) {
                const float num = 3.0f * hist[prev] - 4.0f * hist[bin] + hist[next];
                const float denB = 2.0f * ( hist[prev] - 2.0f * hist[bin] + hist[next] );
                float newbin = num / denB; // * M_PI/18.0f; // * 10.0f;
                if( newbin >= 0 && newbin <= 2 ) {
                    xcoord[bin] = prev + newbin;
                    yval[bin]   = -(num*num) / (4.0f * denB) + hist[prev];

                    if( yval[bin] > y_max ) {
                        y_max = yval[bin];
                        maxbin = bin;
                    }
                }
            }
        }
        float th = ((M_PI2 * xcoord[maxbin]) / NBINS_V1) - M_PI;

        ext->orientation = th;
#endif // not OLD_ORIENTATION
}

/*************************************************************
 * V4: host side
 *************************************************************/
#ifdef USE_DYNAMIC_PARALLELISM // defined in_s_pyramid.h

__global__
void orientation_starter_v1( ExtremumCandidate* extremum,
                             ExtremaMgmt*       mgmt_array,
                             uint32_t           mgmt_level,
                             Plane2D_float      layer )
{
    ExtremaMgmt* mgmt = &mgmt_array[mgmt_level];

    dim3 block;
    dim3 grid;
    grid.x  = mgmt->counter;
    block.x = ORI_V1_NUM_THREADS;

    if( grid.x != 0 ) {
        compute_keypoint_orientations_v1
            <<<grid,block>>>
            ( extremum,
              mgmt_array,
              mgmt_level,
              layer );
    }
}

__host__
void Pyramid::orientation_v1( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=1; level<_levels-1; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level);

            orientation_starter_v1
                <<<1,1,0,oct_str>>>
                ( oct_obj.getExtrema( level ),
                  oct_obj.getExtremaMgmtD( ),
                  level,
                  oct_obj.getData( level ) );
        }
    }
}

#else // not USE_DYNAMIC_PARALLELISM

__global__
void orientation_starter_v1( ExtremumCandidate*,
                             ExtremaMgmt*,
                             uint32_t,
                             Plane2D_float )
{
    /* dummy to make the linker happy */
}

__host__
void Pyramid::orientation_v1( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=1; level<_levels-1; level++ ) {
            hipStreamSynchronize( oct_obj.getStream(level) );
        }

        oct_obj.readExtremaCount( );
        hipDeviceSynchronize( );

        for( int level=1; level<_levels-1; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level);

            dim3 block;
            dim3 grid;
            // grid.x  = _octaves[octave].getExtremaMgmtH(level)->max1;
            grid.x  = oct_obj.getExtremaMgmtH(level)->counter;
            block.x = ORI_V1_NUM_THREADS;
            if( grid.x != 0 ) {
                compute_keypoint_orientations_v1
                    <<<grid,block,0,oct_str>>>
                    ( oct_obj.getExtrema( level ),
                      oct_obj.getExtremaMgmtD( ),
                      level,
                      oct_obj.getData( level ) );
            }
        }
    }
}
#endif // not USE_DYNAMIC_PARALLELISM

