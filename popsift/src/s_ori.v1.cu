#include "hip/hip_runtime.h"
#include "s_ori.v1.h"
#include "s_gradiant.h"

#define ORI_V1_NUM_THREADS 16
#define NBINS_V1           36
#define WINFACTOR_V1       1.5F

/*************************************************************
 * V1: device side
 *************************************************************/

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 */
__global__
void compute_keypoint_orientations_v1( ExtremumCandidate* extremum,
                                       ExtremaMgmt*       mgmt_array,
                                       uint32_t           mgmt_level,
                                       const float*       layer,
                                       int                layer_pitch,
                                       int                layer_height )
{
    uint32_t w   = layer_pitch;
    uint32_t h   = layer_height;

    ExtremaMgmt* mgmt = &mgmt_array[mgmt_level];

    if( threadIdx.y >= mgmt->counter ) return;

    ExtremumCandidate* ext = &extremum[threadIdx.y];

    float hist[NBINS_V1];
    for (int i = 0; i < NBINS_V1; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    float x    = ext->xpos;
    float y    = ext->ypos;
    float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = WINFACTOR_V1 * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = -0.5f / (sigw * sigw);
    int sq_thres  = rad * rad;
    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer,
                      layer_pitch,
                      layer_height );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * exp(sq_dist * factor);
            int bidx = (int)rintf(NBINS_V1 * (theta + M_PI) / M_PI2);
            bidx = (bidx < NBINS_V1) ? bidx : 0;
            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < NBINS_V1; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }

    if( threadIdx.x != 0 ) return;

    /* new oris */
    float ang[2] = {NINF, NINF};

    /* smooth histogram */
    for( int iter = 0; iter < 2; iter++ ) {
        float first = hist[0];
        float prev  = hist[(NBINS_V1-1)];

        int bin;
        for (bin = 0; bin < NBINS_V1 - 1; bin++) {
            float temp = hist[bin];
            hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * hist[bin+1];
            prev = temp;
        }
        hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * first;
    }

    /* find histogram maximum */
    float maxh = NINF;
    for (int bin = 0; bin < NBINS_V1; bin++) {
        maxh = fmaxf(maxh, hist[bin]);
    }

    /* find other peaks, boundary of 80% of max */
    int nangles = 0;
    for (int bin = 0; bin < NBINS_V1; bin++) {
        float hc = hist[bin];
        float hn = hist[((bin+1+NBINS_V1)%NBINS_V1)];
        float hp = hist[((bin-1+NBINS_V1)%NBINS_V1)];

        /* find if a peak */
        if (hc >= (0.8f * maxh) && hc > hn && hc > hp) {
    
            /* interpolate */
            float di = bin + 0.5f * (hn - hp) / (hc+hc-hn-hp);
            
            /* clamp */
            di = (di < 0) ? 
                (di + NBINS_V1) : 
                ((di >= NBINS_V1) ? (di - NBINS_V1) : (di));
            
            double th = ((M_PI2 * di) / NBINS_V1) - M_PI;
            ang[nangles++] = th;
            if(nangles >= 2) break;
        }
    }

    if( nangles < 1 ) return;

    // ext->xpos             = x;
    // ext->ypos             = y;
    // ext->sigma            = sig;
    ext->angle_from_bemap = ang[0];

    for( int bin=1; bin<nangles; bin++ ) {
        int idx = atomicAdd( &mgmt->counter, 1 );
        if( idx < mgmt->max2 ) {
            ext = &extremum[idx];
            ext->xpos             = x;
            ext->ypos             = y;
            ext->sigma            = sig;
            ext->angle_from_bemap = ang[bin];
        }
    }
}

/*************************************************************
 * V4: host side
 *************************************************************/
__host__
void Pyramid::orientation_v1( )
{
    _keep_time_orient_v1.start();
    for( int octave=0; octave<_octaves; octave++ ) {
        _layers[octave].readExtremaCount( _stream );
        hipStreamSynchronize( _stream );
        for( int level=1; level<_levels-1; level++ ) {
            dim3 block;
            dim3 grid;
            // grid.x  = _layers[octave].getExtremaMgmtH(level)->max1;
            grid.x  = _layers[octave].getExtremaMgmtH(level)->counter;
            block.x = ORI_V1_NUM_THREADS;
            if( grid.x != 0 ) {
                compute_keypoint_orientations_v1
                    <<<grid,block,0,_stream>>>
                    ( _layers[octave].getExtrema( level ),
                      _layers[octave].getExtremaMgmtD( ),
                      level,
                      _layers[octave].getData( level ),
                      _layers[octave].getPitch(),
                      _layers[octave].getHeight() );
            }
        }
    }
    _keep_time_orient_v1.stop();
}

