#include "hip/hip_runtime.h"
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "s_gradiant.h"
#include "debug_macros.h"

#include <stdio.h>
#include <inttypes.h>

using namespace popart;

/*************************************************************
 * V1: device side
 *************************************************************/

__device__
inline float compute_angle( int bin, float hc, float hn, float hp )
{
    /* interpolate */
    float di = bin + 0.5f * (hn - hp) / (hc+hc-hn-hp);

    /* clamp */
    di = (di < 0) ? 
            (di + ORI_NBINS) : 
            ((di >= ORI_NBINS) ? (di - ORI_NBINS) : (di));

    float th = ((M_PI2 * di) / ORI_NBINS) - M_PI;
    // float th = ((M_PI2 * di) / ORI_NBINS);
    return th;
}

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 */
__global__
void compute_keypoint_orientations_v1( Extremum*     extremum,
                                       int*          extrema_counter,
                                       Plane2D_float layer )
{
    uint32_t w   = layer.getWidth();
    uint32_t h   = layer.getHeight();

    // if( threadIdx.y >= mgmt->getCounter() ) return;

    Extremum* ext = &extremum[blockIdx.x];

    float hist[ORI_NBINS];
    for (int i = 0; i < ORI_NBINS; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = ORI_WINFACTOR * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = -0.5f / (sigw * sigw);
    int sq_thres  = rad * rad;
    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * exp(sq_dist * factor);

            // int bidx = (int)rintf(ORI_NBINS * (theta + M_PI) / M_PI2);
            int bidx = (int)roundf(ORI_NBINS * (theta + M_PI) / M_PI2);
            // int bidx = (int)roundf(ORI_NBINS * (theta + M_PI) / M_PI2 - 0.5f);

            if( bidx > ORI_NBINS ) {
                printf("Crashing: bin %d theta %f :-)\n", bidx, theta);
            }

            bidx = (bidx == ORI_NBINS) ? 0 : bidx;

            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < ORI_NBINS; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }


#define OLD_ORIENTATION

        if(threadIdx.x != 0) return;

    // for (int bin = 0; bin < ORI_NBINS; bin++) {
        // printf( "%f %f %d %f\n", x, y, bin, hist[bin] );
    // }

#ifdef OLD_ORIENTATION
        for (int iter = 0; iter < 2; iter++) {
            float first = hist[0];
            float prev = hist[(ORI_NBINS - 1)];

            int bin;
            //0,35
            for (bin = 0; bin < ORI_NBINS - 1; bin++) {
                float temp = hist[bin];
                hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * hist[bin + 1];
                prev = temp;
            }

            hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * first;
            //z vprintf("val: %f, indx: %d\n", hist[bin], bin);
        }
	
        /* find histogram maximum */
        float maxh = NINF;
        int binh = 0;
        for (int bin = 0; bin < ORI_NBINS; bin++) {
            // maxh = fmaxf(maxh, hist[bin]);
            if (hist[bin] > maxh) {
                maxh = hist[bin];
                binh = bin;
            }
        }

        {
            float hc = hist[binh];
            float hn = hist[((binh + 1 + ORI_NBINS) % ORI_NBINS)];
            float hp = hist[((binh - 1 + ORI_NBINS) % ORI_NBINS)];
            float th = compute_angle(binh, hc, hn, hp);

            if( isnan(th) ) {
                printf("NAN value in compute_angle\n");
            }

            ext->orientation = th;
        }

        /* find other peaks, boundary of 80% of max */
        int nangles = 1;

        for (int numloops = 1; numloops < ORI_NBINS; numloops++) {
            int bin = (binh + numloops) % ORI_NBINS;

            float hc = hist[bin];
            float hn = hist[((bin + 1 + ORI_NBINS) % ORI_NBINS)];
            float hp = hist[((bin - 1 + ORI_NBINS) % ORI_NBINS)];

            /* find if a peak */
            if (hc >= (0.8f * maxh) && hc > hn && hc > hp) {
                int idx = atomicAdd( extrema_counter, 1 );
                if( idx >= d_max_orientations ) break;

                float th = compute_angle(bin, hc, hn, hp);

                ext = &extremum[idx];
                ext->xpos = x;
                ext->ypos = y;
                ext->sigma = sig;
                ext->orientation = th;

                nangles++;
                if (nangles > 2) break;
            }
        }
#else // not OLD_ORIENTATION
        float xcoord[ORI_NBINS];
        float yval[ORI_NBINS];

        int   maxbin = 0;
        float y_max = 0;
        for(int bin = 0; bin < ORI_NBINS; bin++) {
            int prev = bin - 1;
            if( prev < 0 ) prev = ORI_NBINS - 1;
            int next = bin + 1;
            if( next == ORI_NBINS ) next = 0;

            if( hist[bin] > max( hist[prev], hist[next] ) ) {
                const float num = 3.0f * hist[prev] - 4.0f * hist[bin] + hist[next];
                const float denB = 2.0f * ( hist[prev] - 2.0f * hist[bin] + hist[next] );
                float newbin = num / denB; // * M_PI/18.0f; // * 10.0f;
                if( newbin >= 0 && newbin <= 2 ) {
                    xcoord[bin] = prev + newbin;
                    yval[bin]   = -(num*num) / (4.0f * denB) + hist[prev];

                    if( yval[bin] > y_max ) {
                        y_max = yval[bin];
                        maxbin = bin;
                    }
                }
            }
        }
        float th = ((M_PI2 * xcoord[maxbin]) / ORI_NBINS) - M_PI;

        ext->orientation = th;
#endif // not OLD_ORIENTATION
}

/*************************************************************
 * V4: host side
 *************************************************************/
#ifdef USE_DYNAMIC_PARALLELISM // defined in_s_pyramid.h

__global__
void orientation_starter_v1( Extremum*     extremum,
                             int*          extrema_counter,
                             Plane2D_float layer )
{
    dim3 block;
    dim3 grid;
    grid.x  = *extrema_counter;
    block.x = ORI_V1_NUM_THREADS;

    if( grid.x != 0 ) {
        compute_keypoint_orientations_v1
            <<<grid,block>>>
            ( extremum,
              extrema_counter,
              layer );
    }
}

__host__
void Pyramid::orientation_v1( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level);

            int* extrema_counters = oct_obj.getExtremaMgmtD( );
            int* extrema_counter  = &extrema_counters[level];
            orientation_starter_v1
                <<<1,1,0,oct_str>>>
                ( oct_obj.getExtrema( level ),
                  extrema_counter,
                  oct_obj.getData( level ) );
        }
    }
}

#else // not USE_DYNAMIC_PARALLELISM

__global__
void orientation_starter_v1( Extremum*,
                             ExtremaMgmt*,
                             uint32_t,
                             Plane2D_float )
{
    /* dummy to make the linker happy */
}

__host__
void Pyramid::orientation_v1( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=1; level<_levels-2; level++ ) {
            hipStreamSynchronize( oct_obj.getStream(level) );
        }

        oct_obj.readExtremaCount( );
        hipDeviceSynchronize( );

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level);

            dim3 block;
            dim3 grid;
            // grid.x  = _octaves[octave].getExtremaMgmtH(level)->max1;
            grid.x  = oct_obj.getExtremaMgmtH(level)->getCounter();
            block.x = ORI_V1_NUM_THREADS;
            if( grid.x != 0 ) {
                compute_keypoint_orientations_v1
                    <<<grid,block,0,oct_str>>>
                    ( oct_obj.getExtrema( level ),
                      oct_obj.getExtremaMgmtD( ),
                      level,
                      oct_obj.getData( level ) );
            }
        }
    }
}
#endif // not USE_DYNAMIC_PARALLELISM

