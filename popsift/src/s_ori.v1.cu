#include "hip/hip_runtime.h"
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "s_gradiant.h"
#include "debug_macros.h"

#include <math.h>
#include <stdio.h>
#include <inttypes.h>

using namespace popart;

/*************************************************************
 * V1: device side
 *************************************************************/

__device__
inline float compute_angle( int bin, float hc, float hn, float hp )
{
    /* interpolate */
    float di = bin + 0.5f * (hn - hp) / (hc+hc-hn-hp);

    /* clamp */
    di = (di < 0) ? 
            (di + ORI_NBINS) : 
            ((di >= ORI_NBINS) ? (di - ORI_NBINS) : (di));

    float th = __fdividef( M_PI2 * di, ORI_NBINS ) - M_PI;
    // float th = ((M_PI2 * di) / ORI_NBINS);
    return th;
}

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 */
__global__
void compute_keypoint_orientations_v1( Extremum*     extremum,
                                       int*          extrema_counter,
                                       Plane2D_float layer )
{
    uint32_t w   = layer.getWidth();
    uint32_t h   = layer.getHeight();

    // if( threadIdx.y >= mgmt->getCounter() ) return;

    Extremum* ext = &extremum[blockIdx.x];

    float hist[ORI_NBINS];
    for (int i = 0; i < ORI_NBINS; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = ORI_WINFACTOR * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = __fdividef( -0.5f, (sigw * sigw) );
    int sq_thres = rad * rad;
    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * expf(sq_dist * factor);

            int bidx = (int)rintf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );
            // int bidx = (int)roundf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );

            if( bidx > ORI_NBINS ) {
                printf("Crashing: bin %d theta %f :-)\n", bidx, theta);
            }

            bidx = (bidx == ORI_NBINS) ? 0 : bidx;

            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < ORI_NBINS; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }


    if(threadIdx.x != 0) return;

    for( int iter = 0; iter < 2; iter++ ) {
        float first = hist[0];
        float prev = hist[(ORI_NBINS - 1)];

        int bin;
        //0,35
        for( bin = 0; bin < ORI_NBINS - 1; bin++ ) {
            float temp = hist[bin];
            hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * hist[bin + 1];
            prev = temp;
        }

        hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * first;
        //z vprintf("val: %f, indx: %d\n", hist[bin], bin);
    }
	
    /* find histogram maximum */
    float maxh = NINF;
    int binh = 0;
    for (int bin = 0; bin < ORI_NBINS; bin++) {
        // maxh = fmaxf(maxh, hist[bin]);
        if (hist[bin] > maxh) {
            maxh = hist[bin];
            binh = bin;
        }
    }

    {
        float hc = hist[binh];
        float hn = hist[((binh + 1 + ORI_NBINS) % ORI_NBINS)];
        float hp = hist[((binh - 1 + ORI_NBINS) % ORI_NBINS)];
        float th = compute_angle(binh, hc, hn, hp);

#ifdef DEBUG_SEARCH_FOR_NANS
        if( isnan(th) ) {
            ext->invalid |= ANGLE_IS_NAN;
        }
#endif // DEBUG_SEARCH_FOR_NANS

        ext->orientation = th;
    }

    /* find other peaks, boundary of 80% of max */
    int nangles = 1;

    for (int numloops = 1; numloops < ORI_NBINS; numloops++) {
        int bin = (binh + numloops) % ORI_NBINS;

        float hc = hist[bin];
        float hn = hist[((bin + 1 + ORI_NBINS) % ORI_NBINS)];
        float hp = hist[((bin - 1 + ORI_NBINS) % ORI_NBINS)];

        /* find if a peak */
        if (hc >= (0.8f * maxh) && hc > hn && hc > hp) {
            int idx = atomicAdd( extrema_counter, 1 );
            if( idx >= d_max_orientations ) break;

            float th = compute_angle(bin, hc, hn, hp);

            ext = &extremum[idx];
            ext->xpos = x;
            ext->ypos = y;
            ext->sigma = sig;
            ext->orientation = th;
#ifdef DEBUG_SEARCH_FOR_NANS
            ext->invalid = 0;
#endif // DEBUG_SEARCH_FOR_NANS

            nangles++;
            if (nangles > 2) break;
        }
    }
}

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 * direct curve fitting approach
 */
__global__
void compute_keypoint_orientations_v2( Extremum*     extremum,
                                       int*          extrema_counter,
                                       Plane2D_float layer,
                                       int*          d_number_of_blocks,
                                       int           number_of_blocks )
{
#ifdef DEBUG_SEARCH_FOR_NANS
    int debug_invalid = 0;
#endif // DEBUG_SEARCH_FOR_NANS

    uint32_t w   = layer.getWidth();
    uint32_t h   = layer.getHeight();

    // if( threadIdx.y >= mgmt->getCounter() ) return;

    Extremum* ext = &extremum[blockIdx.x];

    float hist[ORI_NBINS];
    for (int i = 0; i < ORI_NBINS; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = ORI_WINFACTOR * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = __fdividef( -0.5f, (sigw * sigw) );
    int sq_thres  = rad * rad;
    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * expf(sq_dist * factor);

            int bidx = (int)rintf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );
            // int bidx = (int)roundf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );

            if( bidx > ORI_NBINS ) {
                printf("Crashing: bin %d theta %f :-)\n", bidx, theta);
            }

            bidx = (bidx == ORI_NBINS) ? 0 : bidx;

            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < ORI_NBINS; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }


    if(threadIdx.x != 0) return;

    float xcoord[ORI_NBINS];
    float yval[ORI_NBINS];

    int   maxbin[3];
    float y_max[3];

    #pragma unroll
    for( int i=0; i<3; i++ ) {
        maxbin[i] = 0;
        y_max[i] = -INFINITY;
    }

    for(int bin = 0; bin < ORI_NBINS; bin++) {
        int prev = bin - 1;
        if( prev < 0 ) prev = ORI_NBINS - 1;
        int next = bin + 1;
        if( next == ORI_NBINS ) next = 0;

        if( hist[bin] > max( hist[prev], hist[next] ) ) {
            const float num = 3.0f * hist[prev] - 4.0f * hist[bin] + hist[next];
            const float denB = 2.0f * ( hist[prev] - 2.0f * hist[bin] + hist[next] );
#ifdef DEBUG_SEARCH_FOR_NANS
            if( denB == 0 ) debug_invalid = ZERO_HISTOGRAM;
#endif // DEBUG_SEARCH_FOR_NANS

            float newbin = __fdividef( num, denB ); // * M_PI/18.0f; // * 10.0f;
            if( newbin >= 0 && newbin <= 2 ) {
                xcoord[bin] = prev + newbin;
                yval[bin]   = -(num*num) / (4.0f * denB) + hist[prev];

                if( yval[bin] > y_max[0] ) {
                    y_max[2]  = y_max[1];
                    y_max[1]  = y_max[0];
                    y_max[0]  = yval[bin];
                    maxbin[2] = maxbin[1];
                    maxbin[1] = maxbin[0];
                    maxbin[0] = bin;
                }
            }
        }
    }

    float th = __fdividef(M_PI2 * xcoord[maxbin[0]], ORI_NBINS) - M_PI;

    ext->orientation = th;
#ifdef DEBUG_SEARCH_FOR_NANS
    ext->invalid     = debug_invalid;
#endif // DEBUG_SEARCH_FOR_NANS

    for( int i=1; i<=2; i++ ) {
        if( y_max[i] < -1000.0f ) break; // this is a random number: no orientation can be this small

        if( y_max[i] < 0.8f * y_max[0] ) break;

        int idx = atomicAdd( extrema_counter, 1 );
        if( idx >= d_max_orientations ) break;

        float th = __fdividef(M_PI2 * xcoord[maxbin[i]], ORI_NBINS) - M_PI;

        ext = &extremum[idx];
        ext->xpos = x;
        ext->ypos = y;
        ext->sigma = sig;
        ext->orientation = th;
#ifdef DEBUG_SEARCH_FOR_NANS
        ext->invalid = debug_invalid;
#endif // DEBUG_SEARCH_FOR_NANS
    }

    __syncthreads();

    if( threadIdx.x == 0 && threadIdx.y == 0 ) {
        int ct = atomicAdd( d_number_of_blocks, 1 );
        if( ct >= number_of_blocks-1 ) {
            int num_ext = atomicMin( extrema_counter, d_max_orientations );
        }
    }

}

/*************************************************************
 * V4: host side
 *************************************************************/
#ifdef USE_DYNAMIC_PARALLELISM // defined in_s_pyramid.h

__global__
void orientation_starter_v1( Extremum*     extremum,
                             int*          extrema_counter,
                             Plane2D_float layer )
{
    dim3 block;
    dim3 grid;
    grid.x  = *extrema_counter;
    block.x = ORI_V1_NUM_THREADS;

    if( grid.x != 0 ) {
        compute_keypoint_orientations_v1
            <<<grid,block>>>
            ( extremum,
              extrema_counter,
              layer );
    }
}

__global__
void orientation_starter_v2( Extremum*     extremum,
                             int*          extrema_counter,
                             Plane2D_float layer,
                             int*          d_number_of_blocks )
{
    dim3 block;
    dim3 grid;
    grid.x  = *extrema_counter;
    block.x = ORI_V1_NUM_THREADS;

    if( grid.x != 0 ) {
        compute_keypoint_orientations_v2
            <<<grid,block>>>
            ( extremum,
              extrema_counter,
              layer,
              d_number_of_blocks,
              grid.x * grid.y );
    }
}

__host__
void Pyramid::orientation_v1( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        int*  orientation_num_blocks = oct_obj.getNumberOfOriBlocks( );

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level+2);

            int* extrema_counters = oct_obj.getExtremaMgmtD( );
            int* extrema_counter  = &extrema_counters[level];
            if( _bemap_orientation_mode ) {
                orientation_starter_v1
                    <<<1,1,0,oct_str>>>
                    ( oct_obj.getExtrema( level ),
                      extrema_counter,
                      oct_obj.getData( level ) );
            }  else {
                int*  num_blocks = &orientation_num_blocks[level];

                orientation_starter_v2
                    <<<1,1,0,oct_str>>>
                    ( oct_obj.getExtrema( level ),
                      extrema_counter,
                      oct_obj.getData( level ),
                      num_blocks );
            }
        }
    }
}

#else // not USE_DYNAMIC_PARALLELISM

__global__
void orientation_starter_v1( Extremum*,
                             int*,
                             Plane2D_float,
                             int* )
{
    /* dummy to make the linker happy */
}

__host__
void Pyramid::orientation_v1( )
{
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=3; level<_levels; level++ ) {
            hipStreamSynchronize( oct_obj.getStream(level) );
        }

        oct_obj.readExtremaCount( );
        hipDeviceSynchronize( );

        int* h_num_extrema = oct_obj.getExtremaMgmtH();
        int* d_num_extrema = oct_obj.getExtremaMgmtD();
        int* orientation_num_blocks = oct_obj.getNumberOfOriBlocks( );

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level+2);

            dim3 block;
            dim3 grid;
            grid.x  = h_num_extrema[level];
            block.x = ORI_V1_NUM_THREADS;
            if( grid.x != 0 ) {
                if( _bemap_orientation_mode ) {
                    compute_keypoint_orientations_v1
                        <<<grid,block,0,oct_str>>>
                        ( oct_obj.getExtrema( level ),
                          &d_num_extrema[level],
                          oct_obj.getData( level ) );
                } else {
                    compute_keypoint_orientations_v2
                        <<<grid,block,0,oct_str>>>
                        ( oct_obj.getExtrema( level ),
                          &d_num_extrema[level],
                          oct_obj.getData( level ),
                          &orientation_num_blocks[level],
                          grid.x * grid.y );
                }
            }
        }
    }
}
#endif // not USE_DYNAMIC_PARALLELISM

