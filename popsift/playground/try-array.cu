#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__
void testPrintArray( hipTextureObject_t obj, int width, int height, int levels )
{
    for( int z=0; z<levels; z++ ) {
        for( int y=0; y<height; y++ ) {
            for( int x=0; x<width; x++ ) {
                float v = tex2DLayered<float>( obj, x, y, z );
                int d = v;
                printf( "%d ", d );
            }
            printf("\n");
        }
        printf("\n");
    }
}

int main( )
{
    int width  = 10;
    int height = 4;
    int levels = 2;

    hipArray_t           _dog_3d;
    hipChannelFormatDesc _dog_3d_desc;
    hipExtent            _dog_3d_ext;
    hipTextureObject_t   _dog_3d_tex;

    _dog_3d_desc.f = hipChannelFormatKindFloat;
    _dog_3d_desc.x = 32;
    _dog_3d_desc.y = 0;
    _dog_3d_desc.z = 0;
    _dog_3d_desc.w = 0;

    _dog_3d_ext.width  = width; // for cudaMalloc3DArray, width in elements
    _dog_3d_ext.height = height;
    _dog_3d_ext.depth  = levels;

    hipError_t err;
    err = hipMalloc3DArray( &_dog_3d,
                             &_dog_3d_desc,
                             _dog_3d_ext,
                             hipArrayLayered | hipArraySurfaceLoadStore );
    if( err != hipSuccess ) {
        cerr << "CUDA malloc 3D array failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA malloc 3D array worked" << endl;
    }

    hipResourceDesc dog_res_desc;
    dog_res_desc.resType         = hipResourceTypeArray;
    dog_res_desc.res.array.array = _dog_3d;

    hipTextureDesc      dog_tex_desc;
    memset( &dog_tex_desc, 0, sizeof(hipTextureDesc) );
    dog_tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    dog_tex_desc.addressMode[0]   = hipAddressModeClamp;
    dog_tex_desc.addressMode[1]   = hipAddressModeClamp;
    dog_tex_desc.addressMode[2]   = hipAddressModeClamp;
    dog_tex_desc.readMode         = hipReadModeElementType; // read as float
    dog_tex_desc.filterMode       = hipFilterModePoint; // no interpolation

    err = hipCreateTextureObject( &_dog_3d_tex,
                                   &dog_res_desc,
                                   &dog_tex_desc, 0 );
    if( err != hipSuccess ) {
        cerr << "CUDA create texture failed: " << hipGetErrorString(err) << endl;
    } else {
        cerr << "CUDA create texture succeeded" << endl;
    }

    float* array;
    err = hipHostMalloc( &array, 2*4*10*sizeof(float) , hipHostMallocDefault);
    if( err != hipSuccess ) {
        cerr << "CUDA malloc host failed: " << hipGetErrorString(err) << endl;
    } else {
        cerr << "CUDA malloc host succeeded" << endl;
    }

    for( int z=0; z<2; z++ ) {
        for( int y=0; y<4; y++ ) {
            for( int x=0; x<10; x++ ) {
                array[z*4*10 + y*10 + x] = x + y + z;
                cerr << x+y+z << " ";
            }
            cerr << endl;
        }
        cerr << endl;
    }

#if 0
    err = cudaMemcpyToArray( _dog_3d,
                             0, 0,
                             array,
                             10*4*2*sizeof(float),
                             // 10*4*sizeof(float),
                             cudaMemcpyHostToDevice );
#else
    hipMemcpy3DParms s = { 0 };
    s.srcPtr = make_hipPitchedPtr( array, 10*sizeof(float), 10, 4 );
    s.dstArray = _dog_3d;
    s.extent = make_hipExtent( width, height, levels );
    s.kind = hipMemcpyHostToDevice;
    err = hipMemcpy3D( &s );
#endif
    if( err != hipSuccess ) {
        cerr << "CUDA memcpy 3D failed: " << hipGetErrorString(err) << endl;
    } else {
        cerr << "CUDA memcpy 3D succeeded" << endl;
    }

    testPrintArray
        <<<1,1>>>
        ( _dog_3d_tex, width, height, levels );

    err = hipHostFree( array );
    if( err != hipSuccess ) {
        cerr << "CUDA free host failed" << endl;
    } else {
        cerr << "CUDA free host success" << endl;
    }

    err = hipDestroyTextureObject( _dog_3d_tex );
    if( err != hipSuccess ) {
        cerr << "CUDA destroy texture failed" << endl;
    } else {
        cerr << "CUDA destroy texture success" << endl;
    }

    err = hipFreeArray( _dog_3d );
    if( err != hipSuccess ) {
        cerr << "CUDA array free failed" << endl;
    } else {
        cerr << "CUDA array free success" << endl;
    }
}

