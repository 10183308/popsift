
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>

#define POP_CUDA_FATAL(err,s) { \
        std::cerr << __FILE__ << ":" << __LINE__ << std::endl; \
        std::cerr << "    " << s << hipGetErrorString(err) << std::endl; \
        exit( -__LINE__ ); \
    }
#define POP_CUDA_FATAL_TEST(err,s) if( err != hipSuccess ) { POP_CUDA_FATAL(err,s); }


using namespace std;

#define ASIZE 3

__global__ void printme( hipTextureObject_t tex )
{
    printf("call printme\n");
    for( float i=0; i<1.0; i+=0.01 ) {
        float offset = i+1.0;
        float res = tex1D<float>( tex, offset );
        printf( "a[%f] = %f (%f)\n", i, res, (i-res) );
        i += 0.0001;
        offset = i+1.0;
        res = tex1D<float>( tex, offset );
        printf( "a[%f] = %f (%f)\n", i, res, (i-res) );
    }
}

int main( )
{
    float in_array[ASIZE];
    in_array[0] = -0.5;
    in_array[1] = 0.5;
    in_array[2] = 1.5;

    hipError_t err;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );
    // cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc( 8, 0, 0, 0, cudaChannelFormatKindUnsigned );
    hipArray_t array;

    err = hipMallocArray( &array, &channelDesc, ASIZE, 1 );
    POP_CUDA_FATAL_TEST( err, "hipMallocArray failed" );

    err = hipMemcpyToArray( array, 0, 0, in_array, (ASIZE)*sizeof(float), hipMemcpyHostToDevice );
    POP_CUDA_FATAL_TEST( err, "hipMemcpyToArray failed" );

    hipResourceDesc resDesc;
    memset( &resDesc, 0, sizeof(hipResourceDesc ) );
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    // resDesc.resType = cudaResourceTypeLinear;
    // resDesc.res.linear.devPtr      = array;
    // resDesc.res.linear.desc        = channelDesc;
    // resDesc.res.linear.sizeInBytes = 10000*sizeof(float);

    hipTextureDesc texDesc;
    memset( &texDesc, 0, sizeof(hipTextureDesc) );
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    // texDesc.filterMode       = cudaFilterModePoint;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    // texDesc.readMode         = cudaReadModeNormalizedFloat;
    texDesc.normalizedCoords = 0;

    hipTextureObject_t texObj = 0;

    err = hipCreateTextureObject ( &texObj, &resDesc, &texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "hipCreateTextureObject failed" );

    printme<<<1,1>>>( texObj );

    hipDeviceSynchronize();
}

