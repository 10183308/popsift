#include "hip/hip_runtime.h"
#include "write_plane_2d.h"

#include <iostream>
#include <iomanip>
#include <fstream>
#include <limits>

// #include "debug_macros.h"
// #include "align_macro.h"
// #include "assist.h"
// #include <stdio.h>
// #include <assert.h>

using namespace std;

namespace popart {

__host__
void write_plane2D( const char* filename, bool onDevice, Plane2D_float& f )
{
    if( onDevice ) {
        // cerr << __FILE__ << ":" << __LINE__ << ": copying from device" << endl;
        Plane2D_float g;
        g.allocHost( f.getCols(), f.getRows(), CudaAllocated );
        g.memcpyFromDevice( f );
        write_plane2D( filename, g );
        g.freeHost( CudaAllocated );
    } else {
        write_plane2D( filename, f );
    }
}

__host__
void write_plane2Dunscaled( const char* filename, bool onDevice, Plane2D_float& f )
{
    if( onDevice ) {
        // cerr << __FILE__ << ":" << __LINE__ << ": copying from device" << endl;
        Plane2D_float g;
        g.allocHost( f.getCols(), f.getRows(), CudaAllocated );
        g.memcpyFromDevice( f );
        write_plane2Dunscaled( filename, g );
        g.freeHost( CudaAllocated );
    } else {
        write_plane2Dunscaled( filename, f );
    }
}

__host__
void write_plane2D( const char* filename, Plane2D_float& f )
{
    // cerr << "Enter " << __FUNCTION__ << endl;

    int rows = f.getRows();
    int cols = f.getCols();
    // cerr << "    size: " << cols << "x" << rows << endl;

    unsigned char* c = new unsigned char[rows * cols];
    float minval = std::numeric_limits<float>::max();
    float maxval = std::numeric_limits<float>::min();
    for( int y=0; y<rows; y++ ) {
        for( int x=0; x<cols; x++ ) {
            float v = f.ptr(y)[x];
            // cerr << " " << v;
            minval = min( minval, v );
            maxval = max( maxval, v );
        }
    }
    // cerr << endl;

    // cerr << "    minval: " << minval << endl;
    // cerr << "    maxval: " << maxval << endl;

    float fmaxval = 255.0f / ( maxval - minval );
    for( int y=0; y<rows; y++ ) {
        for( int x=0; x<cols; x++ ) {
            float v = f.ptr(y)[x];
            v = ( v - minval ) * fmaxval;
            c[y*cols+x] = (unsigned char)v;
        }
    }
#if 1
    ofstream of( filename );
    of << "P2" << endl
       << cols << " " << rows << endl
       << "255" << endl;
    unsigned char* cx = c;
    for( int row=0; row<rows; row++ ) {
        for( int col=0; col<cols; col++ ) {
            int val = *cx;
            cx++;
            of << val << " ";
        }
        of << endl;
    }
    delete [] c;
#else
    ofstream of( filename );
    of << "P5" << endl
       << cols << " " << rows << endl
       << "255" << endl;
    of.write( (char*)c, cols * rows );
    delete [] c;
#endif

    // cerr << "Leave " << __FUNCTION__ << endl;
}

__host__
void write_plane2Dunscaled( const char* filename, Plane2D_float& f )
{
    int rows = f.getRows();
    int cols = f.getCols();

    float* c = new float[rows * cols];
    for( int y=0; y<rows; y++ ) {
        for( int x=0; x<cols; x++ ) {
            float v = f.ptr(y)[x];
            c[y*cols+x] = v;
        }
    }

    ofstream of( filename );
    of << "nonsense" << endl
       << cols << " " << rows << endl
       << "maxint" << endl;
    float* cx = c;
    for( int row=0; row<rows; row++ ) {
        for( int col=0; col<cols; col++ ) {
            float val = *cx;
            cx++;
            of << setprecision(2) << val << " ";
        }
        of << endl;
    }
    delete [] c;

    // cerr << "Leave " << __FUNCTION__ << endl;
}

} // namespace popart

