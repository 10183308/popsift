#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <iomanip>
#include <iostream>
#include <unistd.h>
#ifndef __APPLE__
#include <malloc.h>
#endif
#include <stdlib.h>
#include <errno.h>
#include <hip/hip_math_constants.h>
#include "features.h"
#include "sift_extremum.h"
#include "common/debug_macros.h"
#include "sift_conf.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

using namespace std;

namespace popsift {
    

/*************************************************************
 * Features
 *************************************************************/

    Features::Features( )
	: _num_ext( 0 )
	, _num_ori( 0 )
    { }
    
    Features::~Features( )
    { }
    
/*************************************************************
 * HostFeatures
 *************************************************************/
    
    HostFeatures::HostFeatures( )
	: _ext( 0 )
	, _ori( 0 )
    { }

    HostFeatures::HostFeatures( int num_ext, int num_ori )
	: _ext( 0 )
	, _ori( 0 )
    {
	reset( num_ext, num_ori );
    }

    HostFeatures::~HostFeatures( )
    {
	free( _ext );
	free( _ori );
    }

#ifdef __APPLE__
    static void* memalign( size_t alignment, size_t size )
    {
	void* ret;
	int err = posix_memalign( &ret, alignment, size );
	if( err != 0 ) {
	    errno = err;
	    ret = 0;
	}
	return ret;
    }
#endif

    void HostFeatures::reset( int num_ext, int num_ori )
    {
	if( _ext != 0 ) { free( _ext ); _ext = 0; }
	if( _ori != 0 ) { free( _ori ); _ori = 0; }

	_ext = (Feature*)memalign( sysconf(_SC_PAGESIZE), num_ext * sizeof(Feature) );
	if( _ext == 0 ) {
	    cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
		 << "    Failed to (re)allocate memory for downloading " << num_ext << " features" << endl;
	    if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
	    if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
	    exit( -1 );
	}
	_ori = (Descriptor*)memalign( sysconf(_SC_PAGESIZE), num_ori * sizeof(Descriptor) );
	if( _ori == 0 ) {
	    cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
		 << "    Failed to (re)allocate memory for downloading " << num_ori << " descriptors" << endl;
	    if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
	    if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
	    exit( -1 );
	}

	setFeatureCount( num_ext );
	setDescriptorCount( num_ori );
    }

    void HostFeatures::pin( )
    {
	hipError_t err;
	err = hipHostRegister( _ext, getFeatureCount() * sizeof(Feature), 0 );
	if( err != hipSuccess ) {
	    cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
		 << "    Failed to register feature memory in CUDA." << endl
		 << "    " << hipGetErrorString(err) << endl;
	}
	err = hipHostRegister( _ori, getDescriptorCount() * sizeof(Descriptor), 0 );
	if( err != hipSuccess ) {
	    cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
		 << "    Failed to register descriptor memory in CUDA." << endl
		 << "    " << hipGetErrorString(err) << endl;
	}
    }

    void HostFeatures::unpin( )
    {
	hipHostUnregister( _ext );
	hipHostUnregister( _ori );
    }

    void HostFeatures::print( std::ostream& ostr, bool write_as_uchar ) const
    {
	for( int i=0; i<size(); i++ ) {
	    _ext[i].print( ostr, write_as_uchar );
	}
    }

    std::ostream& operator<<( std::ostream& ostr, const HostFeatures& feature )
    {
	feature.print( ostr, false );
	return ostr;
    }

/*************************************************************
 * DeviceFeatures
 *************************************************************/

    DeviceFeatures::DeviceFeatures( )
	: _ext( 0 )
	, _ori( 0 )
	, _rev( 0 )
    { }

    DeviceFeatures::DeviceFeatures( int num_ext, int num_ori )
	: _ext( 0 )
	, _ori( 0 )
	, _rev( 0 )
    {
	reset( num_ext, num_ori );
    }

    DeviceFeatures::~DeviceFeatures( )
    {
	hipFree( _ext );
	hipFree( _ori );
	hipFree( _rev );
    }

    void DeviceFeatures::reset( int num_ext, int num_ori )
    {
	if( _ext != 0 ) { hipFree( _ext ); _ext = 0; }
	if( _ori != 0 ) { hipFree( _ori ); _ori = 0; }
	if( _rev != 0 ) { hipFree( _rev ); _rev = 0; }

	_ext = popsift::cuda::malloc_devT<Feature>   ( num_ext, __FILE__, __LINE__ );
	_ori = popsift::cuda::malloc_devT<Descriptor>( num_ori, __FILE__, __LINE__ );
	_rev = popsift::cuda::malloc_devT<int>       ( num_ori, __FILE__, __LINE__ );

	setFeatureCount( num_ext );
	setDescriptorCount( num_ori );
    }

    __device__ inline float
    l2_in_t0( const float4* lptr, const float4* rptr )
    {
	const float4  lval = lptr[threadIdx.x];
	const float4  rval = rptr[threadIdx.x];
	const float4  mval = make_float4( lval.x - rval.x,
					  lval.y - rval.y,
					  lval.z - rval.z,
					  lval.w - rval.w );
	float   res = mval.x * mval.x
	    + mval.y * mval.y
	    + mval.z * mval.z
	    + mval.w * mval.w;

	res += __shfl_down( res, 16 );
	res += __shfl_down( res,  8 );
	res += __shfl_down( res,  4 );
	res += __shfl_down( res,  2 );
	res += __shfl_down( res,  1 );

	return res;
    }
    __device__ inline float
    dot_l2_in_t0( const float4* lptr, const float4* rptr )
    {
	const float4  lval = lptr[threadIdx.x];
	const float4  rval = rptr[threadIdx.x];
	const float4  mval = make_float4( lval.x * rval.x,
					  lval.y * rval.y,
					  lval.z * rval.z,
					  lval.w * rval.w );
	float   res = mval.x
	    + mval.y
	    + mval.z
	    + mval.w;

    
	res += __shfl_down( res, 16 );
	res += __shfl_down( res,  8 );
	res += __shfl_down( res,  4 );
	res += __shfl_down( res,  2 );
	res += __shfl_down( res,  1 );
	return res;
    }
  
    __global__ void
    compute_distance_l2( int3* match_matrix, Descriptor* l, int l_len, Descriptor* r, int r_len )
    {
	if( blockIdx.x >= l_len ) return;
	const int idx = blockIdx.x;

	float match_1st_val = HIP_INF_F;
	float match_2nd_val = HIP_INF_F;
	int   match_1st_idx = 0;
	int   match_2nd_idx = 0;

	const float4* lptr = (const float4*)( &l[idx] );

	for( int i=0; i<r_len; i++ )
	{
	    const float4* rptr = (const float4*)( &r[i] );

	    const float   res  = l2_in_t0( lptr, rptr );

	    if( threadIdx.x == 0 )
	    {
		if( res < match_1st_val )
		{
		    match_2nd_val = match_1st_val;
		    match_2nd_idx = match_1st_idx;
		    match_1st_val = res;
		    match_1st_idx = i;
		}
		else if( res < match_2nd_val )
		{
		    match_2nd_val = res;
		    match_2nd_idx = i;
		}
	    }
	    __syncthreads();
	}

	if( threadIdx.x == 0 )
	{
	    bool accept = ( match_1st_val / match_2nd_val < 0.8f );
	    match_matrix[blockIdx.x] = make_int3( match_1st_idx, match_2nd_idx, accept );
	}
    }
  
  
    __global__ void
    compute_distance_dot( int3* match_matrix, Descriptor* l, int l_len, Descriptor* r, int r_len )
    {
	if( blockIdx.x >= l_len ) return;
	const int idx = blockIdx.x;

	float match_1st_val = -1.0f;
	float match_2nd_val = -1.0f;
	int   match_1st_idx = 0;
	int   match_2nd_idx = 0;

  

	const float4* lptr = (const float4*)( &l[idx] );

	for( int i=0; i<r_len; i++ )
	{
	    const float4* rptr = (const float4*)( &r[i] );
	    const float   res  = dot_l2_in_t0( lptr, rptr );

	
	    if( threadIdx.x == 0 )
	    {
		if( res > match_1st_val )
		{
		    match_2nd_val = match_1st_val;
		    match_2nd_idx = match_1st_idx;
		    match_1st_val = res;
		    match_1st_idx = i;
		}
		else if( res > match_2nd_val )
		{
		    match_2nd_val = res;
		    match_2nd_idx = i;
		}
	    }

	    __syncthreads();	
	}
    
    
	const int one = __shfl(match_1st_idx, 0);
	const int two = __shfl(match_2nd_idx, 0);
  
	const float4* rptr = (const float4*)( &r[one] );
	const float res2 = l2_in_t0( lptr, rptr );
	const float4* rptr2 = (const float4*)( &r[two] );
	const float res3 = l2_in_t0( lptr, rptr2 );

	if( threadIdx.x == 0 )
	{
	    bool accept = (res2/res3 < 0.8f );
	    match_matrix[blockIdx.x] = make_int3( match_1st_idx, match_2nd_idx, accept );
	}
    }

    __device__ void
    printBits( unsigned int num )
    {
        for ( int bit = 0; bit < 32; bit++ )
	{
	    printf("%i", num & 0x01);
	    num = num >> 1;
	}
    }

  
    __device__ void
    printFeature( unsigned int *num )
    {
        for ( int i = 0; i < 128; i += 4 ) {
            for (int j = 0; j < 4; j++) {
		printBits(num[ i + j]);
		printf( " " );
            }
            
            printf( "\n" ); 
        }
        
	printf( "\n\n" );
    }

    __device__ void
    print32x32( unsigned int *num )
    {
        for ( int i = 0; i < 32; i++ ) {
            printBits(num[i]);
            printf( "\n" ); 
        }
        
        printf( "\n\n" );
    }
    
    __device__ void
    transpose32(unsigned int *A) {
        int j, k;
        unsigned m, t;
        
        m = 0x0000FFFF;
        for (j = 16; j != 0; j = j >> 1, m = m ^ (m << j)) {
            for (k = 0; k < 32; k = (k + j + 1) & ~j) {
                t = (A[k] ^ (A[k+j] >> j)) & m;
                A[k] = A[k] ^ t;
                A[k+j] = A[k+j] ^ (t << j);
            }
        }
    }
    
    __device__ void
    organize_32( unsigned int* A, unsigned int* B )
    {
        int i = threadIdx.x;
        int cnt = threadIdx.x * 4;
        for (int j = 0; j < 128; j +=32)
	{
	    B[cnt] = A[i + j];
	    cnt++;   
                
	}
    }
	
    // Using __shared__ variables within a descriptor
    __global__ void
    compute_distance_shared_32(unsigned int *featurex, unsigned int *featurey)
    {
//int i = blockIdx.x*blockDim.x + threadIdx.x;
	int s = threadIdx.x * 4;
	int i;
	__shared__ unsigned int T[128];
	for (i = s; i < s + 4; i++)
	    T[i] = *(featurex + i + 128 * blockIdx.x);
        
	__syncthreads();
	if (threadIdx.x < 4)
	    transpose32(T + 32 * threadIdx.x);
	__syncthreads();
	organize_32(T, featurey + 128 * blockIdx.x);
        
        
//__syncthreads();
//if (threadIdx.x == 0 && blockIdx.x == 32)
//printFeature(featurey + DIMENSIONS * blockIdx.x);
        
    }
    
  
    
    __device__ void
    transpose(Descriptor * src, Descriptor *des, int size) {
              
        int idx = blockIdx.x;

        
	
        unsigned int * featurex = (unsigned int*)(&src[idx]);       
	//const float4* fx = (const float4*)(&src[idx]);
	//const float4 fx1 = fx[0];
	//unsigned int * featurex = (unsigned int*)&fx1.x;
	
        unsigned int * featurey = (unsigned int*)(&des[idx]);
        
        //int i = blockIdx.x*blockDim.x + threadIdx.x;
        int s = threadIdx.x * 4;
        int i;
        __shared__ unsigned int T[128];
	for (i = s; i < s + 4; i++)
            T[i] = *(featurex + i + 128 * idx);


//	    if(idx == 0 && threadIdx.x == 0) 
//		printFeature(featurex);
	    
        __syncthreads();        
        if (threadIdx.x < 4)
            transpose32(T + 32 * threadIdx.x);
        __syncthreads();
        organize_32(T, featurey + 128 * blockIdx.x);


//	    if(idx == 0 && threadIdx.x == 0)
//		printFeature(featurey+128*idx);

	__syncthreads();

	//if(idx == 0 && threadIdx.x == 0)
	//   printFeature((unsigned int*)des+128*idx);
	//const int N = 2;	
	//std::string s1((char*)featurey+128*idx, 16);
	//std::string s2((char*)featurex, 16);
	//int keys[N] = {1, 2};
	//char values[N] = {'b', 'a'};	    
	//thrust::stable_sort_by_key(thrust::device, keys, keys+N, values);
        
       
    }
    
    __global__ void
    compute_distance_transposed_hamming( int3* match_matrix, Descriptor* l, int l_len, Descriptor* r, int r_len , Descriptor * l_tra, Descriptor *r_tra) {

        if(blockIdx.x > l_len)
            return;

	if(blockIdx.x == 2 && threadIdx.x == 0)
	    printFeature((unsigned int*)l+128*blockIdx.x);
	
	
        transpose(l, l_tra, l_len);
	
	if(blockIdx.x == 2 && threadIdx.x == 0)
	printFeature((unsigned int*)l_tra+128*blockIdx.x);
	
	
    }

    __global__ void
    compute_distance_print( int3* match_matrix, Descriptor* l, int l_len, Descriptor* r, int r_len , Descriptor * l_tra, Descriptor *r_tra) {
	printf("address: %d\n", l_tra);


	for(int i = 0; i < 4; i++) {
	    for(int j = 0; j < 10; j++)
		printf("%u\t", l_tra[i].features[j]);
	    printf("\n");
	}
	/*
	unsigned int * t = (unsigned int*)(&l_tra[0]);
	printFeature((unsigned int *)t);
	t = (unsigned int*)(&l_tra[1]);
	printFeature((unsigned int *)t);
	t = (unsigned int*)(&l_tra[2]);
	printFeature((unsigned int *)t);
	t = (unsigned int*)(&l_tra[3]);
	printFeature((unsigned int *)t);
	t = (unsigned int*)(&l_tra[4]);
	printFeature((unsigned int *)t);
	t = (unsigned int*)(&l_tra[5]);
	printFeature((unsigned int *)t);
	t = (unsigned int*)(&l_tra[6]);
	printFeature((unsigned int *)t);
	*/
	printf("-------\n");
	//printFeature((unsigned int *)l_tra + 128);
    }

    struct compare_descriptors {
	template <typename T>
	__host__ __device__
	bool operator()(const T &l, const T &r) const {
	    for(int i = 0; i < 128; i++) {
		if(l.features[i] > r.features[i])
		    return true;
		if(l.features[i] < r.features[i])
		    return false;
	    }
	    return false;
	}
    };

    
    __global__ void
    show_distance( int3*       match_matrix,
		   Feature*    l_ext,
		   Descriptor* l_ori,
		   int*        l_fem,
		   int         l_len,
		   Feature*    r_ext,
		   Descriptor* r_ori,
		   int*        r_fem,
		   int         r_len )
    {
	int counter = 0;
	for( int i=0; i<l_len; i++ )
	{
	    const float4* lptr  = (const float4*)( &l_ori[i] );
	    const float4* rptr1 = (const float4*)( &r_ori[match_matrix[i].x] );
	    const float4* rptr2 = (const float4*)( &r_ori[match_matrix[i].y] );
	    float d1 = l2_in_t0( lptr, rptr1 );
	    float d2 = l2_in_t0( lptr, rptr2 );
	    if( threadIdx.x == 0 )
	    {
	  
		if( match_matrix[i].z )
		    counter++;
		/*printf( "accept feat %4d [%4d] matches feat %4d [%4d] ( 2nd feat %4d [%4d] ) dist %.3f vs %.3f\n",
		  l_fem[i], i,
		  r_fem[match_matrix[i].x], match_matrix[i].x,
		  r_fem[match_matrix[i].y], match_matrix[i].y,
		  d1, d2 );*/
	  
		//else
		/*printf( "reject feat %4d [%4d] matches feat %4d [%4d] ( 2nd feat %4d [%4d] ) dist %.3f vs %.3f\n",
		  l_fem[i], i,
		  r_fem[match_matrix[i].x], match_matrix[i].x,
		  r_fem[match_matrix[i].y], match_matrix[i].y,
		  d1, d2 );*/
	    }
	
	    __syncthreads();
      
	}
	if( threadIdx.x == 0 )
	    printf("Matches: %d\n", counter);
  
    }


    __host__ Descriptor * gpu_init(int SIZE) {
	Descriptor *tmp;

	hipError_t err = hipMalloc((void **)&tmp, SIZE * sizeof(Descriptor));
	if(err != hipSuccess)
	    printf("%s\n", hipGetErrorString(err));

	return tmp;
    }



    
    void DeviceFeatures::match( DeviceFeatures* other, const popsift::Config& config )
    {

	int l_len = getDescriptorCount( );
	int r_len = other->getDescriptorCount( );

	hipDeviceSetLimit(hipLimitPrintfFifoSize, 10000000);

	int3* match_matrix = popsift::cuda::malloc_devT<int3>( l_len, __FILE__, __LINE__ );    
    
	dim3 grid;
	grid.x = l_len;
	grid.y = 1;
	grid.z = 1;
	dim3 block;
	block.x = 32;
	block.y = 1;
	block.z = 1;

	if ( config.getModeMatching() == popsift::Config::l2 )
	{
	    compute_distance_l2
		<<<grid,block>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len );
	}
	else if ( config.getModeMatching() == popsift::Config::dot )
	{
	    compute_distance_dot
		<<<grid,block>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len );
	}
	else
	{


	    //transpose first set of descritors

	    //sort the transposed descriptors

	    //transpose and compare with the second set
	
	
	    Descriptor *l_copy = gpu_init(l_len);
	    Descriptor *r_copy = gpu_init(r_len);
/*
	    compute_distance_print
		<<<1, 1>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len , l_copy, r_copy);
	    hipDeviceSynchronize();*/
	    compute_distance_transposed_hamming
		<<<grid,block>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len , l_copy, r_copy);
	    
	    hipDeviceSynchronize();
	    // printf("%f\n", [0].features[0]);
	    
	    Descriptor *h_data;
	    Descriptor *d_data;
	    Descriptor *tmp_host = (Descriptor*)malloc(l_len * sizeof(Descriptor));
	    //h_data = l_copy;

	    hipMalloc(
		&d_data,
		l_len*sizeof(Descriptor));

	    hipDeviceSynchronize();
	    compute_distance_print
		<<<1, 1>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len , l_copy, r_copy);


	    /*
	    hipMemcpy(
		d_data,
	        l_copy,
		l_len*sizeof(Descriptor),
		hipMemcpyDeviceToDevice);

	    */
	    /*
	     hipDeviceSynchronize();
	    compute_distance_print
		<<<1, 1>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len , d_data, r_copy);
	    */
	    //thrust::device_ptr<Descriptor> d_ptr(d_data);
/*
	    thrust::device_ptr<Descriptor> d_ptr = thrust::device_pointer_cast(l_copy);
	    
	    
	    thrust::sort(
		d_ptr,
		d_ptr+l_len,
		compare_descriptors());
*/
	    hipDeviceSynchronize();
	    compute_distance_print
		<<<1, 1>>>
		( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len , l_copy, r_copy);
/*
	    hipMemcpy(
		tmp_host,
		d_data,
		l_len*sizeof(Descriptor),
		hipMemcpyDeviceToHost);
*/

  
	    //int tmp_size = l_len;
	    
	    // for(int i = 0; i < tmp_size; i++)
	    //	printf("%f %f\n", tmp_host[i].features[126], tmp_host[i].features[127]);
	
	}

	show_distance
	    <<<1,32>>>
	    ( match_matrix,
	      getFeatures(),
	      getDescriptors(),
	      getReverseMap(),
	      l_len,
	      other->getFeatures(),
	      other->getDescriptors(),
	      other->getReverseMap(),
	      r_len );


	hipFree( match_matrix );
    }

/*************************************************************
 * Feature
 *************************************************************/

    void Feature::print( std::ostream& ostr, bool write_as_uchar ) const
    {
	float sigval =  1.0f / ( sigma * sigma );

	for( int ori=0; ori<num_ori; ori++ ) {
	    ostr << xpos << " " << ypos << " "
		 << sigval << " 0 " << sigval << " ";
	    if( write_as_uchar ) {
		for( int i=0; i<128; i++ ) {
		    ostr << roundf(desc[ori]->features[i]) << " ";
		}
	    } else {
		ostr << std::setprecision(3);
		for( int i=0; i<128; i++ ) {
		    ostr << desc[ori]->features[i] << " ";
		}
		ostr << std::setprecision(6);
	    }
	    ostr << std::endl;
	}
    }

    std::ostream& operator<<( std::ostream& ostr, const Feature& feature )
    {
	feature.print( ostr, false );
	return ostr;
    }

} // namespace popsift
