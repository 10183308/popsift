#include <stdlib.h>
#include <iostream>
#include <hipblas.h>

#include "cublas_init.h"

void cublas_init( hipblasHandle_t* handle, const char* file, int line )
{
    hipblasStatus_t stat;
    stat = hipblasCreate( handle );
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS initialization filed in " << file << ":" << line << std::endl;
        exit( -1 );
    }
}

void cublas_uninit( hipblasHandle_t handle )
{
    hipblasDestroy( handle );
}

