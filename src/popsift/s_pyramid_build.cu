#include "hip/hip_runtime.h"
/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "gauss_filter.h"
#include "common/debug_macros.h"
#include "assist.h"
#include "common/clamp.h"

#include <iostream>
#include <stdio.h>

/* It makes no sense whatsoever to change this value */
#define PREV_LEVEL 3

#define MAKE_DOG_LATER

namespace popsift {

namespace gauss {

namespace variableSpan {

namespace absoluteTexAddress {
__global__
void horiz( hipTextureObject_t src_data,
            Plane2D_float       dst_data,
            int                 level )
{
    const int dst_w = dst_data.getWidth();

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = d_gauss.inc_span[level]; offset>0; offset-- ) {
        const float& g  = popsift::d_gauss.inc_filter[level*GAUSS_ALIGN + offset];
        const float  v1 = tex2D<float>( src_data, off_x - offset + 0.5f, blockIdx.y + 0.5f );
        out += ( v1 * g );

        const float  v2 = tex2D<float>( src_data, off_x + offset + 0.5f, blockIdx.y + 0.5f );
        out += ( v2 * g );
    }
    const float& g  = popsift::d_gauss.inc_filter[level*GAUSS_ALIGN];
    const float v3 = tex2D<float>( src_data, off_x+0.5f, blockIdx.y+0.5f );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out;
}

__device__
inline void vert( hipTextureObject_t src_data,
                  Plane2D_float       dst_data,
                  int                 span,
                  float*              filter )
{
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = span; offset>0; offset-- ) {
        g  = filter[offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );
    }

    g  = filter[0];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void vert( hipTextureObject_t src_data,
           Plane2D_float       dst_data,
           int                 level )
{
    vert( src_data, dst_data, d_gauss.inc_span[level], &popsift::d_gauss.inc_filter[level*GAUSS_ALIGN] );
}

} // namespace absoluteTexAddress

namespace relativeTexAddress {

__device__
inline void horiz( hipTextureObject_t src_data,
                   Plane2D_float       dst_data,
                   float               shift,
                   int                 span,
                   float*              filter )
{
    const float dst_w  = dst_data.getWidth();
    const float dst_h  = dst_data.getHeight();
    const float read_y = ( blockIdx.y + shift ) / dst_h;

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = span; offset>0; offset-- ) {
        const float& g  = filter[offset];
        const float read_x_l = ( off_x - offset );
        const float  v1 = tex2D<float>( src_data, ( read_x_l + shift ) / dst_w, read_y );
        out += ( v1 * g );

        const float read_x_r = ( off_x + offset );
        const float  v2 = tex2D<float>( src_data, ( read_x_r + shift ) / dst_w, read_y );
        out += ( v2 * g );
    }
    const float& g  = filter[0];
    const float read_x = off_x;
    const float v3 = tex2D<float>( src_data, ( read_x + shift ) / dst_w, read_y );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out * 255.0f;
}

__global__
void horiz( hipTextureObject_t src_data,
            Plane2D_float       dst_data,
            float               shift )
{
    horiz( src_data, dst_data, shift, d_gauss.inc_span[0], &d_gauss.inc_filter[0*GAUSS_ALIGN] );
}

} // namespace relativeTexAddress

} // namespace variableSpan

namespace fixedSpan {

template<int SPAN, bool isOctave0>
__global__
void octave( Plane2D_float src_data,
             Plane2D_float dst_data
#ifndef MAKE_DOG_LATER
             , hipSurfaceObject_t dog_data
#endif // MAKE_DOG_LATER
             )
{
    /* Idea: Process a 16x16 square of the src image and compute
     *       all levels except level 0 (must be computed before)
     *       blockIdx.x and blockIdx.y are the X and Y blocks.
     *       threadIdx.x runs 0 ... 16 + 2*(span-1)
     *       threadIdx.y runs 0 ... LEVELS-2
     */
    const int stride = 16 + 2 * (SPAN-1);
    const int noff   = SPAN - 1;
    const int level  = threadIdx.y + 1;

    __shared__ float inn_block[stride][stride];
    __shared__ float out_block[MAX_LEVELS][stride][stride];

    const int w = src_data.getWidth();
    const int h = src_data.getHeight();

    Plane2D_float destination( w, h,
                               dst_data.ptr( threadIdx.y * h ),
                               dst_data.getPitch() );

    for( int row=threadIdx.y; row<stride; row+=blockDim.y ) {
        const int idx = clamp( blockIdx.x * 16 + threadIdx.x - noff, w );
        const int idy = clamp( blockIdx.y * 16 + row         - noff, h );
        inn_block[row][threadIdx.x] = src_data.ptr(idy)[idx];
    }
    __syncthreads();

    float* filter = isOctave0
                  ? &d_gauss.abs_filter_o0[ level * GAUSS_ALIGN ]
                  : &d_gauss.abs_filter_oN[ level * GAUSS_ALIGN ];

    for( int row=0; row<stride; row++ ) {
        float in = inn_block[row][threadIdx.x];
        float out = in * filter[0];
#if 0
// do no filtering at all
        float g;
        if( SPAN==4 ) {
            g = __shfl_up( in, 1 ) + __shfl_down( in, 1, 22 ); out += g * filter[1];
            g = __shfl_up( in, 2 ) + __shfl_down( in, 2, 22 ); out += g * filter[2];
            g = __shfl_up( in, 3 ) + __shfl_down( in, 3, 22 ); out += g * filter[3];
        }
#endif

#if 0
// perhaps manual unrolling is safe --- stick with that until it works
        #pragma unroll
        for( int s=1; s<SPAN; s++ ) {
            float g = __shfl_up( in, s ) + __shfl_down( in, s );
            out += g * filter[s];
        }
#endif
        out_block[level][row][threadIdx.x] = out;
    }
    __syncthreads();

#if 0
    float out_col[stride];
    for( int col=0; col<stride; col++ ) {
        float in = out_block[level][threadIdx.x][col];
        float out = in * filter[0];

        #pragma unroll
        for( int s=1; s<SPAN; s++ ) {
            float g = __shfl_up( in, -s ) + __shfl_down( in, s );
            out += g * filter[s];
        }
        out_col[col] = out;
    }
    __syncthreads();

    for( int col=0; col<stride; col++ ) {
        out_block[level][threadIdx.x][col] = out_col[col];
    }
#endif

    for( int row=0; row<16; row++ ) {
        const int base_x = blockIdx.x * 16;
        const int base_y = blockIdx.y * 16;
        if( threadIdx.x < 16 ) {
            float val = out_block[level][row+noff][threadIdx.x+noff];
#ifndef MAKE_DOG_LATER
            float dog = ( level == 0
                        ? inn_block[row+noff][threadIdx.x+noff]
                        : out_block[level-1][row+noff][threadIdx.x+noff] )
                      - val;
#endif // MAKE_DOG_LATER
            if( base_y+row < h && base_x+threadIdx.x < w ) {
                int idx = base_x+threadIdx.x;
                int idy = base_y+row;

                destination.ptr(idy)[idx] = val;
                // destination.ptr(idy)[idx] = inn_block[row+noff][threadIdx.x+noff];
                // destination.ptr(idy)[idx] = src_data.ptr(idy)[idx];
                // destination.ptr(idy)[idx] = 0;
                // destination.ptr(idy)[idx] = val;
#ifndef MAKE_DOG_LATER
                surf2DLayeredwrite( dog,
                                    dog_data,
                                    idx*4,
                                    idy,
                                    level,
                                    hipBoundaryModeZero );
#endif // MAKE_DOG_LATER
            }
        }
    }
    __syncthreads();
}

} // namespace fixedSpan



__global__
void get_by_2_interpolate( hipTextureObject_t src_data,
                           Plane2D_float       dst_data,
                           int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const float val = tex2D<float>( src_data, 2.0f * idx + 1.0f, 2.0f * idy + 1.0f );
    dst_data.ptr(idy)[idx] = val;
}

__global__
void get_by_2_pick_every_second( Plane2D_float src_data,
                                 Plane2D_float dst_data,
                                 int           level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();
    const int read_x = clamp( idx << 1, 0, src_w );
    const int read_y = clamp( idy << 1, 0, src_h );

    const float val = src_data.ptr(read_y)[read_x];

    dst_data.ptr(idy)[idx] = val;
}


__global__
void make_dog( Plane2D_float       this_data,
               Plane2D_float       top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int cols = this_data.getWidth();
    const int rows = this_data.getHeight();
    
    const int r_x = clamp( idx, cols );
    const int r_y = clamp( idy, rows );

    const float b = this_data.ptr(r_y)[r_x];
    const float a = top_data .ptr(r_y)[r_x];
    const float c = b - a;

    surf2DLayeredwrite( c, dog_data, idx*4, idy, level, hipBoundaryModeZero );
}

} // namespace gauss

__host__
inline void Pyramid::horiz_from_input_image( const Config& conf, Image* base, int octave, hipStream_t stream, Config::SiftMode mode )
{
    Octave&   oct_obj = _octaves[octave];

    const int width   = oct_obj.getWidth();
    const int height  = oct_obj.getHeight();

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    float shift  = 0.5f;

    if( octave == 0 && ( mode == Config::PopSift || mode == Config::VLFeat ) ) {
        shift  = 0.5f * powf( 2.0f, conf.getUpscaleFactor() );
    }

    gauss::variableSpan::relativeTexAddress::horiz
        <<<grid,block,0,stream>>>
        ( base->getInputTexture(),
          oct_obj.getIntermediateData( ),
          shift );
}


__host__
inline void Pyramid::downscale_from_prev_octave( int octave, int level, hipStream_t stream, Config::SiftMode mode )
{
    Octave&      oct_obj = _octaves[octave];
    Octave& prev_oct_obj = _octaves[octave-1];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* Necessary to wait for a lower level in the previous octave */
    hipEvent_t ev = prev_oct_obj.getEventGaussDone( _levels-PREV_LEVEL );
    hipStreamWaitEvent( stream, ev, 0 );

    dim3 h_block( 64, 2 );
    dim3 h_grid;
    h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
    h_grid.y = (unsigned int)grid_divide( height, h_block.y );

    switch( mode )
    {
    case Config::PopSift :
    case Config::VLFeat :
    case Config::OpenCV :
        gauss::get_by_2_pick_every_second
            <<<h_grid,h_block,0,stream>>>
            ( prev_oct_obj.getData( _levels-PREV_LEVEL ),
              oct_obj.getData( level ),
              level );
        break;
    default :
        gauss::get_by_2_interpolate
            <<<h_grid,h_block,0,stream>>>
            ( prev_oct_obj._data_tex[ _levels-PREV_LEVEL ],
              oct_obj.getData( level ),
              level );
        break;
    }
}

__host__
inline void Pyramid::horiz_from_prev_level( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* waiting for previous level in same octave */
    hipEvent_t ev = oct_obj.getEventGaussDone( level-1 );
    hipStreamWaitEvent( stream, ev, 0 );

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    gauss::variableSpan::absoluteTexAddress::horiz
        <<<grid,block,0,stream>>>
        ( oct_obj._data_tex[ level-1 ],
          oct_obj.getIntermediateData( ),
          level );
}

__host__
inline void Pyramid::vert_from_interm( int octave, int level, hipStream_t stream )
{
    Octave& oct_obj = _octaves[octave];

    /* waiting for any events is not necessary, it's in the same stream as horiz
     */

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 64, 2 );
    dim3 grid;
    grid.x = (unsigned int)grid_divide( width,  block.x );
    grid.y = (unsigned int)grid_divide( height, block.y );

    gauss::variableSpan::absoluteTexAddress::vert
        <<<grid,block,0,stream>>>
        ( oct_obj._interm_data_tex,
          oct_obj.getData( level ),
          level );
}

__host__
inline void Pyramid::dog_from_blurred( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 128, 2 );
    dim3 grid;
    grid.x = grid_divide( width,  block.x );
    grid.y = grid_divide( height, block.y );

    /* waiting for lower level is automatic, it's in the same stream.
     * waiting for upper level is necessary, it's in another stream.
     */
    hipEvent_t  ev     = oct_obj.getEventGaussDone( level-1 );
    hipStreamWaitEvent( stream, ev, 0 );

    gauss::make_dog
        <<<grid,block,0,stream>>>
        ( oct_obj.getData(level),
          oct_obj.getData(level-1),
          oct_obj.getDogSurface( ),
          level-1 );
}

__host__
inline void Pyramid::make_octave( const Config& conf, Octave& oct_obj, hipStream_t stream, bool isOctaveZero )
{
    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    if( conf.getGaussMode() == Config::Fixed4 ) {
        dim3 block( 22, _levels-1 );
        dim3 grid;
        grid.x = grid_divide( width,  16 );
        grid.y = grid_divide( height, 16 );

        if( isOctaveZero ) {
            gauss::fixedSpan::octave
                <4,true>
                <<<grid,block,0,stream>>>
                ( oct_obj.getData(0)
                  , oct_obj.getData(1)
#ifndef MAKE_DOG_LATER
                  , oct_obj.getDogSurface( )
#endif // MAKE_DOG_LATER
                );
        } else {
            gauss::fixedSpan::octave
                <4,false>
                <<<grid,block,0,stream>>>
                ( oct_obj.getData(0)
                  , oct_obj.getData(1)
#ifndef MAKE_DOG_LATER
                  , oct_obj.getDogSurface( )
#endif // MAKE_DOG_LATER
                );
        }
    } else {
        dim3 block( 30, _levels-1 );
        dim3 grid;
        grid.x = grid_divide( width,  16 );
        grid.y = grid_divide( height, 16 );

        if( isOctaveZero ) {
            gauss::fixedSpan::octave
                <8,true>
                <<<grid,block,0,stream>>>
                ( oct_obj.getData(0)
                  , oct_obj.getData(1)
#ifndef MAKE_DOG_LATER
                  , oct_obj.getDogSurface( )
#endif // MAKE_DOG_LATER
                );
        } else {
            gauss::fixedSpan::octave
                <8,false>
                <<<grid,block,0,stream>>>
                ( oct_obj.getData(0)
                  , oct_obj.getData(1)
#ifndef MAKE_DOG_LATER
                  , oct_obj.getDogSurface( )
#endif // MAKE_DOG_LATER
                );
        }
    }
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_pyramid( const Config& conf, Image* base )
{
    hipError_t err;

#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    hipDeviceSynchronize();

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
      Octave& oct_obj   = _octaves[octave];

      if( conf.getGaussMode() == Config::Fixed4 || conf.getGaussMode() == Config::Fixed8 ) {
        hipStream_t stream = oct_obj.getStream(0);
        if( octave == 0 ) {
            horiz_from_input_image( conf, base, 0, stream, conf.getSiftMode() );
            vert_from_interm( octave, 0, stream );
            make_octave( conf, oct_obj, stream, true );
        } else {
            downscale_from_prev_octave( octave, 0, stream, conf.getSiftMode() );
            make_octave( conf, oct_obj, stream, false );
        }

        for( uint32_t level=0; level<_levels; level++ ) {
            hipEvent_t  ev     = oct_obj.getEventGaussDone(level);
            hipEvent_t  dog_ev = oct_obj.getEventDogDone(level);

            err = hipEventRecord( ev, stream );
            POP_CUDA_FATAL_TEST( err, "Could not record a Gauss done event: " );

            if( level > 0 ) {
#ifdef MAKE_DOG_LATER
                dog_from_blurred( octave, level, stream );
#endif // MAKE_DOG_LATER

                err = hipEventRecord( dog_ev, stream );
                POP_CUDA_FATAL_TEST( err, "Could not record a Gauss done event: " );
            }
        }
      } else {

        for( uint32_t level=0; level<_levels; level++ ) {
            const int width  = oct_obj.getWidth();
            const int height = oct_obj.getHeight();

            hipStream_t stream = oct_obj.getStream(level);
            hipEvent_t  ev     = oct_obj.getEventGaussDone(level);
            hipEvent_t  dog_ev = oct_obj.getEventDogDone(level);

            if( level == 0 )
            {
                if( octave == 0 )
                {
                    horiz_from_input_image( conf, base, 0, stream, conf.getSiftMode() );
                    vert_from_interm( octave, 0, stream );
                }
                else
                {
                    switch( _scaling_mode )
                    {
                    case Config::ScaleDirect :
                        // Does not work yet
                        horiz_from_input_image( conf, base, octave, stream, conf.getSiftMode() );
                        vert_from_interm( octave, level, stream );
                        break;
                    case Config::ScaleDefault :
                    default :
                        downscale_from_prev_octave( octave, level, stream, conf.getSiftMode() );
                        break;
                    }
                }
            }
            else
            {
                horiz_from_prev_level( octave, level, stream );
                vert_from_interm( octave, level, stream );
            }


            err = hipEventRecord( ev, stream );
            POP_CUDA_FATAL_TEST( err, "Could not record a Gauss done event: " );

            if( level > 0 ) {
                dog_from_blurred( octave, level, stream );

                err = hipEventRecord( dog_ev, stream );
                POP_CUDA_FATAL_TEST( err, "Could not record a Gauss done event: " );
            }
        }
      }
    }
}

} // namespace popsift

