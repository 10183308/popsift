#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <iomanip>
#include <iostream>
#include <unistd.h>
#ifndef __APPLE__
#include <malloc.h>
#endif
#include <stdlib.h>
#include <errno.h>
#include <hip/hip_math_constants.h>

#include <thrust/sequence.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/unique.h>

#include "sift_features.h"
#include "sift_extremum.h"
#include "common/debug_macros.h"

using namespace std;

namespace popsift {

/*************************************************************
 * FeaturesBase
 *************************************************************/

FeaturesBase::FeaturesBase( )
    : _num_ext( 0 )
    , _num_ori( 0 )
{ }

FeaturesBase::~FeaturesBase( )
{ }

/*************************************************************
 * FeaturesHost
 *************************************************************/

FeaturesHost::FeaturesHost( )
    : _ext( 0 )
    , _ori( 0 )
{ }

FeaturesHost::FeaturesHost( int num_ext, int num_ori )
    : _ext( 0 )
    , _ori( 0 )
{
    reset( num_ext, num_ori );
}

FeaturesHost::~FeaturesHost( )
{
    free( _ext );
    free( _ori );
}

#ifdef __APPLE__
static void* memalign( size_t alignment, size_t size )
{
    void* ret;
    int err = posix_memalign( &ret, alignment, size );
    if( err != 0 ) {
        errno = err;
        ret = 0;
    }
    return ret;
}
#endif

void FeaturesHost::reset( int num_ext, int num_ori )
{
    if( _ext != 0 ) { free( _ext ); _ext = 0; }
    if( _ori != 0 ) { free( _ori ); _ori = 0; }

    _ext = (Feature*)memalign( sysconf(_SC_PAGESIZE), num_ext * sizeof(Feature) );
    if( _ext == 0 ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
             << "    Failed to (re)allocate memory for downloading " << num_ext << " features" << endl;
        if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
        if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
        exit( -1 );
    }
    _ori = (Descriptor*)memalign( sysconf(_SC_PAGESIZE), num_ori * sizeof(Descriptor) );
    if( _ori == 0 ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
             << "    Failed to (re)allocate memory for downloading " << num_ori << " descriptors" << endl;
        if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
        if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
        exit( -1 );
    }

    setFeatureCount( num_ext );
    setDescriptorCount( num_ori );
}

void FeaturesHost::pin( )
{
    hipError_t err;
    err = hipHostRegister( _ext, getFeatureCount() * sizeof(Feature), 0 );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
             << "    Failed to register feature memory in CUDA." << endl
             << "    Features count: " << getFeatureCount() << endl
             << "    Memory size requested: " << getFeatureCount() * sizeof(Feature) << endl
             << "    " << hipGetErrorString(err) << endl;
    }
    err = hipHostRegister( _ori, getDescriptorCount() * sizeof(Descriptor), 0 );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
             << "    Failed to register descriptor memory in CUDA." << endl
             << "    Descriptors count: " << getDescriptorCount() << endl
             << "    Memory size requested: " << getDescriptorCount() * sizeof(Descriptor) << endl
             << "    " << hipGetErrorString(err) << endl;
    }
}

void FeaturesHost::unpin( )
{
    hipHostUnregister( _ext );
    hipHostUnregister( _ori );
}

void FeaturesHost::print( std::ostream& ostr, bool write_as_uchar ) const
{
    for( int i=0; i<size(); i++ ) {
        _ext[i].print( ostr, write_as_uchar );
    }
}

std::ostream& operator<<( std::ostream& ostr, const FeaturesHost& feature )
{
    feature.print( ostr, false );
    return ostr;
}

/*************************************************************
 * FeaturesDev
 *************************************************************/

FeaturesDev::FeaturesDev( )
    : _ext( 0 )
    , _ori( 0 )
    , _rev( 0 )
{ }

FeaturesDev::FeaturesDev( int num_ext, int num_ori )
    : _ext( 0 )
    , _ori( 0 )
    , _rev( 0 )
{
    reset( num_ext, num_ori );
}

FeaturesDev::~FeaturesDev( )
{
    hipFree( _ext );
    hipFree( _ori );
    hipFree( _rev );
}

void FeaturesDev::reset( int num_ext, int num_ori )
{
    if( _ext != 0 ) { hipFree( _ext ); _ext = 0; }
    if( _ori != 0 ) { hipFree( _ori ); _ori = 0; }
    if( _rev != 0 ) { hipFree( _rev ); _rev = 0; }

    _ext = popsift::cuda::malloc_devT<Feature>   ( num_ext, __FILE__, __LINE__ );
    _ori = popsift::cuda::malloc_devT<Descriptor>( num_ori, __FILE__, __LINE__ );
    _rev = popsift::cuda::malloc_devT<int>       ( num_ori, __FILE__, __LINE__ );

    POP_CHECK_NON_NULL( _ext, "failed to allocate Feature array" );
    POP_CHECK_NON_NULL( _ori, "failed to allocate Descriptor array" );
    POP_CHECK_NON_NULL( _rev, "failed to allocate Reverse Mapping array" );

    setFeatureCount( num_ext );
    setDescriptorCount( num_ori );
}

/*************************************************************
 * Feature
 *************************************************************/

void Feature::print( std::ostream& ostr, bool write_as_uchar ) const
{
    float sigval =  1.0f / ( scale * scale );

    for( int ori=0; ori<num_ori; ori++ ) {
        ostr << xpos << " " << ypos << " "
             << sigval << " 0 " << sigval << " ";
        if( write_as_uchar ) {
            for( int i=0; i<128; i++ ) {
                ostr << roundf(desc[ori]->features[i]) << " ";
            }
        } else {
            ostr << std::setprecision(3);
            for( int i=0; i<128; i++ ) {
                ostr << desc[ori]->features[i] << " ";
            }
            ostr << std::setprecision(6);
        }
        ostr << std::endl;
    }
}

std::ostream& operator<<( std::ostream& ostr, const Feature& feature )
{
    feature.print( ostr, false );
    return ostr;
}

} // namespace popsift

