#include "hip/hip_runtime.h"
/*
* Copyright 2017, Simula Research Laboratory
*
* This Source Code Form is subject to the terms of the Mozilla Public
* License, v. 2.0. If a copy of the MPL was not distributed with this
* file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include <float.h>


#include "sift_matching.h"
#include "assist.h"
#include "sift_conf.h"
#include "sift_octave.h"
#include "sift_pyramid.h"
#include "sift_extremum.h"
#include "popsift.h"
#include "common/debug_macros.h"
#include "hip/hip_runtime.h"
#include ""

namespace popsift {

Matching::Matching(Config& config)
 : config(config) {

}

Matching::~Matching() {

}

__device__
float calc_distance(float* a, float* b) {
    float sum = 0.0f;
    for (int i = 0; i < 128; i++) {
        float sub = a[i] - b[i];
        sum += sub*sub;
    }
    return sum;
}

__global__
void test(Descriptor* d_desc_a, int desc_a_count, Descriptor* d_desc_b, int desc_b_count, int* output) {
    int tid = threadIdx.x;
    
    for (int i = tid; i < desc_a_count; i += blockDim.x) {
        Descriptor& a = d_desc_a[i];
        float min1 = FLT_MAX, min2 = FLT_MAX;
        int min_index;

        for (int x = 0; x < desc_b_count; x++) {
            float dst = calc_distance(&a.features[0], &d_desc_b[x].features[0]);
            printf("%f", dst);
            if (dst < min1) {
                min2 = min1;
                min1 = dst;
                min_index = x;
            }
            else if (dst < min2) {
                min2 = dst;
            }
        }

        if (min1 / min2 < 0.64f) {
            output[i] = min_index;
        }
        else {
            output[i] = -1;
        }
    }
}

std::vector<int> Matching::Match(popsift::Descriptor* d_desc_a, size_t num_desc_a,
    popsift::Descriptor* d_desc_b, size_t num_desc_b) {
    
    int* d_result = popsift::cuda::malloc_devT<int>(num_desc_a, __FILE__, __LINE__);

    dim3 threadsPerBlock(1024);
    dim3 numBlocks(1);
    test <<<numBlocks, threadsPerBlock >>>(d_desc_a, num_desc_a, d_desc_b, num_desc_b, d_result);

    std::vector<int> h_result(num_desc_a);

    hipMemcpyAsync(h_result.data(), d_result, num_desc_a * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    return h_result;
}

}
