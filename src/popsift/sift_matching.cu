#include "hip/hip_runtime.h"
/*
* Copyright 2017, Simula Research Laboratory
*
* This Source Code Form is subject to the terms of the Mozilla Public
* License, v. 2.0. If a copy of the MPL was not distributed with this
* file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include <float.h>


#include "sift_matching.h"
#include "assist.h"
#include "sift_conf.h"
#include "sift_octave.h"
#include "sift_pyramid.h"
#include "sift_extremum.h"
#include "popsift.h"
#include "common/debug_macros.h"
#include "hip/hip_runtime.h"
#include ""

namespace popsift {

Matching::Matching(Config& config)
 : config(config) {

}

Matching::~Matching() {

}

template<typename T>
__device__
float calc_distance(T* a, T* b) {
    float sum = 0.0f;
    for (int i = 0; i < 128; i++) {
        float sub = a[i] - b[i];
        sum += sub*sub;
    }
    return sum;
}

__global__
void test(Descriptor* d_desc_a, int desc_a_count, Descriptor* d_desc_b, int desc_b_count, int* output) {
    int tid = threadIdx.x;
    
    for (int i = tid; i < desc_a_count; i += blockDim.x) {
        Descriptor& a = d_desc_a[i];
        float min1 = FLT_MAX, min2 = FLT_MAX;
        int min_index;

        for (int x = 0; x < desc_b_count; x++) {
            float dst = calc_distance<float>(&a.features[0], &d_desc_b[x].features[0]);
            //printf("%f", dst);
            if (dst < min1) {
                min2 = min1;
                min1 = dst;
                min_index = x;
            }
            else if (dst < min2) {
                min2 = dst;
            }
        }

        if (min1 / min2 < 0.64f) {
            output[i] = min_index;
        }
        else {
            output[i] = -1;
        }
    }
}

struct u8desc {
    unsigned char features[128];
};
__global__
void convert(Descriptor* d_desc, int count, u8desc* out) {
    int tid = threadIdx.x;

    for (int i = tid; i < count; i += blockDim.x) {
        for (int x = 0; x < 128; x++) {
            unsigned int tmp = d_desc[i].features[x] * 512;
            out[i].features[x] = tmp;
        }
    }
}

__global__
void u8_test(u8desc* d_desc_a, int desc_a_count, u8desc* d_desc_b, int desc_b_count, int* output) {
    int tid = threadIdx.x;

    for (int i = tid; i < desc_a_count; i += blockDim.x) {
        u8desc& a = d_desc_a[i];
        float min1 = FLT_MAX, min2 = FLT_MAX;
        int min_index;

        for (int x = 0; x < desc_b_count; x++) {
            float dst = calc_distance<unsigned char>(&a.features[0], &d_desc_b[x].features[0]);
            if (dst < min1) {
                min2 = min1;
                min1 = dst;
                min_index = x;
            }
            else if (dst < min2) {
                min2 = dst;
            }
        }

        if (min1 / min2 < 0.64f) {
            output[i] = min_index;
        }
        else {
            output[i] = -1;
        }
    }

}
std::vector<int> Matching::Match(popsift::Descriptor* d_desc_a, size_t num_desc_a,
    popsift::Descriptor* d_desc_b, size_t num_desc_b) {
    dim3 threadsPerBlock(1024);
    dim3 numBlocks(1);
    
    int* d_result = popsift::cuda::malloc_devT<int>(num_desc_a, __FILE__, __LINE__);

    std::cout << "starting test";
#if 1
        u8desc* a_u8desc = popsift::cuda::malloc_devT<u8desc>(num_desc_a, __FILE__, __LINE__);
        u8desc* b_u8desc = popsift::cuda::malloc_devT<u8desc>(num_desc_b, __FILE__, __LINE__);
        convert << <numBlocks, threadsPerBlock >> > (d_desc_a, num_desc_a, b_u8desc);
        convert << <numBlocks, threadsPerBlock >> > (d_desc_b, num_desc_b, b_u8desc);
        
        u8_test<<<numBlocks, threadsPerBlock >>>(a_u8desc, num_desc_a, b_u8desc, num_desc_b, d_result);
        std::vector<int> h_result(num_desc_a);

#else
    test << <numBlocks, threadsPerBlock >> >(d_desc_a, num_desc_a, d_desc_b, num_desc_b, d_result);
    std::vector<int> h_result(num_desc_a);
#endif
    hipMemcpyAsync(h_result.data(), d_result, num_desc_a * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::cout << "test done";
    
    
    return h_result;
}

}
