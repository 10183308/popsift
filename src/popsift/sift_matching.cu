#include "hip/hip_runtime.h"
/*
* Copyright 2017, Simula Research Laboratory
*
* This Source Code Form is subject to the terms of the Mozilla Public
* License, v. 2.0. If a copy of the MPL was not distributed with this
* file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include <float.h>


#include "sift_matching.h"
#include "assist.h"
#include "sift_conf.h"
#include "sift_octave.h"
#include "sift_pyramid.h"
#include "sift_extremum.h"
#include "popsift.h"
#include "common/debug_macros.h"
#include "hip/hip_runtime.h"
#include ""

namespace popsift {

__global__
void ConvertDescriptorsToU8(Descriptor* d_desc, int count, U8Descriptor* out) {
    int tid = threadIdx.x;
    for (int i = tid; i < count; i += blockDim.x) {
        for (int x = 0; x < 128; x++) {
            unsigned int tmp = d_desc[i].features[x] * 512;
            out[i].features[x] = tmp;
        }
    }
}

U8Descriptor* ConvertDescriptorsToU8(Descriptor* d_descriptors, int count)
{
    auto u8d_descriptors = popsift::cuda::malloc_devT<U8Descriptor>(count, __FILE__, __LINE__);
    int threads_per_block = 64;
    int block_count = (int)ceil(count / (float)threads_per_block);
    ConvertDescriptorsToU8<<<block_count, threads_per_block>>> (d_descriptors, count, u8d_descriptors);
    return u8d_descriptors;
}

Matching::Matching(Config& config)
 : config(config) {

}

Matching::~Matching() {

}

template<typename T>
__device__
float calc_distance_minret(const T* a, const T* b, const float* min2) {
    float sum = 0.0f;
    for (int i = 0; i < 128; i++) {
        float sub = a[i] - b[i];
        sum += sub*sub;
        if (sum > *min2) return sum;
    }
    return sum;
}

__device__ inline unsigned int swar_sub(unsigned int a, unsigned int b) {
    const unsigned int h = 0x80808080;
    return ((a | h) - (b & ~h)) ^ ((a ^ ~b) & h);
}

__device__ inline void update_sum(unsigned& sum, unsigned &d)
{
    unsigned v = d & 0xFF; d >>= 8;
    sum += v*v;
}

__device__
float calc_distance(const U8Descriptor& aa, const U8Descriptor& bb) {
    unsigned sum = 0;
#if 1
    for (int i = 0; i < 128; i++) {
        unsigned a = aa.features[i] - bb.features[i];
        sum += a*a;
    }
    return sum;
#else
    for (int i = 0; i < 32; i += 4) {
        unsigned a = *(const unsigned*)(aa.features + 4 * i);
        unsigned b = *(const unsigned*)(bb.features + 4 * i);
        unsigned d = swar_sub(a, b);
        update_sum(sum, d);
        update_sum(sum, d);
        update_sum(sum, d);
        update_sum(sum, d);
    }
    return sum;
    */
}

//~16+sec execution
__global__
void test(Descriptor* d_desc_a, int desc_a_count, Descriptor* d_desc_b, int desc_b_count, int* output) {
    int tid = threadIdx.x;
    
    for (int i = tid; i < desc_a_count; i += blockDim.x) {
        Descriptor& a = d_desc_a[i];
        float min1 = FLT_MAX, min2 = FLT_MAX;
        int min_index;

        for (int x = 0; x < desc_b_count; x++) {
            float dst = calc_distance_minret<float>(&a.features[0], &d_desc_b[x].features[0], &min2);
            //printf("%f", dst);
            if (dst < min1) {
                min2 = min1;
                min1 = dst;
                min_index = x;
            }
            else if (dst < min2) {
                min2 = dst;
            }
        }

        if (min1 / min2 < 0.64f) {
            output[i] = min_index;
        }
        else {
            output[i] = -1;
        }
    }
}



//~1.2sec execution 128x1
__global__
    void u8_test(U8Descriptor* d_desc_a, int desc_a_count, U8Descriptor* d_desc_b, int desc_b_count, int* output) {
        int tid = threadIdx.x + (blockIdx.x * blockDim.x);
        if (tid >= desc_a_count) return;

    __shared__ U8Descriptor a;
    a = d_desc_a[tid];
    float min1 = FLT_MAX, min2 = FLT_MAX;
    int min_index;
    const int cache_size = 128;
    const int skip_len = cache_size;// *2;
    __shared__ U8Descriptor cached[cache_size];

    for (int x = 0; x < desc_b_count-cache_size; x += cache_size) {
        //memcpy(cached[threadIdx.x].features, d_desc_b[threadIdx.x + x].features, sizeof(U8Descriptor));
        //cached[threadIdx.x] = d_desc_b[threadIdx.x + x];
        /*
        unsigned char* ap = &d_desc_b[x].features[0];
        unsigned char* bp = &cached[0].features[0];
        memcpy(bp + (threadIdx.x*skip_len), ap + (threadIdx.x*skip_len), sizeof(unsigned char) * skip_len);
        */
        /*
        for (int i = 0; i < cache_size; i++) {
            int dst = 0;
#if 0
            for (int s = 0; s < 128; s+=4) {
                unsigned int tmp = swar_sub(*(unsigned int*)&a.features[s], *(unsigned int*)&cached[i].features[s]);     
                #pragma unroll
                for (int k = 0; k < 4; k++) {
                    unsigned char v = (tmp >> (k * 8)) & 0xFF;
                    dst += v*v;
                }
            }
#else
            for (int s = 0; s < 128; s++) {
                unsigned char sub = a.features[s] - cached[i].features[s];
                dst += sub*sub;
            }
#endif
            if (dst < min1) {
                min2 = min1;
                min1 = dst;
                min_index = x;
            }
            else if (dst < min2) {
                min2 = dst;
            }
        }
        */
    }
    /*
    for (int x = 0; x < desc_b_count; x++) {
        float dst = calc_distance<unsigned char>(&a.features[0], &d_desc_b[x].features[0], &min2);
        if (dst < min1) {
            min2 = min1;
            min1 = dst;
            min_index = x;
        }
        else if (dst < min2) {
            min2 = dst;
        }
    }

    if (min1 / min2 < 0.64f) {
        output[tid] = min_index;
    }
    else {
        output[tid] = -1;
    }*/
}

//~3sec execution
__global__
void u8_test_shared(U8Descriptor* d_desc_a, int desc_a_count, U8Descriptor* d_desc_b, int desc_b_count, int* output) {
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    if (tid >= desc_a_count) return;

    __shared__ U8Descriptor b[32];
    U8Descriptor desc = d_desc_a[tid];
    float min1 = FLT_MAX, min2 = FLT_MAX;
    int min_index;

    for (int x = 0; x < desc_b_count; x += 32) {
        memcpy(b[threadIdx.x].features, d_desc_b[x + threadIdx.x].features, sizeof(U8Descriptor));

        for (int i = 0; i < 32; i++) {
            float dst = calc_distance_minret<unsigned char>(desc.features, b[i].features, &min2);
            if (dst < min1) {
                min2 = min1;
                min1 = dst;
                min_index = x + i;
            }
            else if (dst < min2) {
                min2 = dst;
            }
        }
    }

    if (min1 / min2 < 0.64f) {
        output[tid] = min_index;
    }
    else {
        output[tid] = -1;
    }
}


__device__ void reduce(float* vals) {
    int tid = threadIdx.x;
    if (tid > 15) return;
    vals[tid] += vals[tid + 16];
    vals[tid] += vals[tid + 8];
    vals[tid] += vals[tid + 4];
    vals[tid] += vals[tid + 2];
    vals[tid] += vals[tid + 1];
}

//needs 32x1 blocksize ~5sec execution
__global__
void char_32thread_1desc(U8Descriptor* d_desc_a, int desc_a_count, U8Descriptor* d_desc_b, int desc_b_count, int* output) {
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    if (tid >= desc_a_count) return;

    float min1 = FLT_MAX, min2 = FLT_MAX;
    int min_index;
    
    U8Descriptor a;
    memcpy(&a.features[threadIdx.x * 4], &d_desc_a[tid].features[threadIdx.x * 4], sizeof(unsigned char) * 4);

    __shared__ U8Descriptor b[32];
    __shared__ float sums[32];

    //could it be benefitial if different blocks started on different B's?
    for (int i = 0; i < desc_b_count; i+=32) {                   
        //memcpy(&b.features[threadIdx.x * 4], &d_desc_b[i].features[threadIdx.x * 4], sizeof(unsigned char) * 4);
        memcpy(&b[threadIdx.x].features[0], &d_desc_b[threadIdx.x + i].features[0], sizeof(U8Descriptor));


        sums[threadIdx.x] = 0.0f;
        for (int x = threadIdx.x*4; x < 128; x++) {
            float sub;// = a.features[x] - b.features[x];
            sub = sub*sub;
            sums[threadIdx.x] += sub;
        }
        __syncthreads();
        reduce(&sums[0]);
        if (threadIdx.x == 0) {
            if (sums[0] < min1) {
                min2 = min1;
                min1 = sums[0];
                min_index = i;
            }
            else if (sums[0] < min2) {
                min2 = sums[0];
            }
        }
    }
    if (threadIdx.x == 0) {
        if (min1 / min2 < 0.64f) {
            output[tid] = min_index;
        }
        else {
            output[tid] = -1;
        }
    }
}


struct MinDiff {
    float m[2];
    int idx;
};

__global__
void char_32x32(U8Descriptor* d_desc_a, int desc_a_count, 
    U8Descriptor* d_desc_b, int desc_b_count, int* output) {

    
    __shared__ U8Descriptor a[32]; //4096B
    __shared__ U8Descriptor b[32]; //4096B
    __shared__ MinDiff c[32]; //check if enough registers to remove shared

    int ltid = threadIdx.y * blockDim.x + threadIdx.x; // 0, 1023
    int gtid = ltid + blockIdx.x + (blockIdx.y*gridDim.x);
    //if (blockDim.x*blockIdx.x + threadIdx.y > desc_a_count) return; //add with ceil in blockdim on launch
    
    memcpy(&a[threadIdx.y].features[threadIdx.x * 4], &d_desc_a[blockIdx.x*blockDim.x].features[threadIdx.x * 4], sizeof(unsigned));
    memcpy(&b[threadIdx.y].features[threadIdx.x * 4], &d_desc_b[blockIdx.x*blockDim.x].features[threadIdx.x * 4], sizeof(unsigned));

    *(unsigned int*)(&a[threadIdx.y].features[threadIdx.x * 4]) = *(unsigned int*)(&d_desc_a[blockIdx.x*blockDim.x].features[threadIdx.x * 4]);
    *(unsigned int*)(&b[threadIdx.y].features[threadIdx.x * 4]) = *(unsigned int*)(&d_desc_b[blockIdx.y*blockDim.y].features[threadIdx.x * 4]);
    __syncthreads();

    //float dst = calc_distance(a[threadIdx.x], b[threadIdx.y]);
    /*
    if (dst < c[threadIdx.y].m[0]) {
        c[threadIdx.y].m[1] = c[threadIdx.y].m[0];
        c[threadIdx.y].m[0]  = dst;
        c[threadIdx.y].idx = gtid;
    }
    else if (dst < c[threadIdx.y].m[1]) {
        c[threadIdx.y].m[1] = dst;
    }
    */

    //memcpy(&a[threadIdx.y].features[threadIdx.x], &d_desc_a[]

}


__global__ 
void distance_test(int* output) {
    int tid = threadIdx.y * blockDim.x + threadIdx.x; // 0, 1023
    int gtid = tid + blockIdx.x + (blockIdx.y*gridDim.x);
    __shared__ U8Descriptor a;
    __shared__ U8Descriptor b;
    if (tid < 128) {
        a.features[tid] = tid;
        b.features[tid] = tid;
    }
    float dst = calc_distance(a, b);
    if(tid==0)
        output[gtid] = (int)dst;
    if (gtid > 14000) printf("asd");

}

std::vector<int> Matching::Match(popsift::Descriptor* d_desc_a, size_t num_desc_a,
    popsift::Descriptor* d_desc_b, size_t num_desc_b) {
        
    

    //dim3 numBlocks((int)ceil(num_desc_a / (float)(threadsPerBlock.x*threadsPerBlock.y)));
    //dim3 numBlocks((int)ceil(num_desc_a / (float)threadsPerBlock.y));
    int* d_result = popsift::cuda::malloc_devT<int>(num_desc_a, __FILE__, __LINE__);

    std::cout << "starting test\n";
#if 1

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(num_desc_a / threadsPerBlock.x, num_desc_a / threadsPerBlock.y);
    distance_test<<<numBlocks, threadsPerBlock >>>(d_result);
#endif

#if 0
    U8Descriptor* a_U8Descriptor = ConvertDescriptorsToU8(d_desc_a, num_desc_a);
    U8Descriptor* b_U8Descriptor = ConvertDescriptorsToU8(d_desc_b, num_desc_b);
#endif

#if 0
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(num_desc_a / threadsPerBlock.x, num_desc_b / threadsPerBlock.y); //need ceiling
    char_32x32<<<numBlocks,threadsPerBlock>>>(a_U8Descriptor, num_desc_a, b_U8Descriptor, num_desc_b, d_result);
#endif

#if 0
    dim3 threadsPerBlock(128, 1);
    dim3 numBlocks(num_desc_a / threadsPerBlock.x); //need ceiling
    u8_test<< <numBlocks, threadsPerBlock >> >(a_U8Descriptor, num_desc_a, b_U8Descriptor, num_desc_b, d_result);
#endif

    //char_32thread_1desc <<<numBlocks, threadsPerBlock >>>(a_U8Descriptor, num_desc_a, b_U8Descriptor, num_desc_b, d_result);
    std::vector<int> h_result(num_desc_a);

    //hipMemcpyAsync(h_result.data(), d_result, num_desc_a * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::cout << "test done";
    
    
    return h_result;
}

}
