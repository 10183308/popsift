#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define WITH_SURFACE

using namespace std;

__global__
void testPrintArray( hipTextureObject_t obj, int width, int height, int levels )
{
    for( int z=0; z<levels; z++ ) {
        for( int y=0; y<height; y++ ) {
            for( int x=0; x<width; x++ ) {
                float v = tex2DLayered<float>( obj, x, y, z );
                int d = v;
                printf( "%d ", d );
            }
            printf("\n");
        }
        printf("\n");
    }
}

#ifdef WITH_SURFACE
__global__
void writeToSurface( hipSurfaceObject_t obj, int width, int height, int levels )
{
    for( int z=threadIdx.z; z<levels; z += blockDim.z ) {
        for( int y=threadIdx.y; y<height; y += blockDim.y ) {
            for( int x=threadIdx.x; x<width; x += blockDim.x ) {
                float v = x - 10*y + 100*z;
                surf2DLayeredwrite( v,
                                    obj,
                                    x*sizeof(float), y, z,
                                    hipBoundaryModeZero );
                                    // x, y, z,
                                    // cudaBoundaryModeTrap );
            }
        }
    }
}
#endif // WITH_SURFACE

int main( )
{
    int width  = 10;
    int height = 4;
    int levels = 2;

    hipArray_t           _dog_3d;
    hipChannelFormatDesc _dog_3d_desc;
    hipExtent            _dog_3d_ext;
    hipTextureObject_t   _dog_3d_tex;
#ifdef WITH_SURFACE
    hipSurfaceObject_t   _dog_3d_surf;
#endif // WITH_SURFACE

    _dog_3d_desc.f = hipChannelFormatKindFloat;
    _dog_3d_desc.x = 32;
    _dog_3d_desc.y = 0;
    _dog_3d_desc.z = 0;
    _dog_3d_desc.w = 0;

    _dog_3d_ext.width  = width; // for cudaMalloc3DArray, width in elements
    _dog_3d_ext.height = height;
    _dog_3d_ext.depth  = levels;

    hipError_t err;
    err = hipMalloc3DArray( &_dog_3d,
                             &_dog_3d_desc,
                             _dog_3d_ext,
                             hipArrayLayered | hipArraySurfaceLoadStore );
    if( err != hipSuccess ) {
        cerr << "CUDA malloc 3D array failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA malloc 3D array worked" << endl;
    }

    hipResourceDesc dog_res_desc;
    dog_res_desc.resType         = hipResourceTypeArray;
    dog_res_desc.res.array.array = _dog_3d;

    hipTextureDesc      dog_tex_desc;
    memset( &dog_tex_desc, 0, sizeof(hipTextureDesc) );
    dog_tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    dog_tex_desc.addressMode[0]   = hipAddressModeClamp;
    dog_tex_desc.addressMode[1]   = hipAddressModeClamp;
    dog_tex_desc.addressMode[2]   = hipAddressModeClamp;
    dog_tex_desc.readMode         = hipReadModeElementType; // read as float
    dog_tex_desc.filterMode       = hipFilterModePoint; // no interpolation

    err = hipCreateTextureObject( &_dog_3d_tex,
                                   &dog_res_desc,
                                   &dog_tex_desc, 0 );
    if( err != hipSuccess ) {
        cerr << "CUDA create texture failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA create texture succeeded" << endl;
    }

    // cudaResourceDesc dog_res_desc;
    // dog_res_desc.resType         = cudaResourceTypeArray;
    // dog_res_desc.res.array.array = _dog_3d;

#ifdef WITH_SURFACE
    err = hipCreateSurfaceObject( &_dog_3d_surf, &dog_res_desc );
    if( err != hipSuccess ) {
        cerr << "CUDA create surface failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA create surface succeeded" << endl;
    }
#endif // WITH_SURFACE

    float* array;
    err = hipHostMalloc( &array, width*height*levels*sizeof(float) , hipHostMallocDefault);
    if( err != hipSuccess ) {
        cerr << "CUDA malloc host failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA malloc host succeeded" << endl;
    }

    cerr << "== host-sided printing of array ==" << endl;
    for( int z=0; z<levels; z++ ) {
        for( int y=0; y<height; y++ ) {
            for( int x=0; x<width; x++ ) {
                int val = x + 10*y + 100*z;
                array[(z*height + y)*width + x] = val;
                cerr << val << " ";
            }
            cerr << endl;
        }
        cerr << endl;
    }

#if 0
    // This does not work
    err = cudaMemcpyToArray( _dog_3d,
                             0, 0,
                             array,
                             10*4*2*sizeof(float),
                             // 10*4*sizeof(float),
                             cudaMemcpyHostToDevice );
#endif
    hipMemcpy3DParms s = { 0 };
    s.srcPtr = make_hipPitchedPtr( array, 10*sizeof(float), 10, 4 );
    s.dstArray = _dog_3d;
    s.extent = make_hipExtent( width, height, levels );
    s.kind = hipMemcpyHostToDevice;
    err = hipMemcpy3D( &s );

    if( err != hipSuccess ) {
        cerr << "CUDA memcpy 3D failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA memcpy 3D succeeded" << endl;
    }

    hipDeviceSynchronize();

    cerr << "== CUDA-sided printing of array ==" << endl;

    testPrintArray
        <<<1,1>>>
        ( _dog_3d_tex, 10, 4, 2 );
        // ( _dog_3d_tex, width, height, levels );

    err = hipGetLastError( );
    if( err != hipSuccess ) {
        cerr << "testPrintArray: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "testPrintArray succeeded" << endl;
    }

#ifdef WITH_SURFACE
    cerr << "== CUDA-sided writing to surface" << endl;

    dim3 block( 8, 8, 1 );
    writeToSurface
        <<<1,block>>>
        ( _dog_3d_surf, width, height, levels );

    err = hipGetLastError( );
    if( err != hipSuccess ) {
        cerr << "writeToSurface: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "writeToSurface succeeded" << endl;
    }

    cerr << "== CUDA-sided printing of array ==" << endl;

    testPrintArray
        <<<1,1>>>
        ( _dog_3d_tex, width, height, levels );

    err = hipGetLastError( );
    if( err != hipSuccess ) {
        cerr << "testPrintArray: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "testPrintArray succeeded" << endl;
    }
#endif // WITH_SURFACE

    hipDeviceSynchronize();

    memset( array, 0, width*height*levels*sizeof(float) );

    memset( &s, 0, sizeof(hipMemcpy3DParms) );
    s.srcArray = _dog_3d;
    s.dstPtr = make_hipPitchedPtr( array, width*sizeof(float), width, height );
    s.extent = make_hipExtent( width, height, levels );
    s.kind = hipMemcpyDeviceToHost;
    err = hipMemcpy3D( &s );

    if( err != hipSuccess ) {
        cerr << "CUDA memcpy 3D failed: " << hipGetErrorString(err) << endl;
        exit( -1 );
    } else {
        cerr << "CUDA memcpy 3D succeeded" << endl;
    }

    cerr << "== host-sided printing of array ==" << endl;
    for( int z=0; z<2; z++ ) {
        for( int y=0; y<4; y++ ) {
            for( int x=0; x<10; x++ ) {
                cerr << array[(z*height + y)*width + x] << " ";
            }
            cerr << endl;
        }
        cerr << endl;
    }

    err = hipHostFree( array );
    if( err != hipSuccess ) {
        cerr << "CUDA free host failed" << endl;
        exit( -1 );
    } else {
        cerr << "CUDA free host success" << endl;
    }

#ifdef WITH_SURFACE
    err = hipDestroySurfaceObject( _dog_3d_surf );
    if( err != hipSuccess ) {
        cerr << "CUDA destroy surface failed" << endl;
        exit( -1 );
    } else {
        cerr << "CUDA destroy surface success" << endl;
    }
#endif // WITH_SURFACE

    err = hipDestroyTextureObject( _dog_3d_tex );
    if( err != hipSuccess ) {
        cerr << "CUDA destroy texture failed" << endl;
        exit( -1 );
    } else {
        cerr << "CUDA destroy texture success" << endl;
    }

    err = hipFreeArray( _dog_3d );
    if( err != hipSuccess ) {
        cerr << "CUDA array free failed" << endl;
        exit( -1 );
    } else {
        cerr << "CUDA array free success" << endl;
    }
}

