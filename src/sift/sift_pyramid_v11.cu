#include "hip/hip_runtime.h"
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "gauss_filter.h"
#include "debug_macros.h"
#include "assist.h"
#include "clamp.h"

#include <iostream>
#include <stdio.h>

/*************************************************************
 * V11: device side
 *************************************************************/

namespace popart {

#ifdef DEBUG_SEARCH_FOR_NANS
__global__
void post_gauss_validate_data_layer( int octave, int level, Plane2D_float layer )
{
    const int width  = layer.getWidth();
    const int height = layer.getHeight();
    for( int y=0; y<height; y++ ) {
        for( int x=0; x<width; x++ ) {
            if( isnan( layer.ptr(y)[x] ) ) {
                printf( "POST GAUSS: Found a NAN value in blur layer octave %d level %d at (%d,%d)\n", octave, level, x,y );
                return;
            }
            if( isinf( layer.ptr(y)[x] ) ) {
                printf( "POST GAUSS: Found an INF value in blur layer octave %d level %d at (%d,%d)\n", octave, level, x,y );
                return;
            }
        }
    }
}
#endif // DEBUG_SEARCH_FOR_NANS

namespace gauss {
namespace v11 {

__global__
void horiz_tex_128x1( hipTextureObject_t src_data,
                      Plane2D_float       dst_data,
                      int                 level )
{
    const float dst_w  = dst_data.getWidth();
    const float dst_h  = dst_data.getHeight();
    const float read_y = ( blockIdx.y + 0.5f ) / dst_h;

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];
        const float read_x_l = ( off_x - offset );
        const float  v1 = tex2D<float>( src_data, ( read_x_l + 0.5f ) / dst_w, read_y );
        out += ( v1 * g );

        const float read_x_r = ( off_x + offset );
        const float  v2 = tex2D<float>( src_data, ( read_x_r + 0.5f ) / dst_w, read_y );
        out += ( v2 * g );
    }
    const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    const float read_x = off_x;
    const float v3 = tex2D<float>( src_data, ( read_x + 0.5f ) / dst_w, read_y );
    out += ( v3 * g );

#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( out ) ) printf( "horiz_tex_128x1 yielded NAN\n" );
    if( isinf( out ) ) printf( "horiz_tex_128x1 yielded INF\n" );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(blockIdx.y)[off_x] = out;
}

__global__
void horiz_tex_128x1_initial_blur( hipTextureObject_t src_data,
                                          Plane2D_float       dst_data )
{
    const float dst_w  = dst_data.getWidth();
    const float dst_h  = dst_data.getHeight();
    const float read_y = ( blockIdx.y + 0.5f ) / dst_h;

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        const float& g  = popart::d_gauss_filter_initial_blur[offset];
        const float read_x_l = ( off_x - offset );
        const float  v1 = tex2D<float>( src_data, ( read_x_l + 0.5f ) / dst_w, read_y );
        out += ( v1 * g );

        const float read_x_r = ( off_x + offset );
        const float  v2 = tex2D<float>( src_data, ( read_x_r + 0.5f ) / dst_w, read_y );
        out += ( v2 * g );
    }
    const float& g  = popart::d_gauss_filter_initial_blur[0];
    const float read_x = off_x;
    const float v3 = tex2D<float>( src_data, ( read_x + 0.5f ) / dst_w, read_y );
    out += ( v3 * g );

#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( out ) ) printf( "horiz_tex_128x1_initial_blur yielded NAN\n" );
    if( isinf( out ) ) printf( "horiz_tex_128x1_initial_blur yielded INF\n" );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(blockIdx.y)[off_x] = out;
}


__global__
void horiz_128x1( hipTextureObject_t src_data,
                  Plane2D_float       dst_data,
                  int                 level )
{
    const int dst_w = dst_data.getWidth();

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];
        const float  v1 = tex2D<float>( src_data, off_x - offset + 0.5f, blockIdx.y + 0.5f );
        out += ( v1 * g );

        const float  v2 = tex2D<float>( src_data, off_x + offset + 0.5f, blockIdx.y + 0.5f );
        out += ( v2 * g );
    }
    const float& g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    const float v3 = tex2D<float>( src_data, off_x+0.5f, blockIdx.y+0.5f );
    out += ( v3 * g );

#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( out ) ) printf( "horiz_128x1 yielded NAN\n" );
    if( isinf( out ) ) printf( "horiz_128x1 yielded INF\n" );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(blockIdx.y)[off_x] = out;
}

__global__
void get_by_2( hipTextureObject_t src_data,
               Plane2D_float       dst_data,
               int level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const float val = tex2D<float>( src_data, 2.0f * idx + 1.0f, 2.0f * idy + 1.0f );
#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( val ) ) printf( "get_by_2 yielded NAN\n" );
    if( isinf( val ) ) printf( "get_by_2 yielded INF\n" );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(idy)[idx] = val;
}

__global__
void get_by_2_opencv( Plane2D_float src_data,
                      Plane2D_float       dst_data,
                      int level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();
    const int read_x = clamp( idx << 1, 0, src_w );
    const int read_y = clamp( idy << 1, 0, src_h );

    const float val = src_data.ptr(read_y)[read_x];

    dst_data.ptr(idy)[idx] = val;
}

__global__
void horiz_by_2( hipTextureObject_t src_data,
                 Plane2D_float       dst_data,
                 int level )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idx = threadIdx.x - offset;
        // add +1.0f because we must shift by 0.5 pixels upscaled by 2 in the previous octave
        val = tex2D<float>( src_data, 2 * ( block_x + idx ) + 1.0f, 2 * ( block_y + idy ) + 1.0f );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ) + 1.0f, 2 * ( block_y + idy ) + 1.0f );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, 2 * ( block_x + idx ) + 1.0f, 2 * ( block_y + idy ) + 1.0f );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( out ) ) printf( "horiz_by_2 yielded NAN\n" );
    if( isinf( out ) ) printf( "horiz_by_2 yielded INF\n" );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(idy)[idx] = out;
}

__global__
void vert( hipTextureObject_t src_data,
           Plane2D_float       dst_data,
           int level )
{
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

#ifdef GAUSS_INTERM_FILTER_MODE_POINT
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );
#else // not GAUSS_INTERM_FILTER_MODE_POINT
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[level*GAUSS_ALIGN + offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[level*GAUSS_ALIGN];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
    out += ( val * g );
#endif // not GAUSS_INTERM_FILTER_MODE_POINT

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( out ) ) printf( "vert yielded NAN at (%d,%d)\n", idx, idy );
    if( isinf( out ) ) printf( "vert yielded INF at (%d,%d)\n", idx, idy );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(idy)[idx] = out;
}

__global__
void vert_initial_blur( hipTextureObject_t src_data,
                        Plane2D_float       dst_data )
{
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();

    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

#ifdef GAUSS_INTERM_FILTER_MODE_POINT
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter_initial_blur[offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter_initial_blur[0];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );
#else // not GAUSS_INTERM_FILTER_MODE_POINT
    for( int offset = GAUSS_SPAN; offset>0; offset-- ) {
        g  = popart::d_gauss_filter_initial_blur[offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter_initial_blur[0];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
    out += ( val * g );
#endif // not GAUSS_INTERM_FILTER_MODE_POINT

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

#ifdef DEBUG_SEARCH_FOR_NANS
    if( isnan( out ) ) printf( "vert_initial_blur yielded NAN\n" );
    if( isinf( out ) ) printf( "vert_initial_blur yielded INF\n" );
#endif // DEBUG_SEARCH_FOR_NANS
    dst_data.ptr(idy)[idx] = out;
}


#if 1
__global__
void make_dog( Plane2D_float       this_data,
               Plane2D_float       top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const int cols = this_data.getWidth();
    const int rows = this_data.getHeight();
    
    const int r_x = clamp( idx, cols );
    const int r_y = clamp( idy, rows );

    const float b = this_data.ptr(r_y)[r_x];
    const float a = top_data .ptr(r_y)[r_x];
    const float c = b - a; // c = fabs( a - b );

    surf2DLayeredwrite( c, dog_data, idx*4, idy, level, hipBoundaryModeZero );
}
#else
__global__
void make_dog( hipTextureObject_t this_data,
               hipTextureObject_t top_data,
               hipSurfaceObject_t dog_data,
               int                 level )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const float b = tex2D<float>( this_data, idx, idy );
    const float a = tex2D<float>( top_data, idx, idy );
    const float c = b - a; // c = fabs( a - b );

    surf2DLayeredwrite( c, dog_data, idx*4, idy, level, hipBoundaryModeZero );
}
#endif

} // namespace gauss
} // namespace v11

__host__
inline void Pyramid::horiz_from_upscaled_orig_tex( hipTextureObject_t src_data, int octave, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* I believe that waiting is not necessary because image is upscaled
     * in default stream */

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    gauss::v11::horiz_tex_128x1
        <<<grid,block,0,stream>>>
        ( src_data,
          oct_obj.getIntermediateData( ),
          0 ); // level is always 0
}

__host__
inline void Pyramid::horiz_from_upscaled_orig_tex_initial_blur( hipTextureObject_t src_data, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[0];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* I believe that waiting is not necessary because image is upscaled
     * in default stream */

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    gauss::v11::horiz_tex_128x1_initial_blur
        <<<grid,block,0,stream>>>
        ( src_data,
          oct_obj.getIntermediateData( ) );
}

#define PREV_LEVEL 3
// #define PREV_LEVEL 5

__host__
inline void Pyramid::downscale_from_prev_octave( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];
    Octave& prev_oct_obj = _octaves[octave-1];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* Necessary to wait for a lower level in the previous octave */
    hipEvent_t ev = prev_oct_obj.getEventGaussDone( _levels-PREV_LEVEL );
    hipStreamWaitEvent( stream, ev, 0 );

    dim3 h_block( 64, 2 );
    dim3 h_grid;
    h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
    h_grid.y = (unsigned int)grid_divide( height, h_block.y );

#ifdef USE_OPENCV_INTERPRETATION
    gauss::v11::get_by_2_opencv
        <<<h_grid,h_block,0,stream>>>
        ( prev_oct_obj.getData( _levels-PREV_LEVEL ),
          oct_obj.getData( level ),
          level );
#else // not USE_OPENCV_INTERPRETATION
    gauss::v11::get_by_2
        <<<h_grid,h_block,0,stream>>>
        ( prev_oct_obj._data_tex[ _levels-PREV_LEVEL ],
          oct_obj.getData( level ),
          level );
#endif // not USE_OPENCV_INTERPRETATION
}

__host__
inline void Pyramid::downscale_from_prev_octave_and_horiz_blur( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj  = _octaves[octave];
    Octave& prev_oct_obj  = _octaves[octave-1];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* Necessary to wait for a lower level in the previous octave */
    hipEvent_t ev = prev_oct_obj.getEventGaussDone( _levels-PREV_LEVEL );
    hipStreamWaitEvent( stream, ev, 0 );

    dim3 h_block( 64, 2 );
    dim3 h_grid;
    h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
    h_grid.y = (unsigned int)grid_divide( height, h_block.y );

    gauss::v11::horiz_by_2
        <<<h_grid,h_block,0,stream>>>
        ( prev_oct_obj._data_tex[ _levels-PREV_LEVEL ],
          oct_obj.getIntermediateData( ),
          level );
}

__host__
inline void Pyramid::horiz_from_prev_level( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    /* waiting for previous level in same octave */
    hipEvent_t ev = oct_obj.getEventGaussDone( level-1 );
    hipStreamWaitEvent( stream, ev, 0 );

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;
    gauss::v11::horiz_128x1
        <<<grid,block,0,stream>>>
        ( oct_obj._data_tex[ level-1 ],
          oct_obj.getIntermediateData( ),
          level );
}

__host__
inline void Pyramid::vert_from_interm( int octave, int level, hipStream_t stream )
{
    Octave& oct_obj = _octaves[octave];

    /* waiting for any events is not necessary, it's in the same stream as horiz
     */

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 64, 2 );
    dim3 grid;
    grid.x = (unsigned int)grid_divide( width,  block.x );
    grid.y = (unsigned int)grid_divide( height, block.y );

    gauss::v11::vert
        <<<grid,block,0,stream>>>
        ( oct_obj._interm_data_tex,
          oct_obj.getData( level ),
          level );
}

__host__
inline void Pyramid::vert_from_interm_initial_blur( hipStream_t stream )
{
    Octave& oct_obj = _octaves[0];

    /* waiting for any events is not necessary, it's in the same stream as horiz
     */

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 64, 2 );
    dim3 grid;
    grid.x = (unsigned int)grid_divide( width,  block.x );
    grid.y = (unsigned int)grid_divide( height, block.y );

    gauss::v11::vert_initial_blur
        <<<grid,block,0,stream>>>
        ( oct_obj._interm_data_tex,
          oct_obj.getData( 0 ) );
}

__host__
inline void Pyramid::dog_from_blurred( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 128, 2 );
    dim3 grid;
    grid.x = grid_divide( width,  block.x );
    grid.y = grid_divide( height, block.y );

    /* waiting for lower level is automatic, it's in the same stream.
     * waiting for upper level is necessary, it's in another stream.
     */
    hipEvent_t  ev     = oct_obj.getEventGaussDone( level-1 );
    hipStreamWaitEvent( stream, ev, 0 );

#if 1
    gauss::v11::make_dog
        <<<grid,block,0,stream>>>
        ( oct_obj.getData(level),
          oct_obj.getData(level-1),
          oct_obj.getDogSurface( ),
          level-1 );
#else
    gauss::v11::make_dog
        <<<grid,block,0,stream>>>
        ( oct_obj._data_tex[level],
          oct_obj._data_tex[level-1],
          oct_obj.getDogSurface( ),
          level-1 );
#endif
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_v11( Image* base )
{
    hipError_t err;

#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    hipDeviceSynchronize();

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
        Octave& oct_obj   = _octaves[octave];

        for( uint32_t level=0; level<_levels; level++ ) {
            const int width  = oct_obj.getWidth();
            const int height = oct_obj.getHeight();

            hipStream_t stream = oct_obj.getStream(level);
            hipEvent_t  ev     = oct_obj.getEventGaussDone(level);
            hipEvent_t  dog_ev = oct_obj.getEventDogDone(level);

            if( level == 0 )
            {
                if( octave == 0 )
                {
                    hipTextureObject_t& tex = base->getUpscaledTexture();
                    if( _assume_initial_blur ) {
                        horiz_from_upscaled_orig_tex_initial_blur( tex, stream );
                        vert_from_interm_initial_blur( stream );
                    } else {
                        horiz_from_upscaled_orig_tex( tex, octave, stream );
                        vert_from_interm( octave, level, stream );
                    }
                }
                else 
                {
                    switch( _scaling_mode )
                    {
                    case Config::DirectDownscaling :
                        {
                            hipTextureObject_t& tex = base->getUpscaledTexture();
                            horiz_from_upscaled_orig_tex( tex, octave, stream );
                            vert_from_interm( octave, level, stream );
                        }
                        break;
                    case Config::IndirectUnfilteredDownscaling :
                        downscale_from_prev_octave( octave, level, stream );
                        break;
                    case Config::IndirectDownscaling :
                        downscale_from_prev_octave_and_horiz_blur( octave, level, stream );
                        vert_from_interm( octave, level, stream );
                        break;
                    default :
                        cerr << __FILE__ << ":" << __LINE__ << ": unknown scaling mode" << endl;
                        break;
                    }
                }
            }
            else
            {
                horiz_from_prev_level( octave, level, stream );
                vert_from_interm( octave, level, stream );
            }

            err = hipEventRecord( ev, stream );
            POP_CUDA_FATAL_TEST( err, "Could not record a Gauss done event: " );

            if( level > 0 ) {
                dog_from_blurred( octave, level, stream );

                err = hipEventRecord( dog_ev, stream );
                POP_CUDA_FATAL_TEST( err, "Could not record a Gauss done event: " );
            }
#ifdef DEBUG_SEARCH_FOR_NANS
            post_gauss_validate_data_layer
                <<<1,1,0,stream>>>
                ( octave, level, oct_obj.getData( level ) );
#endif // DEBUG_SEARCH_FOR_NANS
        }
    }
}

} // namespace popart

